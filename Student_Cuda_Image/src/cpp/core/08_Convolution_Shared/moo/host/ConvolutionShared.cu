#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <assert.h>

#include "ConvolutionShared.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void convolutionKernelShared(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t);
extern __global__ void colorToGreyShared(uchar4* ptrDevPixels, int w, int h);
extern __global__ void findMinMaxShared(uchar4* ptrDevPixels, uchar* ptrDevResult,int w, int h);
extern __global__ void affineTransformShared(uchar4* ptrDevPixels, float a, float b, int w, int h, int offset);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

ConvolutionShared::ConvolutionShared()
    {
    // Inputs
    this->w = 640;
    this->h = 360;
    this->k = 9;

    // Tools
    this->dg = dim3(16, 1, 1); // disons a optimiser
    this->db = dim3(128, 1, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : Convolution CUDA Shared";
    //this->videoPath="/media/Data/Video/nasaFHD_short.avi";
    //this->videoTitle="NASA FHD SHORT";

    this->videoPath="/media/Data/Video/neilPryde.avi";
    this->videoTitle="neilPryde";

    size_t sizeOctets=w*h*sizeof(uchar4);
    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostMemory), sizeOctets,hipHostMallocDefault ) );

    this->captureur = new CaptureVideo(videoPath,videoTitle,this->ptrHostMemory);

    const int N = k*k;
    sizeConvSM = N*sizeof(float);

    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostNoyau), N*sizeof(float),hipHostMallocDefault ) );

    fillDetourage(ptrHostNoyau);

    HANDLE_ERROR(hipMalloc((void **)&ptrDeviceNoyau,N*sizeof(float)));
    HANDLE_ERROR(hipMemcpy(ptrDeviceNoyau,ptrHostNoyau,N*sizeof(float),hipMemcpyHostToDevice));


    sizeSM = 2*db.x*sizeof(uchar);
    sizeResult=2*dg.x*sizeof(uchar);
    ptrHostResult = (uchar*) malloc(sizeResult);

    // first contains min of each block, then max of each block
    HANDLE_ERROR(hipMalloc((void**) &ptrDevResult, sizeResult));

    //cout << endl<<"[CBI] Convolution dt =" << dt << endl;
    }

ConvolutionShared::~ConvolutionShared()
    {
    free(ptrHostResult);
    hipFree(ptrDevResult);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Noyau detourage 9x9
 */
void ConvolutionShared::fillDetourage(float* ptrNoyau)
    {
    const int N = 9*9;
    // Tab auto temporaire
    float tab[N] =
	    {
		    0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.6063, 1.2765, 0.4061, -6.4829, -12.7462, -6.4829, 0.4061, 1.2765, 0.6063, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828
	    };

    for (int i = 1; i <= N; i++)
	{
	ptrNoyau[i - 1] = tab[i - 1] / 100.0f;
	}
    }

/**
 * Override
 * Call periodicly by the API
 */
void ConvolutionShared::animationStep()
    {
    //t+=dt;
    }

/**
 * Override
 */
void ConvolutionShared::runGPU(uchar4* ptrDevPixels)
    {
    Chronos chrono;
    Mat matImage = captureur->capturer();
    uchar4* image = CaptureVideo::castToUChar4(&matImage);
    HANDLE_ERROR(hipMemcpy(ptrDevPixels,image,(w*h)*sizeof(ptrDevPixels[0]),hipMemcpyHostToDevice));
    colorToGreyShared<<<dg,db>>>(ptrDevPixels,w,h);
    HANDLE_ERROR(hipDeviceSynchronize());
    convolutionKernelShared<<<dg,db,sizeConvSM>>>(ptrDevPixels,ptrDeviceNoyau,k,w,h,t);
    HANDLE_ERROR(hipDeviceSynchronize());
    findMinMaxShared<<<dg,db,sizeSM>>>(ptrDevPixels,ptrDevResult,w,h);
    HANDLE_ERROR(hipMemcpy(ptrHostResult, ptrDevResult, sizeResult, hipMemcpyDeviceToHost));
    uchar max = 0;
    uchar min = 255;
    for(int i = 0;i<dg.x;i++)
	{
	uchar crt = ptrHostResult[i];
	if(crt < min)
	    min = crt;
	crt = ptrHostResult[i+dg.x];
	if(crt > max)
	    max = crt;
	}
    // affine transformation
    float a = 255.0f/(float)(max-min);
    float b = 0;
    if(min != 0)
	b = 255.0f/((-max/(float)min)+1.0f);
    affineTransformShared<<<dg,db>>>(ptrDevPixels, a, b, w, h,0);
    //printf("min: %d, max: %d\n",min,max);
    hipDeviceSynchronize();
    chrono.stop();
    cout << "ElapseTime:  " << chrono.getDeltaTime()<< " (s)" << endl;
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float ConvolutionShared::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int ConvolutionShared::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int ConvolutionShared::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string ConvolutionShared::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

