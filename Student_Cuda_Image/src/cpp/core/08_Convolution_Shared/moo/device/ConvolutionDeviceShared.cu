#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"
#include "Lock.h"

#include "ConvolutionMathShared.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__device__ int mutex = 0;

__global__ void convolutionKernelShared(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h,float t);

__global__ void colorToGreyShared(uchar4* ptrDevPixels, int w, int h);

__global__ void affineTransformShared(uchar4* ptrDevPixels, float a, float b, int w, int h, int offset);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__device__ void ecrasementShared(uchar* tabSM, int halfThread)
    {
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    const int NB_THREAD_BLOCK = Indice1D::nbThreadBlock();
    int s = TID_LOCAL;
    while(s<halfThread)
	{
	if(tabSM[s]>tabSM[s+halfThread]) // if tabSM not min
	    tabSM[s] = tabSM[s+halfThread];
	if(tabSM[NB_THREAD_BLOCK+s]<tabSM[NB_THREAD_BLOCK+s+halfThread]) // if tabSM not max
	    tabSM[NB_THREAD_BLOCK+s] = tabSM[NB_THREAD_BLOCK+s+halfThread];
	s+=NB_THREAD_BLOCK;
	}
    }

__device__ void reductionIntraBShared(uchar* tabSM)
    {
    const int NB_THREAD = Indice1D::nbThread();
    const int NB_THREAD_BLOCK=Indice1D::nbThreadBlock();
    int halfThread = NB_THREAD_BLOCK/2;
    while(halfThread>=1)
	{
	ecrasementShared(tabSM,halfThread);
	__syncthreads();
	halfThread/=2;
	}
    }

__device__ void reductionInterBShared(uchar* tabSM, uchar* ptrDevResult)
    {
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    const int NB_THREADS = Indice1D::nbThreadBlock();
    const int NB_BLOCKS = gridDim.x;
    const int BID = blockIdx.x;
    if(TID_LOCAL==0)
	{
	// bad idea with Lock, finish on CPU instead, only NB_BLOCKS items
	ptrDevResult[BID] = tabSM[0]; // min
	ptrDevResult[NB_BLOCKS+BID] = tabSM[NB_THREADS]; // max
	}

    }

__device__ void reductionIntraTShared(uchar* tabSM, uchar4* ptrDevPixels,int n)
    {
    const int NB_THREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocalBlock();

    int s = TID;
    uchar minCrtThread = 255;
    uchar maxCrtThread = 0;
    while(s<n)
	{
	uchar crtVal = ptrDevPixels[s].x;
	if(crtVal > maxCrtThread)
	    maxCrtThread = crtVal;
	if(crtVal < minCrtThread)
	    minCrtThread = crtVal;
	s+=NB_THREAD;
	}
    //printf("currentSum=%f PI=%f\n in reduce intrathread", sumCurrentThread,sumCurrentThread*DX);
    //cout<<"currentSum"<<sumCurrentThread<<endl;
    tabSM[TID_LOCAL] = minCrtThread;
    tabSM[Indice1D::nbThreadBlock()+TID_LOCAL] = maxCrtThread; // tabSM is 2*n size
    }

__global__ void colorToGreyShared(uchar4* ptrDevPixels, int w, int h)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    int s = TID;
    while(s<WH)
	{
	float x = ptrDevPixels[s].x;
	float y = ptrDevPixels[s].y;
	float z = ptrDevPixels[s].z;
	float average = (x/(float)3.0+y/(float)3.0+z/(float)3.0);
	ptrDevPixels[s].x = average;
	ptrDevPixels[s].y = average;
	ptrDevPixels[s].z = average;
	ptrDevPixels[s].w = 255;
	s += NB_THREAD;
	}
    }

__global__ void affineTransformShared(uchar4* ptrDevPixels, float a, float b, int w, int h, int offset)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    int s = TID;
    while(s<WH)
	{
	uchar newValue = (uchar)((float)(ptrDevPixels[s].x)*a+b);
	newValue += offset;
	if(newValue > 255)
	    newValue = 255;
	ptrDevPixels[s].x = newValue;
	ptrDevPixels[s].y = newValue;
	ptrDevPixels[s].z = newValue;
	s += NB_THREAD;
	}
    }

__global__ void convolutionKernelShared(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t)
    {

    extern __shared__ float convSM[];// 1 instance per block !

    ConvolutionMathShared convMath = ConvolutionMathShared(w, h);

    const int TID = Indice2D::tid();
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID_LOCAL;
    if(s < (k*k))
	convSM[s] = ptrDeviceNoyau[s];
    __syncthreads();

    s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)
	color = ptrDevPixels[s];
	convMath.colorIJ(&color,ptrDevPixels,convSM,k,pixelI, pixelJ, s); 	// update color
	ptrDevPixels[s] = color;
	s += NB_THREAD;
	}
    }

/*
 * ptrDevResult should contain min in [0] and max in [1]
 */
__global__ void findMinMaxShared(uchar4* ptrDevPixels, uchar* ptrDevResult,int w, int h)
    {
    // one shared memory per block
    extern __shared__ uchar tabSM[];
    //const int TID_LOCAL = Indice1D::tidLocalBlock();

    int sizePtrDevPixels = w*h;
    reductionIntraTShared(tabSM, ptrDevPixels,sizePtrDevPixels);
    reductionIntraBShared(tabSM);
    reductionInterBShared(tabSM, ptrDevResult);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

