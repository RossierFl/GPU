#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"

#include "Rippling0Math.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/



/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __global__ void rippling0(uchar4* ptrDevPixels,int w, int h,float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

void launchKernelRippling0(uchar4* ptrDevPixels, int w, int h, float t)
    {
    dim3 dg = dim3(8, 8, 1); // disons, a optimiser
    dim3 db = dim3(16, 16, 1); // disons, a optimiser

    //Device::print(dg, db);
     Device::checkDimError(dg,db);

    rippling0<<<dg,db>>>(ptrDevPixels,w,h,t);
    Device::checkKernelError("rippling0");
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rippling0(uchar4* ptrDevPixels, int w, int h, float t)
    {
    Rippling0Math rippling0Math = Rippling0Math(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;
    color.w = 255; // alpha

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	rippling0Math.color(pixelI, pixelJ, t, &color); // update color
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}

    }



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

