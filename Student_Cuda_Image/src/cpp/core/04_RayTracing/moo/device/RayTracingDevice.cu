#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"
#include "Sphere.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, float t, Sphere* ptrSpheresDevGM, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, float t, Sphere* ptrSpheresDevGM, int n)
{
	RayTracingMath* math = new RayTracingMath();

	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;

	uchar4 color;
	int x;
	int y;

	int s = TID;
	while (s < WH)
	{
		IndiceTools::toIJ(s, w, &x, &y);
		math->colorXY(&color, x, y, t, ptrSpheresDevGM, n);
		ptrDevPixels[s] = color;
		s += NB_THREAD;
	}

	delete math;
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
