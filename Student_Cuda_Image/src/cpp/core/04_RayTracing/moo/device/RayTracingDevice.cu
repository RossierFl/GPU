#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"
#include "Sphere.h"

__global__ void rayTracingGPU(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n) {
	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;

	RayTracingMath* rayTracingMath = new RayTracingMath();

	int s = TID;
	while (s < WH) {
		uchar4 color;
		int x;
		int y;
		IndiceTools::toIJ(s, w, &x, &y);
		rayTracingMath->colorXY(&color, x, y, t, spheres, n);
		ptrDevPixels[s] = color;
		s += NB_THREAD;
	}

	delete rayTracingMath;
}

