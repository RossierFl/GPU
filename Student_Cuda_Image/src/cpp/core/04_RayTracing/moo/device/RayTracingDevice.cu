#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"
#include "Sphere.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracingGPU(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, float t, Sphere* spheres, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rayTracingGPU(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n)
    {
    RayTracingMath* rayTracingMath = new RayTracingMath();

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int x;
    int y;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &x, &y );
	rayTracingMath->colorXY(&color,x, y,t,spheres, n);
	ptrDevPixels[s] = color;
	s += NB_THREAD;
	}
    delete rayTracingMath;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
