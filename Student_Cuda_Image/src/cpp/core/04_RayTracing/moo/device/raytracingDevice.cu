#include "hip/hip_runtime.h"

#include "Sphere.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "RaytracingMath.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void raytracingKernel(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void raytracingKernel(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    RayTracingMath* math = new RayTracingMath();

    int s = TID;

    uchar4 color;
    int x;
    int y;

    while (s < WH) {
	IndiceTools::toIJ(s, w, &x, &y);
	math->colorXY(&color, x, y, t, spheres, n);
	ptrDevPixels[s] = color;
	s += NB_THREAD;
    }

    delete math;

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

