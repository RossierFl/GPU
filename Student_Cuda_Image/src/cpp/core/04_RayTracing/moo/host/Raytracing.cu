#include "hip/hip_runtime.h"

#include "Raytracing.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void raytracingKernel(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
Raytracing::Raytracing(int w, int h, Sphere* spheres, int n)
    {
    // Inputs
    this->w = w;
    this->h = h;

    this->t = 0;

    this->spheres = spheres;
    this->n = n;
    // Tools
    this->dg = dim3(8, 8, 1); // TODO disons a optimiser
    this->db = dim3(16, 16, 1); // TODO disons a optimiser
    this->t = variateurAnimation.varierAndGet();
    //Outputs
    this->title = "RayTracing non-zoomable CUDA";
    // control
    Device::assertDim(dg, db);
    assert(w == h);
    }

Raytracing::~Raytracing()
    {

    }

/*-------------------------*\
|*	Methode	           *|
\*-------------------------*/

void Raytracing::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/**
 * Override
 */
void Raytracing::runGPU(uchar4* ptrDevPixels)
    {
    Sphere* spheresDevGRAM = NULL;
    HANDLE_ERROR(hipMalloc(&spheresDevGRAM, n * sizeof(Sphere)));
    HANDLE_ERROR(hipMemcpy(spheresDevGRAM, spheres, n * sizeof(Sphere), hipMemcpyHostToDevice));
    raytracingKernel<<<dg,db>>>(ptrDevPixels, w, h, t, spheresDevGRAM, n);
    hipFree(spheresDevGRAM);
    hipDeviceSynchronize();
    }

/*-------------------------*\
|*	Get	           *|
\*-------------------------*/

/**
 * Override
 */
float Raytracing::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Raytracing::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Raytracing::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Raytracing::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

