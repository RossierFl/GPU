#include "hip/hip_runtime.h"

#include "Raytracing.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

extern __global__ void raytracingKernel(uchar4* ptrDevPixels,int w, int h,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
Raytracing::Raytracing(int w, int h)
    {
    // Inputs
    this->w = w;
    this->h = h;

    this->t = 0;
    }

Raytracing::~Raytracing()
    {

    }

/*-------------------------*\
|*	Methode	           *|
\*-------------------------*/

void Raytracing::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/**
 * Override
 */
void Raytracing::runGPU(uchar4* ptrDevPixels)
    {
    raytracingKernel<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/*-------------------------*\
|*	Get	           *|
\*-------------------------*/

/**
 * Override
 */
float Raytracing::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Raytracing::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Raytracing::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Raytracing::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

