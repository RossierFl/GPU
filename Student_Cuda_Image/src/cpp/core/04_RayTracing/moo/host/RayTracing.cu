#include "hip/hip_runtime.h"
#include <assert.h>

#include "RayTracing.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

__global__ void rayTracingGPU(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n);

RayTracing::RayTracing(int w, int h, float dt, double x1, double y1, double x2, double y2, Sphere* spheres, int n) :
		variateurAnimation(IntervalF(0, 10), dt) {
	// Inputs
	this->w = w;
	this->h = h;

	this->spheres = spheres;
	this->n = n;

	// Tools
	this->dg = dim3(8, 8, 1); // TODO disons a optimiser
	this->db = dim3(16, 16, 1); // TODO disons a optimiser
	this->t = variateurAnimation.varierAndGet();

	//Outputs
	this->title = "RayTracing non-zoomable CUDA";

	// control
	Device::assertDim(dg, db);
	assert(w == h);
}

RayTracing::~RayTracing() {
}

void RayTracing::runGPU(uchar4* ptrDevPixels) {
	Sphere* spheresDevGRAM = NULL;
	HANDLE_ERROR(hipMalloc(&spheresDevGRAM, n * sizeof(Sphere)));
	HANDLE_ERROR(hipMemcpy(spheresDevGRAM, spheres, n * sizeof(Sphere), hipMemcpyHostToDevice));
	rayTracingGPU<<<dg,db>>>(ptrDevPixels, w, h, t, spheresDevGRAM, n);
	hipFree(spheresDevGRAM);
	hipDeviceSynchronize();
}

/**
 * Override
 */
void RayTracing::animationStep() {
	this->t = variateurAnimation.varierAndGet();
}

/**
 * Override
 */
float RayTracing::getT() {
	return t;
}

/**
 * Override
 */
int RayTracing::getW() {
	return w;
}

/**
 * Override
 */
int RayTracing::getH() {
	return h;
}

/**
 * Override
 */
string RayTracing::getTitle() {
	return title;
}
