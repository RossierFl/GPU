#include "hip/hip_runtime.h"
#include <assert.h>

#include "RayTracing.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void rayTracingGPU(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(int w, int h, float dt, double x1, double y1, double x2, double y2,double z1,double z2 ,double rayonMax,int nbrBalls) :
		variateurAnimation(IntervalF(0,10), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
	

      spheres= new Sphere[nbrBalls];
      std::cout<<"NBR BALLLS"<<nbrBalls<<std::endl;
      for(int i = 0;i<nbrBalls;i++)
      {
  	    float x = drand48();
  	    float y = drand48();
  	    float z = drand48();
  	    float3 centre;
  	    centre.x = x1+x*(x2-x1);
  	    centre.y = y1+y*(y2-y1);
  	    centre.z = z1+z*(z2-z1);
  	    float r = drand48();
  	    float hue = drand48();
  	    spheres[i] = Sphere(centre, 20+r*rayonMax, hue);
      }
      for(int i =0;i<nbrBalls;i++){
	  //std::cout<<spheres[i].hue(2)<<std::endl;
      }
    this->nbrBalls = nbrBalls;
	
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = variateurAnimation.varierAndGet();
    InitSphereMemory();

    //Outputs
    this->title = "[API Image Normale] : RayTracing non-zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }



RayTracing::~RayTracing()
    {
    hipFree(ptrSpheresDev);//free mat
    delete spheres;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void RayTracing::animationStep()
    {
    this->t = variateurAnimation.varierAndGet();
    }

/**
 * Override
 */
void RayTracing::runGPU(uchar4* ptrDevPixels)
    {

    rayTracingGPU<<<dg,db>>>(ptrDevPixels,w,h,t,ptrSpheresDev,nbrBalls);//call

    hipDeviceSynchronize();

    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float RayTracing::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int RayTracing::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int RayTracing::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string RayTracing::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void RayTracing::InitSphereMemory(){

        HANDLE_ERROR(hipMalloc(&ptrSpheresDev,nbrBalls*sizeof(Sphere)));//malloc all spheres
        HANDLE_ERROR(hipMemcpy(ptrSpheresDev,spheres,nbrBalls*sizeof(Sphere),hipMemcpyHostToDevice));//fill with spheres

}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
