#include "hip/hip_runtime.h"
#include <assert.h>

#include "RayTracing.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, float t, Sphere* spheres, int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(int w, int h, float dt, double x1, double y1, double x2, double y2, Sphere* spheres, int n) :
		variateurAnimation(IntervalF(0,10), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
	
    this->spheres = spheres;
    this->n = n;
	
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = variateurAnimation.varierAndGet();

    //Outputs
    this->title = "[API Image Normale] : RayTracing non-zoomable CUDA -- Emmanuel Dafflon";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

RayTracing::~RayTracing()
    {
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void RayTracing::animationStep()
    {
    this->t = variateurAnimation.varierAndGet();
    }

/**
 * Override
 */
void RayTracing::runGPU(uchar4* ptrDevPixels)
    {
    Sphere* spheresToDev = NULL; //create pointer
    HANDLE_ERROR(hipMalloc(&spheresToDev,n*sizeof(Sphere)));//malloc all spheres
    HANDLE_ERROR(hipMemcpy(spheresToDev,spheres,n*sizeof(Sphere),hipMemcpyHostToDevice));//fill with spheres
    rayTracing<<<dg,db>>>(ptrDevPixels,w,h,t,spheresToDev,n);//call
    hipFree(spheresToDev);//free mat
    hipDeviceSynchronize(); // in case of issues
    //printf("\n");
    //exit(0);
    //get fps to do smooth animation ??
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float RayTracing::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int RayTracing::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int RayTracing::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string RayTracing::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
