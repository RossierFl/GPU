#include "hip/hip_runtime.h"
#include <assert.h>

#include "RayTracing.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, float t, Sphere* ptrSpheresDevGM, int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(int w, int h, float dt, double x1, double y1, double x2, double y2, Sphere* spheres, int n) :
		variateurAnimation(IntervalF(0, 15), dt)
{
	// Check
	assert(w == h);

	// Inputs
	this->w = w;
	this->h = h;
	this->spheres = spheres;
	this->n = n;

	// Tools
	this->dg = dim3(8, 8, 1); // disons, à optimiser
	this->db = dim3(16, 16, 1); // disons, à optimiser
	this->t = variateurAnimation.varierAndGet();

	Device::assertDim(dg, db);
}

RayTracing::~RayTracing()
{
}

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
* Override
*/
void RayTracing::animationStep()
{
	this->t = variateurAnimation.varierAndGet();
}

/**
* Override
*/
void RayTracing::runGPU(uchar4* ptrDevPixels)
{
	/* Pointer on global memory for the spheres */
	Sphere* ptrSpheresDevGM = NULL;

	/* Allocate in global memory */
	HANDLE_ERROR(hipMalloc(&ptrSpheresDevGM, n * sizeof(Sphere)));

	/* Copy from RAM to GRAM */
	HANDLE_ERROR(hipMemcpy(ptrSpheresDevGM, spheres, n * sizeof(Sphere), hipMemcpyHostToDevice));

	/* Launch kernel */
	// @formatter:off
	rayTracing<<<dg,db>>>(ptrDevPixels, w, h, t, ptrSpheresDevGM, n);
	// @formatter:on

	/* Free memory */
	hipFree(ptrSpheresDevGM);

	/* DEBUG */
	hipDeviceSynchronize();
	printf("\n");
}

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
* Override
*/
float RayTracing::getT()
{
	return t;
}

/**
* Override
*/
int RayTracing::getW()
{
	return w;
}

/**
* Override
*/
int RayTracing::getH()
{
	return h;
}

/**
* Override
*/
string RayTracing::getTitle()
{
	return "Ray tracing";
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
