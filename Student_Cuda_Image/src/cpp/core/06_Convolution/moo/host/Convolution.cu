#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <assert.h>

#include "Convolution.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void convolutionKernel(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t);
extern __global__ void colorToGrey(uchar4* ptrDevPixels, int w, int h);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Convolution::Convolution(int w, int h,float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;
    this->k = 9;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : Convolution CUDA";
    this->videoPath="/media/Data/Video/nasaFHD_short.avi";
    this->videoTitle="NASA FHD SHORT";

    size_t sizeOctets=w*h*sizeof(uchar4);
    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostMemory), sizeOctets,hipHostMallocDefault ) );

    this->captureur = new CaptureVideo(videoPath,videoTitle,this->ptrHostMemory);

    const int N = k*k;

    printf("N: %d, k: %d, half of k: %d, ss: %d\n",N,k,k/2,(int)(k*((float)k/2.0f)));

    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostNoyau), N*sizeof(float),hipHostMallocDefault ) );

    fillDetourage(ptrHostNoyau);

    HANDLE_ERROR(hipMalloc((void **)&ptrDeviceNoyau,N*sizeof(float)));
    HANDLE_ERROR(hipMemcpy(ptrDeviceNoyau,ptrHostNoyau,N*sizeof(float),hipMemcpyHostToDevice));

    //cout << endl<<"[CBI] Convolution dt =" << dt << endl;
    }

Convolution::~Convolution()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Noyau detourage 9x9
 */
void Convolution::fillDetourage(float* ptrNoyau)
    {
    const int N = 9*9;
    // Tab auto temporaire
    float tab[N] =
	    {
		    0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.6063, 1.2765, 0.4061, -6.4829, -12.7462, -6.4829, 0.4061, 1.2765, 0.6063, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828
	    };

    for (int i = 1; i <= N; i++)
	{
	ptrNoyau[i - 1] = tab[i - 1] / 100;
	}
    }

/**
 * Override
 * Call periodicly by the API
 */
void Convolution::animationStep()
    {
    t+=dt;
    }

/**
 * Override
 */
void Convolution::runGPU(uchar4* ptrDevPixels)
    {
    Mat matImage = captureur->capturer();
    uchar4* image = CaptureVideo::castToUChar4(&matImage);
    HANDLE_ERROR(hipMemcpy(ptrDevPixels,image,(w*h)*sizeof(ptrDevPixels[0]),hipMemcpyHostToDevice));
    colorToGrey<<<dg,db>>>(ptrDevPixels,w,h);
    HANDLE_ERROR(hipDeviceSynchronize());
    convolutionKernel<<<dg,db>>>(ptrDevPixels,ptrDeviceNoyau,k,w,h,t);
    HANDLE_ERROR(hipDeviceSynchronize());
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Convolution::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Convolution::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Convolution::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string Convolution::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

