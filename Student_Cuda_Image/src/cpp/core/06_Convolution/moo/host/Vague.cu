#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Convolution.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void convolution(uchar4* ptrDevPixels,int w, int h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Convolution::Convolution(int w, int h,float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : Convolution CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w== h);

    cout << endl<<"[CBI] Convolution dt =" << dt;
    }

Convolution::~Convolution()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Convolution::animationStep()
    {
    t+=dt;
    }

/**
 * Override
 */
void Convolution::runGPU(uchar4* ptrDevPixels)
    {
    rippling<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Convolution::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Convolution::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Convolution::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string Convolution::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

