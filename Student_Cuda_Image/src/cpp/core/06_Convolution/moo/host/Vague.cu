#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Vague.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void vague(uchar4* ptrDevPixels,int w, int h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Vague::Vague(int w, int h,float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : Vague CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w== h);

    cout << endl<<"[CBI] Vague dt =" << dt;
    }

Vague::~Vague()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Vague::animationStep()
    {
    t+=dt;
    }

/**
 * Override
 */
void Vague::runGPU(uchar4* ptrDevPixels)
    {
    rippling<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Vague::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Vague::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Vague::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string Vague::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

