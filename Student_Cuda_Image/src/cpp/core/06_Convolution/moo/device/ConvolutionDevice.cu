#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"

#include "ConvolutionMath.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void convolutionKernel(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h,float t);

__global__ void colorToGrey(uchar4* ptrDevPixels, int w, int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void colorToGrey(uchar4* ptrDevPixels, int w, int h)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    int s = TID;
    while(s<WH)
	{
	float x = ptrDevPixels[s].x;
	float y = ptrDevPixels[s].y;
	float z = ptrDevPixels[s].z;
	float average = (x/(float)3.0+y/(float)3.0+z/(float)3.0);
	ptrDevPixels[s].x = average;
	ptrDevPixels[s].y = average;
	ptrDevPixels[s].z = average;
	ptrDevPixels[s].w = 255;
	s += NB_THREAD;
	}
    }

__global__ void convolutionKernel(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t)
    {
    ConvolutionMath convMath = ConvolutionMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)
	color = ptrDevPixels[s];
	convMath.colorIJ(&color,ptrDevPixels,ptrDeviceNoyau,k,pixelI, pixelJ, s, t); 	// update color
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

