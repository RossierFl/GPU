#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJuliaMGPU.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractale(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	int isJulia, double c1, double c2);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJuliaMGPU::MandelbrotJuliaMGPU(int w, int h, float dt, int n, int isJulia,
	double c1, double c2, double x1, double y1, double x2, double y2) :
								variateurAnimation(IntervalF(30, 100), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    this->c1 = c1;
    this->c2 = c2;

    this->isJulia = isJulia;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(x1,y1,x2,y2);

    //Outputs
    this->title = "[API Image Fonctionelle] : Mandelbrot Julia zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);

    Device::p2pEnableALL();

    nbGPUs = 3;
    splittedH = h/nbGPUs;
    tailleSplitted = w*splittedH*sizeof(uchar4);

    for(int i = 1;i<nbGPUs;i++)
	{
	hipSetDevice(i);
	HANDLE_ERROR(hipMalloc((void**)&ptrDevZoneMemGPU[i],tailleSplitted));
	}
    hipSetDevice(0);

    assert(w == h);
    }

MandelbrotJuliaMGPU::~MandelbrotJuliaMGPU()
    {
    delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJuliaMGPU::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/**
 * Override
 */
void MandelbrotJuliaMGPU::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    for(int i = 0;i<nbGPUs;i++)
	{
	hipSetDevice(i);
	int offset = w*splittedH;
	uchar4* ptrDevBandISrc = ptrDevPixels+offset;
	if(i == 0)
	    {
	    fractale<<<dg,db>>>(ptrDevPixels,w,splittedH,domaineMath,getT(),t,isJulia,c1,c2);
	    }
	else if(i > 0)
	    {
	    fractale<<<dg,db>>>(ptrDevZoneMemGPU[i],w,splittedH,domaineMath,getT(),t,isJulia,c1,c2);
	    HANDLE_ERROR(hipMemcpy(ptrDevPixels+offset,ptrDevZoneMemGPU[i],tailleSplitted));
	    }
	}
    hipSetDevice(0);
    //hipDeviceSynchronize();
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJuliaMGPU::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float MandelbrotJuliaMGPU::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJuliaMGPU::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJuliaMGPU::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJuliaMGPU::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

