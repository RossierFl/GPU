#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJuliaMGPU.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractaleMGPU(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	int isJulia, double c1, double c2, int offset);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJuliaMGPU::MandelbrotJuliaMGPU(int w, int h, float dt, int n, int isJulia,
	double c1, double c2, double x1, double y1, double x2, double y2) :
														variateurAnimation(IntervalF(30, 100), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    this->c1 = c1;
    this->c2 = c2;

    this->isJulia = isJulia;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(x1,y1,x2,y2);

    //Outputs
    this->title = "[API Image Fonctionelle] : Mandelbrot Julia multi-gpu zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);

    Device::p2pEnableALL();

    nbGPUs = 3;
    splittedH = h/nbGPUs;
    tailleSplitted = w*splittedH*sizeof(uchar4);

    //ptrDevZoneMemGPU = (uchar4**) malloc(nbGPUs-1*sizeof(uchar4*));

    //    for(int i = 1;i<nbGPUs;i++)
    //	{
    //
    //	}
    hipSetDevice(1);
    HANDLE_ERROR(hipMalloc((void**)&ptrDevGpu1,tailleSplitted));
    hipSetDevice(2);
    HANDLE_ERROR(hipMalloc((void**)&ptrDevGpu2,tailleSplitted));
    hipSetDevice(0);

    assert(w == h);
    }

MandelbrotJuliaMGPU::~MandelbrotJuliaMGPU()
    {
    delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJuliaMGPU::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/**
 * Override
 */
void MandelbrotJuliaMGPU::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    /*
     * TODO implement OMP, but if no OMP, ok runs fine and quick without it
     */
    //    for(int i = 0;i<nbGPUs;i++)
    //	{
    hipSetDevice(0);
    //	if(i == 0)
    //	    {
    int offset = 0;
    fractaleMGPU<<<dg,db>>>(ptrDevPixels,w,splittedH,domaineMath,getT(),t,isJulia,c1,c2,offset);
    //}
    //	else if(i > 0)
    //	    {
    hipSetDevice(1);
    offset = w*splittedH;
    uchar4* ptrDevBandISrc = ptrDevPixels+offset;
    fractaleMGPU<<<dg,db>>>(ptrDevGpu1,w,splittedH,domaineMath,getT(),t,isJulia,c1,c2,offset);
    HANDLE_ERROR(hipMemcpy(ptrDevBandISrc,ptrDevGpu1,tailleSplitted,hipMemcpyDeviceToDevice));
    hipSetDevice(2);
    offset = 2*w*splittedH;
    ptrDevBandISrc = ptrDevPixels+offset;
    fractaleMGPU<<<dg,db>>>(ptrDevGpu2,w,splittedH,domaineMath,getT(),t,isJulia,c1,c2,offset);
    HANDLE_ERROR(hipMemcpy(ptrDevBandISrc,ptrDevGpu2,tailleSplitted,hipMemcpyDeviceToDevice));

    // }
    //}

    hipSetDevice(0);
    //hipDeviceSynchronize();
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJuliaMGPU::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float MandelbrotJuliaMGPU::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJuliaMGPU::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJuliaMGPU::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJuliaMGPU::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

