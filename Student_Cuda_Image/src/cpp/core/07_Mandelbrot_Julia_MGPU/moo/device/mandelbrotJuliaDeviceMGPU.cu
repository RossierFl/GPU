#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "Fractale.h"
#include "MandelbrotMath.h"
#include "JuliaMath.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void fractale(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	int isJulia, double c1, double c2);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void fractale(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	int isJulia, double c1, double c2)
    {
    Fractale* fractaleMath = 0;
    if(isJulia)
	fractaleMath = new JuliaMath(n,c1,c2);
    else
	fractaleMath = new MandelbrotMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	fractaleMath->colorXY(&color,x, y,t); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    delete fractaleMath;

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

