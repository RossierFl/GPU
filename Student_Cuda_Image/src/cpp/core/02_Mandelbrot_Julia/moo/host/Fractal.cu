#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Fractal.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void fractal_gpu(uchar4* ptrDevPixels, int w, int h, int n, DomaineMath& domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Fractal::Fractal(int w, int h, float dt, int n, float x1, float y1, float x2, float y2) :
		variateur(IntervalI(30, 100), dt) {

	// Inputs
	this->w = w;
	this->h = h;
	this->n = n;
	this->x1 = x1;
	this->y1 = y1;
	this->x2 = x2;
	this->y2 = y2;
	this->domaineMath = new DomaineMath(x1, y1, x2, y2);

	// Tools
	this->dg = dim3(16, 16, 1);
	this->db = dim3(8, 8, 1);

	// Outputs
	this->title = "Fractal Cuda";

	//print(dg, db);
	Device::assertDim(dg, db);
}

Fractal::~Fractal() {
	delete domaineMath;
}

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Fractal::animationStep() {
	n = variateur.varierAndGet();
}

/**
 * Override
 */
void Fractal::runGPU(uchar4* ptrDevPixels, DomaineMath& useless) {
	fractal_gpu<<< db, db >>>(ptrDevPixels, w, h, n, useless);
}

/*--------------*\
 |*	get	 *|
 \*--------------*/

float Fractal::getT(void) {
return n;
}

int Fractal::getW(void) {
return w;
}

int Fractal::getH(void) {
return h;
}

DomaineMath* Fractal::getDomaineMathInit() {
return domaineMath;
}

string Fractal::getTitle(void) {
return title;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

