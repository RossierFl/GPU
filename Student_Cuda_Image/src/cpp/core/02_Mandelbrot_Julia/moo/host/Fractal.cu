#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "MathTools.h"
#include "Fractal.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void fractalGPU(uchar4* ptrDevPixels, int w, int h, int n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Fractal::Fractal(int w, int h, float dt, int n, double x1, double y1, double x2, double y2) :
		variateur(IntervalF(30, 100), dt) {

	// Inputs
	this->w = w;
	this->h = h;
	this->n = n;
	this->domaineMath = new DomaineMath(x1, y1, x2, y2);
	this->t = 0;

	// Tools
	this->dg = dim3(8, 8, 1);
	this->db = dim3(16, 16, 1);
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);

	// Outputs
	this->title = "Fractal Cuda";

	//print(dg, db);
	Device::assertDim(dg, db);
	assert(w == h);
}

Fractal::~Fractal() {
	delete domaineMath;
}

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Fractal::animationStep() {
	t = variateur.varierAndGet();
}

/**
 * Override
 */
void Fractal::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath) {
	fractalGPU<<<dg,db>>>(ptrDevPixels, w, h, n, domaineMath);
	//Device::synchronize();
	//Device::checkKernelError("Error>> Fractal::runGPU");
}

/*--------------*\
 |*	get	 *|
 \*--------------*/

float Fractal::getT(void) {
	return t;
}

int Fractal::getW(void) {
	return w;
}

int Fractal::getH(void) {
	return h;
}

DomaineMath* Fractal::getDomaineMathInit() {
	return domaineMath;
}

string Fractal::getTitle(void) {
	return title;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

