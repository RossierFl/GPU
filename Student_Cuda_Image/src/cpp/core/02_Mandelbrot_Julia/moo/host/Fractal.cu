#include "hip/hip_runtime.h"
#include <assert.h>

#include "Fractal.h"
#include "Device.h"
#include "MathTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractal(uchar4* ptrDevPixels, int w, int h,bool julia, DomaineMath domaineMath, int n,double cx, double cy);
//uchar4* ptrDevPixels,int w, int h,bool julia,DomaineMath domaineMath, int n,double cx, double cy

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Fractal::Fractal(int w, int h, float dt,  int nMin,int nMax,bool julia,double cx,double cy,double xMin, double xMax, double yMin, double yMax) :
	variateurAnimation(IntervalI(nMin,nMax), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->julia = julia;
    this->cx = cx;
    this->cy = cy;
    this->n = nMin;
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    ptrDomaineMathInit=new DomaineMath(xMin,yMin,xMax,yMax);

    //Outputs
    this->title = "[API Image Fonctionelle] : Fractal zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    }

Fractal::~Fractal()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Fractal::animationStep()
    {
    this->n = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void Fractal::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    fractal<<<dg,db>>>(ptrDevPixels,this->w,this->h,this->julia,*ptrDomaineMathInit,this->n,this->cx,this->cy);
    //uchar4* ptrDevPixels,int w, int h,bool julia,DomaineMath domaineMath, int n,double cx, double cy
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Fractal::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Fractal::getT(void)
    {
    t=n;
    return t;
    }

/**
 * Override
 */
int Fractal::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Fractal::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Fractal::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

