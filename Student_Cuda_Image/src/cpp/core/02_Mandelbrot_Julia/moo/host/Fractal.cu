#include "hip/hip_runtime.h"
#include <assert.h>

#include "Fractal.h"
#include "Device.h"
#include "MathTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractal(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Fractal::Fractal(int w, int h, float dt, int n) :
	variateurAnimation(IntervalF(0, 2 * PI), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(0,0,2*PI,2*PI);

    //Outputs
    this->title = "[API Image Fonctionelle] : Fractal zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

Fractal::~Fractal()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Fractal::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void Fractal::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    //TODO
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Fractal::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Fractal::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Fractal::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Fractal::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Fractal::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

