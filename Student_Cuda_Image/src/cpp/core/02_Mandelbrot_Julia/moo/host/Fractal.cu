#include "hip/hip_runtime.h"
#include <assert.h>

#include "Fractal.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractal(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, double cx, double cyi, bool isJulia);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Fractal::Fractal(unsigned int w, unsigned int h, unsigned int nMin, unsigned int nMax, double cx, double cyi, bool isJulia, double x0, double x1, double y0, double y1) :
	variateurAnimation(IntervalF(nMin, nMax), 1)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = nMin;
    this->cx = cx;
    this->cyi = cyi;
    this->isJulia = isJulia;
    this->x0=x0;
    this->x1=x1;
    this->y0=y0;
    this->y1=y1;

    // Tools
    this->dg = dim3(16, 16, 1); // disons a optimiser
    this->db = dim3(8, 8, 1); // disons a optimiser
    this->t = 0;
    this->ptrDomaineMathInit=new DomaineMath(x0,y0,x1,y1);

    //Outputs
    this->title = "[API Image Fonctionelle] : Fractal CUDA";

    // Check:
    //print(dg, db);
    Device::checkDimOptimiser(dg, db);
    //assert(w == h);
    }

Fractal::~Fractal()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Fractal::animationStep()
    {
    this->n = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void Fractal::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    fractal<<<dg,db>>>(ptrDevPixels,w,h,domaineMath, n, cx, cyi, isJulia);
    //Device::synchronize();
    //Device::checkKernelError("fractal");
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Fractal::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Fractal::getT(void)
    {
    return n;
    }

/**
 * Override
 */
int Fractal::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Fractal::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Fractal::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

