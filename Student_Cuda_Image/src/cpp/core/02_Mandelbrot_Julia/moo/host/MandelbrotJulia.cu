#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJulia.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

__global__ void fractale(bool isJulia, uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t, float c1, float c2);

MandelbrotJulia::MandelbrotJulia(bool isJulia, bool isMultiGPU, int w, int h, float dt, int n, float c1, float c2, float x1, float x2, float y1, float y2) :
		variateurAnimation(IntervalF(30, 100), dt) {
	// Inputs
	this->w = w;
	this->h = h;
	this->n = n;

	this->c1 = c1;
	this->c2 = c2;

	this->isJulia = isJulia;
	this->isMultiGPU = isMultiGPU;

	// Tools
	this->dg = dim3(8, 8, 1); // disons a optimiser
	this->db = dim3(16, 16, 1); // disons a optimiser
	this->t = 0;
	ptrDomaineMathInit = new DomaineMath(x1, y1, x2, y2);

	//Outputs
	this->title = "[API Image Fonctionelle] : Mandelbrot or Julia zoomable";
	if(isMultiGPU) {
		this->title += " [Multi GPU]";
	}

	Device::assertDim(dg, db);
	assert(w == h);
}

MandelbrotJulia::~MandelbrotJulia() {
	delete ptrDomaineMathInit;
}

/**
 * Override
 */
void MandelbrotJulia::animationStep() {
	this->t = variateurAnimation.varierAndGet();
}

/**
 * Override
 */
void MandelbrotJulia::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath) {
	if (!isMultiGPU) {
		// Call kernel
		fractale<<<dg,db>>>(isJulia, ptrDevPixels, w, h, domaineMath, getT(), t, c1, c2);
		Device::checkKernelError("kernel error: fractale");
		Device::synchronize();
	} else {
		const uint NB_GPU = Device::getDeviceCount();

		/* Enable P2P */
		Device::p2pEnableALL();

		/* Compute steps for each GPU */
		uint remainingH = h;
		const uint stepH = h / NB_GPU;
		uint steps[NB_GPU];
		for (int device = 0; device < NB_GPU; device++) {
			if (device == NB_GPU - 1) {
				steps[device] = remainingH;
			} else {
				steps[device] = stepH;
			}
			remainingH -= steps[device];
		}

		/* Processing */
#pragma omp parallel for
		for (int device = 0; device < NB_GPU; device++) {
			bool isFirst = device == 0;

			// set current device
			hipSetDevice(device);

			// start of data to process on this GPU
			uint offset = w * device * steps[device];
			uchar4* start = ptrDevPixels + offset;
			uchar4* targetCurrentDevResult = ptrDevPixels;

			if(!isFirst) {
				targetCurrentDevResult = NULL;
				HANDLE_ERROR(hipMalloc(&targetCurrentDevResult, sizeof(uchar4) * steps[device]));
			}

			// call kernel to compute a step
			fractale<<<dg,db>>>(isJulia, targetCurrentDevResult, w, steps[device], domaineMath, getT(), t, c1, c2);
			Device::checkKernelError("kernel error: fractale");
			Device::synchronize();

			// Le premier GPU contient toute l'image, mais traite que les premières lignes
			// Les autres sont traitées par les autres GPUs, il faut donc copier leur résultat sur le
			// première GPU afin qu'OpenGL puisse tout afficher
			if(!isFirst) {
				HANDLE_ERROR(hipMemcpy(start, targetCurrentDevResult, sizeof(uchar4) * steps[device], hipMemcpyDeviceToDevice));
				HANDLE_ERROR(hipFree(targetCurrentDevResult));
			}
		}
	}
}

/**
 * Override
 */
DomaineMath* MandelbrotJulia::getDomaineMathInit() {
	return ptrDomaineMathInit;
}

/**
 * Override
 */
float MandelbrotJulia::getT() {
	return t;
}

/**
 * Override
 */
int MandelbrotJulia::getW() {
	return w;
}

/**
 * Override
 */
int MandelbrotJulia::getH() {
	return h;
}

/**
 * Override
 */
string MandelbrotJulia::getTitle() {
	return title;
}
