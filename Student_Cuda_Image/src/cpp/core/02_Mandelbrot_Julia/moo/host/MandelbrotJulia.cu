#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>


#include "MathTools.h"

#include "MandelbrotJulia.h"
#include "Device.h"
#include "DomaineMath.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void MandelbrotJulia(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJulia::MandelbrotJulia(unsigned int w, unsigned int h, float dt, int n, float xMin,float xMax,float yMin,float yMax,bool isJulia,float cX,float cY) :
	variateurAnimation(IntervalF(10, 100), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;
    this->cX=cX;
    this->cY=cY;
    this->isJulia=isJulia;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(xMin,yMin,xMax,yMax);

    //Outputs
    this->title = "MandelbrotJulia  CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

MandelbrotJulia::~MandelbrotJulia()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJulia::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void MandelbrotJulia::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    mandelbrotJulia<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,t,cX,cY,isJulia);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJulia::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float MandelbrotJulia::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJulia::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJulia::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJulia::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
