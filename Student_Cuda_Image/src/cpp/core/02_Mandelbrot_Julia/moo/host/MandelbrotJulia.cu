#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJulia.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

__global__ void fractale(bool isJulia, uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t, float c1, float c2);

MandelbrotJulia::MandelbrotJulia(bool isJulia, int w, int h, float dt, int n, float c1, float c2, float x1, float x2, float y1, float y2) :
		variateurAnimation(IntervalF(30, 100), dt) {
	// Inputs
	this->w = w;
	this->h = h;
	this->n = n;

	this->c1 = c1;
	this->c2 = c2;

	this->isJulia = isJulia;

	// Tools
	this->dg = dim3(8, 8, 1); // disons a optimiser
	this->db = dim3(16, 16, 1); // disons a optimiser
	this->t = 0;
	ptrDomaineMathInit = new DomaineMath(x1, y1, x2, y2);

	//Outputs
	this->title = "[API Image Fonctionelle] : Mandelbrot or Julia zoomable";

	Device::assertDim(dg, db);
	assert(w == h);
}

MandelbrotJulia::~MandelbrotJulia() {
	delete ptrDomaineMathInit;
}

/**
 * Override
 */
void MandelbrotJulia::animationStep() {
	this->t = variateurAnimation.varierAndGet();
}

/**
 * Override
 */
void MandelbrotJulia::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath) {
	fractale<<<dg,db>>>(isJulia, ptrDevPixels, w, h, domaineMath, getT(), t, c1, c2);
}

/**
 * Override
 */
DomaineMath* MandelbrotJulia::getDomaineMathInit() {
	return ptrDomaineMathInit;
}

/**
 * Override
 */
float MandelbrotJulia::getT() {
	return t;
}

/**
 * Override
 */
int MandelbrotJulia::getW() {
	return w;
}

/**
 * Override
 */
int MandelbrotJulia::getH() {
	return h;
}

/**
 * Override
 */
string MandelbrotJulia::getTitle() {
	return title;
}
