#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJulia.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractale(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	int isJulia, double c1, double c2);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJulia::MandelbrotJulia(int w, int h, float dt, int n, int isJulia,
	double c1, double c2, double x1, double y1, double x2, double y2) :
	variateurAnimation(IntervalF(30, 100), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    this->c1 = c1;
    this->c2 = c2;

    this->isJulia = isJulia;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(x1,y1,x2,y2);

    //Outputs
    this->title = "[API Image Fonctionelle] : Mandelbrot Julia zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

MandelbrotJulia::~MandelbrotJulia()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJulia::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/**
 * Override
 */
void MandelbrotJulia::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    fractale<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,getT(),t,isJulia,c1,c2);
    //hipDeviceSynchronize();
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJulia::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float MandelbrotJulia::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJulia::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJulia::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJulia::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

