#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJulia.h"
#include "Device.h"
#include "MathTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void mandelbrotJulia(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJulia::MandelbrotJulia(int w, int h, float dt, int n,float xMin,float xMax,float yMin,float yMax,,bool isJulia,float cX=0,float cY=0) :
	variateurAnimation(IntervalF(20, 200),dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;
	this->cX=cX;
    this->cY=cY;
    this->isJulia=isJulia;
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(xMin,yMin,xMax,yMax);

    //Outputs
    this->title = "[API Image Fonctionelle] : MandelbrotJulia zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

MandelbrotJulia::~MandelbrotJulia()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJulia::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void MandelbrotJulia::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    mandelbrotJulia<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t, isJulia, cX, cY);
    }

/*--------------*\
 |*	get	 *|,
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJulia::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float MandelbrotJulia::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJulia::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJulia::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJulia::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

