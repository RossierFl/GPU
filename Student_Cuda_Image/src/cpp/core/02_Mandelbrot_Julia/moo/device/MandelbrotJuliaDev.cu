#include "hip/hip_runtime.h"
#include "IndiceTools.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "DomaineMath.h"
#include "Fractale.h"
#include "MandelbrotMath.h"
#include "JuliaMath.h"

__global__ void fractale(bool isJulia, uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t, float c1, float c2) {
	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;

	Fractale* math = 0;
	if (isJulia) {
		math = new JuliaMath(n, c1, c2);
	} else {
		math = new MandelbrotMath(n);
	}

	int s = TID;
	while (s < WH) {
		int pixelI;
		int pixelJ;
		IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

		// (i,j) domaine ecran
		// (x,y) domaine math
		double x;
		double y;
		domaineMath.toXY(pixelI, pixelJ, &x, &y);

		uchar4 color;
		math->colorXY(&color, x, y, t); // update color
		ptrDevPixels[s] = color;

		s += NB_THREAD;
	}

	delete math;
}

