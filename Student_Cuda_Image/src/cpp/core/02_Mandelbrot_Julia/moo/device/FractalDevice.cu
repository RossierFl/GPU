#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "FractalMath.h"
#include "FractalMandelbrot.h"
#include "FractalJulia.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void fractal(uchar4* ptrDevPixels,int w, int h,bool julia,DomaineMath domaineMath, int n,float t,double cx, double cy);
__device__ void workPixel(uchar4* ptrColorIJ,int i, int j,int s, const DomaineMath& domaineMath,FractalMath* ptrFractalMath,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void fractal(uchar4* ptrDevPixels, int w, int h,bool julia, DomaineMath domaineMath, int n,float t,double cx, double cy)
    {
    FractalMath* fractalMath;
    if(julia){
      	fractalMath = new FractalJulia(n,cx,cy);// ici pour preparer cuda
    }else{
      	fractalMath = new FractalMandelbrot(n);// ici pour preparer cuda
    }
    const int NB_THREAD = Indice2D::nbThread(); // dans region parallel

    const int TID = Indice2D::tid();
    int s = TID; // in [0,...

    int i;
    int j;
    int WH = w*h;
    while (s < WH)
	{
	IndiceTools::toIJ(s,w,&i,&j); // s[0,W*H[ --> i[0,H[ j[0,W[

	workPixel(&ptrDevPixels[s],i, j,s, domaineMath,fractalMath,t);

	s += NB_THREAD;
	}

    }


__device__ void workPixel(uchar4* ptrColorIJ,int i, int j,int s, const DomaineMath& domaineMath,FractalMath* ptrFractalMath, float t)
    {

    // (i,j) domaine ecran dans N2
    // (x,y) domaine math dans R2
   // std::cout<<"print"<<std::endl;
    double x;
    double y;

    domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

    ptrFractalMath->colorXY(ptrColorIJ,x, y, domaineMath, t); // in [01]
    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

