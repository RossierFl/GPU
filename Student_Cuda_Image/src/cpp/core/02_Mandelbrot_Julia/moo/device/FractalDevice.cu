#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "FractalMath.h"
#include "FractalMathMandelbrot.h"
#include "IndiceTools.h"
#include "DomaineMath.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void fractal_gpu(uchar4* ptrDevPixels, int w, int h, int n, DomaineMath& domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void fractal_gpu(uchar4* ptrDevPixels, int w, int h, int n, DomaineMath& domaineMath) {
	FractalMath* fractalMath = new FractalMathMandelbrot(n);

	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;
	int s = TID;
	int i = 0;
	int j = 0;
	double x = 0;
	double y = 0;

	while (s < WH) {
		// job
		IndiceTools::toIJ(s, w, &i, &j);
		domaineMath.toXY(i, j, &x, &y);
		fractalMath->color(x, y, *ptrDevPixels);
		s += NB_THREAD;
	}

	delete fractalMath;
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

