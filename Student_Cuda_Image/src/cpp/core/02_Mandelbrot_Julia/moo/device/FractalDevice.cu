#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "FractalMath.h"
#include <stdio.h>



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void fractal(uchar4* ptrDevPixels, int w, int h, const DomaineMath& domaineMath, int n, double cx, double cyi, bool isJulia);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Code entrainement Cuda
 */
__global__ void fractal(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, double cx, double cyi, bool isJulia)
    {

    FractalMath fractalMath = FractalMath();

    double x = 0.0;
    double y = 0.0;

    const int TID = Indice2D::tid();
    const int NB_THREADS = Indice2D::nbThread();
    const int SIZE = w*h;
    uchar4 color;
    int s = TID;
    int i =0;
    int j =0;
    while(s<SIZE){
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

	domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

	if(isJulia)
	    fractalMath.colorXY(&color,x, y, n, cx, cyi, x, y); // in [01]
	else
	    fractalMath.colorXY(&color,x, y, n, x,y, 0,0);
	ptrDevPixels[s]=color;

	s+=NB_THREADS;
    }
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

