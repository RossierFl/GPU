#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "FractalMath.h"
#include "FractalMathMandelbrot.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void fractalGPU(uchar4* ptrDevPixels, int w, int h, int n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void fractalGPU(uchar4* ptrDevPixels, int w, int h, int n, DomaineMath domaineMath) {
	FractalMath* fractalMath = new FractalMathMandelbrot(n);
	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;
	int s = TID;
	int i = 0;
	int j = 0;
	double x = 0;
	double y = 0;
	uchar4 color;

	while (s < WH) {
		// job
		IndiceTools::toIJ(s, w, &i, &j);
		domaineMath.toXY(i, j, &x, &y);
		fractalMath->color(x, y, &color);
		ptrDevPixels[s] = color;
		s += NB_THREAD;
	}

	delete fractalMath;
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

