#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "MandelbrotJuliaMath.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrotJulia(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath, int n,float t,bool isJulia,float cX=0,float cY=0);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void mandelbrotJulia(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,bool isJulia,float cX=0,float cY=0)
    {
    MandelbrotJuliaMath mandelbrotJuliaMath = MandelbrotJuliaMath(n,isJulia,cX,cY);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)
	
	mandelbrotJuliaMath.colorXY(&color,x, y,domaineMath,t); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

