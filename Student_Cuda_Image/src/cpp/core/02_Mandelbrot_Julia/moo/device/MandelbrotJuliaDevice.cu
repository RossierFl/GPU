#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "MandelbrotJuliaMath.h"
#include "IndiceTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrotJulia(uchar4* ptrDevPixels,int w, int h, DomaineMath & domaineMath,float t,float cX,float cY,bool isJulia);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void mandelbrotJulia(uchar4* ptrDevPixels,int w, int h, DomaineMath & domaineMath,float t,float cX,float cY,bool isJulia)
    {
    MandelbrotJuliaMath MandelbrotJuliaMath(n,isJulia,cX,cY); // ici pour preparer cuda

    // TODO pattern entrelacement




       const int WH = w * h;


       const int TID = Indice2D::tid();
   	const int NB_THREAD = Indice2D::nbThread();// dans region parallel

   	int s = TID; // in [0,...

   	int i;
   	int j;
   	while (s < WH)
   	    {
   	    IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

   	 // (i,j) domaine ecran dans N2
   	   	    // (x,y) domaine math dans R2

   	   	    double x;
   	   	    double y;

   	   	    domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

   	   	   // float t=variateurT.get();
   	   	   // float t=12;
   	   	    ptrMandelbrotJuliaMath->colorXY(ptrColorIJ,x, y, domaineMath,t);

   	    s += NB_THREAD;
   	    }







    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

