#include "hip/hip_runtime.h"
#include <assert.h>

#include "FractalMulti.h"
#include "Device.h"
#include "MathTools.h"
#include <omp.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractalMulti(uchar4* ptrDevPixels, int w, int h,bool julia, DomaineMath domaineMath, int n,float t,double cx, double cy, int imgOffset);
//uchar4* ptrDevPixels,int w, int h,bool julia,DomaineMath domaineMath, int n,double cx, double cy

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

FractalMulti::FractalMulti(int w, int h, int dt,  int nMin,int nMax,bool julia,double cx,double cy,double xMin, double xMax, double yMin, double yMax) :
	variateurAnimation(IntervalI(nMin,nMax), dt)
    {
    // Inputs
    this->first = true;
    this->w = w;
    this->h = h;
    this->julia = julia;
    this->cx = cx;
    this->cy = cy;
    this->n = nMin;
    this->t=0;
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    Device::assertDim(dg, db);
    ptrDomaineMathInit=new DomaineMath(xMin,yMin,xMax,yMax);

    //Outputs
    this->title = "[API Image Fonctionelle] : Fractal zoomable CUDA MULTIGPU Rossier";
    this->nbGPU = 6;
    this->heightBande = h/nbGPU;
    //memory management for 5 GPU
    this->sizeBande=this->heightBande*w*sizeof(uchar4);
    this->bandeI = new uchar4*[5];

    for(int i=0;i<nbGPU-1;i++)
      {
	hipSetDevice(i);
	//uchar4* ptrDevBande;
	//bandeI[i] = ptrDevBande;
	HANDLE_ERROR(hipMalloc((void**) &bandeI[i], sizeBande)); // Device memory allocation (*)
	HANDLE_ERROR(hipMemset(bandeI[i],0,sizeBande)) ;
      }
    hipSetDevice(5);
    // Check:
    //print(dg, db);
    std::cout<<"construc"<<std::endl;
    }

FractalMulti::~FractalMulti()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void FractalMulti::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void FractalMulti::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
{

   //could be more generic
  assert(h%nbGPU==0);//


  omp_set_num_threads(nbGPU);
/*#pragma omp parallel
	{*/
#pragma omp parralel for
  for(int i=0;i<nbGPU;i++)
    {
     // if(!this->first){
	  hipSetDevice(i);
      /*}else{
	  this->first = false;
      }*/

      int offset = w*h/nbGPU;
      int imgOffset = (h/6)*i;
      if(i==5){
	  fractalMulti<<<dg,db>>>(ptrDevPixels+offset*i,w,this->heightBande,julia,domaineMath,n,t,cx,cy,imgOffset);
	  Device::synchronize();

      }else{

	  uchar4* ptrDevBandeInitialGPU = ptrDevPixels+offset*i;
	  fractalMulti<<<dg,db>>>(bandeI[i],w,this->heightBande,julia,domaineMath,n,t,cx,cy,imgOffset);
	  HANDLE_ERROR(hipMemcpy(ptrDevBandeInitialGPU,  bandeI[i], sizeBande, hipMemcpyDeviceToDevice));// Device -> Host
      }
      Device::checkKernelError("fractalMulti");
      //HANDLE_ERROR(hipFree(ptrDevRes)); // device dispose memory in (*)
    }

  /*for(int i=0;i<nbGPU;i++)
    {
      HANDLE_ERROR(hipMemset(bandeI[i],0,sizeBande)) ;
    }*/
	  /*const int TID = omp_get_thread_num();
	  if(TID==0){
	      hipSetDevice(0);

	  }else{
	      hipSetDevice(TID);
	      //fractalMulti<<<dg,db>>>(ptrDevPixels+,w,heightPerGPU,julia,domaineMath,n,t,cx,cy);
	  }*/
	//}

    Device::synchronize();
    Device::checkKernelError("fractalMulti");

    //uchar4* ptrDevPixels,int w, int h,bool julia,DomaineMath domaineMath, int n,double cx, double cy
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* FractalMulti::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float FractalMulti::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int FractalMulti::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int FractalMulti::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string FractalMulti::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

