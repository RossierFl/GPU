#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "NewtonMath.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	float epsilonx, float* jacobiMatrix);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	float epsilonx, float* jacobiMatrix)
    {
    NewtonMath* newtonMath = new NewtonMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	newtonMath->colorXY(&color,x, y,t,epsilonx,jacobiMatrix); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    delete newtonMath;

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

