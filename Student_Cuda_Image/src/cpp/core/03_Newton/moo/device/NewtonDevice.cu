#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "NewtonMath.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, float t,
	float eX, float eF, float eX_Star, float* jacobianMat);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, float t,
	float eX, float eF, float eX_Star,float* jacobianMat)
    {
    NewtonMath* newtonMath = new NewtonMath();

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ);
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)
	newtonMath->colorXY(&color,x, y,t,eX,eF,eX_Star,jacobianMat);
	ptrDevPixels[s] = color;
	s += NB_THREAD;
	}
    delete newtonMath;

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
