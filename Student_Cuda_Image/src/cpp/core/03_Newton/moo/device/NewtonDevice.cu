#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "NewtonMath.h"



/*----------------------------------------------------------------------*\
|*			Declaration 		                            			*|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported	                	*|
\*--------------------------------------*/

/*--------------------------------------*\
|*		Public			                *|
\*--------------------------------------*/

__global__ void fractalNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float epsilonx, float epsilonf, float epsilonxstar);

/*--------------------------------------*\
|*		Private		                 	*|
\*--------------------------------------*/

/*----------------------------------------------------------------------*\
|*			Implementation 				                            	*|
\*----------------------------------------------------------------------*/

__global__ void fractalNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float epsilonx, float epsilonf, float epsilonxstar)
{
	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;

	uchar4 color;
	double x;
	double y;
	int i;
	int j;
	NewtonMath newtonMath(n);
	int s = TID;
	while (s < WH)
	{
		// Compute (i,j)
		// (i,j) screen domain
		// (x,y) math domain
		IndiceTools::toIJ(s, w, &i, &j);

		// Compute (x,y)
		//  (i,j) -> (x,y)
		domaineMath.toXY(i, j, &x, &y);

		// Compute the color
		newtonMath.colorXY(&color, x, y, epsilonx, epsilonf, epsilonxstar);

		// Apply the color
		ptrDevPixels[s] = color;

		//
		s += NB_THREAD;
	}
}


