#include "hip/hip_runtime.h"
#include <assert.h>

#include "Damier.h"
#include "Device.h"
#include "MathTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damier(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Damier::Damier(int w, int h, float dt, int n) :
	variateurAnimation(IntervalF(0, 2 * PI), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(0,0,2*PI,2*PI);

    //Outputs
    this->title = "[API Image Fonctionelle] : Damier zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

Damier::~Damier()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Damier::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void Damier::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    damier<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Damier::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Damier::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Damier::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Damier::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Damier::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

