#include "hip/hip_runtime.h"
#include <assert.h>

#include "Newton.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

#define MAX_T 40
#define MAX_N 25

/*----------------------------------------------------------------------*\
|*			Declaration 												*|
\*----------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported					 	*|
\*--------------------------------------*/

__global__ void fractalNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float epsilonx, float epsilonf, float epsilonxstar);

/*--------------------------------------*\
|*		Public							*|
\*--------------------------------------*/

/*--------------------------------------*\
|*		Private							*|
\*--------------------------------------*/

/*----------------------------------------------------------------------*\
|*			Implementation 												*|
\*----------------------------------------------------------------------*/

/*-------------------------*\
|*	Constructor		       *|
\*-------------------------*/

Newton::Newton(int w, int h, float dt, float epsilonx, float epsilonf, float epsilonxstar, double x1, double y1, double x2, double y2)
		:
		animationVariator(IntervalF(30, 100), dt)
{
	// Inputs
	this->w = w;
	this->h = h;
	this->n = n;

	this->epsilonx = epsilonx;
	this->epsilonf = epsilonf;
	this->epsilonxstar = epsilonxstar;

	// Tools
	this->dg = dim3(8, 8, 1); // TODO à optimiser
	this->db = dim3(16, 16, 1); // TODO à optimiser
	this->t = 0;
	this->tAdd = true;
	ptrDomaineMathInit = new DomaineMath(x1, y1, x2, y2);

	Device::assertDim(dg, db);
	assert(w == h);
}

Newton::~Newton()
{
	delete ptrDomaineMathInit;
}

/*------------------------*\
|*	Methods	  			  *|
\*------------------------*/

void Newton::animationStep()
{
	t++;
	if(t > MAX_T) {
		t = 0;
		if(tAdd) {
			this->n++;
			if(this->n >= MAX_N) {
				this->tAdd = !this->tAdd;
			}
		} else {
			this->n--;
			if(this->n <= 0) {
				this->tAdd = !this->tAdd;
			}
		}
	}
}

void Newton::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
{
	// Run the computation on the GPU
	// @formatter:off
	fractalNewton<<<dg,db>>>(ptrDevPixels, w, h, domaineMath, n, epsilonx, epsilonf, epsilonxstar);
	// @formatter:on
}

DomaineMath* Newton::getDomaineMathInit()
{
	return ptrDomaineMathInit;
}

float Newton::getT()
{
	return n;
}

int Newton::getW()
{
	return w;
}

int Newton::getH()
{
	return h;
}

std::string Newton::getTitle()
{
	return "Newton";
}


