#include "hip/hip_runtime.h"
#include <assert.h>

#include "Newton.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, float t,
	float eX, float eF, float eX_Star,float* jacobianMat);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Newton::Newton(int w, int h, float dt, double x1, double y1, double x2, double y2, float eX, float eF, float eX_Star) :
		variateurAnimation(IntervalF(0, 50), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;

    this->eX = eX;
    this->eF = eF;
    this->eX_Star = eX_Star;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = variateurAnimation.varierAndGet();
    ptrDomaineMathInit=new DomaineMath(x1,y1,x2,y2);

    //Outputs
    this->title = "[API Image Fonctionelle] : Newton zoomable CUDA -- Emmanuel Dafflon";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

Newton::~Newton()
    {
    delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Newton::animationStep()
    {
    this->t = variateurAnimation.varierAndGet();
    }

/**
 * Override
 */
void Newton::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    float* jacobianMat = NULL; //create pointer
    HANDLE_ERROR(hipMalloc(&jacobianMat,4*sizeof(float)));//malloc 4 floats
	HANDLE_ERROR(hipMemset(jacobianMat,0,4*sizeof(float)));//fill with 0
    fractaleNewton<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,t,eX,eF,eX_Star,jacobianMat);//call the fractal function
    hipFree(jacobianMat);//free mat
    //hipDeviceSynchronize(); // in case of issues 
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Newton::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Newton::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Newton::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Newton::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Newton::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
