#include "hip/hip_runtime.h"
#include <assert.h>

#include "Newton.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	float epsilonx, float* jacobiMatrix);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Newton::Newton(int w, int h, float dt, int n, float epsilonx,
	double c1, double c2, double x1, double y1, double x2, double y2) :
		variateurAnimation(IntervalF(30, 100), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    this->c1 = c1;
    this->c2 = c2;

    this->epsilonx = epsilonx;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(x1,y1,x2,y2);

    //Outputs
    this->title = "[API Image Fonctionelle] : Mandelbrot Julia zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

Newton::~Newton()
    {
    delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Newton::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [30,100]
    }

/**
 * Override
 */
void Newton::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    float* jacobiMatrix;
    hipMalloc((void**)&jacobiMatrix,4*sizeof(jacobiMatrix[0]));
    fractaleNewton<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t,epsilonx,jacobiMatrix);
    hipFree(jacobiMatrix);
    //hipDeviceSynchronize();
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Newton::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Newton::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Newton::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Newton::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Newton::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

