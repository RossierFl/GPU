#include "hip/hip_runtime.h"
#include <assert.h>

#include "Newton.h"
#include "Device.h"
#include "MathTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void newton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Newton::Newton(int w, int h, float dt, int n) :
	variateurAnimation(IntervalF(0, 2 * PI), dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(0,0,2*PI,2*PI);

    //Outputs
    this->title = "[API Image Fonctionelle] : Newton zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

Newton::~Newton()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Newton::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void Newton::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    newton<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Newton::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Newton::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Newton::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Newton::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Newton::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

