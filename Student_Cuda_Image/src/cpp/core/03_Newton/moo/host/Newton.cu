#include "hip/hip_runtime.h"
#include <assert.h>

#include "Newton.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
|*			Declaration 												*|
\*----------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported					 	*|
\*--------------------------------------*/

__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float epsilonx, float epsilonf, float epsilonxstar, float* jacobiMatrix);

/*--------------------------------------*\
|*		Public							*|
\*--------------------------------------*/

/*--------------------------------------*\
|*		Private							*|
\*--------------------------------------*/

/*----------------------------------------------------------------------*\
|*			Implementation 												*|
\*----------------------------------------------------------------------*/

/*-------------------------*\
|*	Constructor		       *|
\*-------------------------*/

Newton::Newton(int w, int h, float dt, int n, float epsilonx, float epsilonf, float epsilonxstar, double x1, double y1, double x2, double y2)
		:
		animationVariator(IntervalF(30, 100), dt)
{
	// Inputs
	this->w = w;
	this->h = h;
	this->n = n;

	this->epsilonx = epsilonx;
	this->epsilonf = epsilonf;
	this->epsilonxstar = epsilonxstar;

	// Tools
	this->dg = dim3(8, 8, 1); // TODO à optimiser
	this->db = dim3(16, 16, 1); // TODO à optimiser
	this->t = 0;
	ptrDomaineMathInit = new DomaineMath(x1, y1, x2, y2);

	Device::assertDim(dg, db);
	assert(w == h);
}

Newton::~Newton()
{
	delete ptrDomaineMathInit;
}

/*------------------------*\
|*	Methods	  			  *|
\*------------------------*/

void Newton::animationStep()
{
	this->t = animationVariator.varierAndGet(); // in [30,100]
}

void Newton::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
{
	// Run the computation on the GPU
	// @formatter:off
	fractalNewton<<<dg,db>>>(ptrDevPixels, w, h, domaineMath, n, epsilonx, epsilonf, epsilonxstar);
	// @formatter:on
}

DomaineMath* Newton::getDomaineMathInit()
{
	return ptrDomaineMathInit;
}

float Newton::getT()
{
	return t;
}

int Newton::getW()
{
	return w;
}

int Newton::getH()
{
	return h;
}

std::string Newton::getTitle()
{
	return "Newton";
}


