#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Rippling.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJulia::MandelbrotJulia(int w, int h, float dt)
    {
    assert(w == h);

    // Inputs
    this->w = w;
    this->h = h;
    this->dt = dt;

    // Tools
    this->dg =  dim3(8,8,1);// TODO
    this->db =  dim3(16,16,1); // TODO
    this->t = 0;

    // Outputs
    this->title = "Rippling Cuda";

    //print(dg, db);
    Device::assertDim(dg, db);
    }

MandelbrotJulia::~MandelbrotJulia()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJulia::animationStep()
    {
    // TODO
     t+=dt;
    }

/**
 * Override
 */
void MandelbrotJulia::runGPU(uchar4* ptrDevPixels)
    {
    // TODO lancer le kernel avec <<<dg,db>>>
    rippling<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float MandelbrotJulia::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJulia::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJulia::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJulia::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

