#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Rippling.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling::Rippling(int w, int h, float dt)
    {
    assert(w == h);

    // Inputs
    this->w = w;
    this->h = h;
    this->dt = dt;

    // Tools
    this->dg =  dim3(8,8,1);// TODO
    this->db =  dim3(16,16,1); // TODO
    this->t = 0;

    // Outputs
    this->title = "Rippling Cuda";

    //print(dg, db);
    Device::assertDim(dg, db);
    }

Rippling::~Rippling()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Rippling::animationStep()
    {
    // TODO
     t+=dt;
    }

/**
 * Override
 */
void Rippling::runGPU(uchar4* ptrDevPixels)
    {
    // TODO lancer le kernel avec <<<dg,db>>>
    rippling<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Rippling::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Rippling::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Rippling::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Rippling::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

