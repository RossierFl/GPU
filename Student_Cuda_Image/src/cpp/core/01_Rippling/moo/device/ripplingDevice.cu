#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "RipplingMath.h"
#include "IndiceTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t)
    {
    RipplingMath ripplingMath = RipplingMath(w, h);

    // TODO pattern entrelacement



       const int WH = w * h;


       const int TID = Indice2D::tid();
   	const int NB_THREAD = Indice2D::nbThread();// dans region parallel

   	int s = TID; // in [0,...

   	int i;
   	int j;
   	while (s < WH)
   	    {
   	    IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

   	    ripplingMath.color(ptrDevPixels[s], i, j,t);

   	    s += NB_THREAD;
   	    }


    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

