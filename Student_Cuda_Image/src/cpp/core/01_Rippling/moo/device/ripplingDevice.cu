#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "RipplingMath.h"
#include "IndiceTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t) {
	RipplingMath ripplingMath = RipplingMath(w, h);

	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;
	int s = TID;
	int i = 0;
	int j = 0;

	while(s < WH) {
		// job
		IndiceTools::toIJ(s, w, &i, &j);
		ripplingMath.color(i, j, t, ptrDevPixels[s]);

		s += NB_THREAD;
	}
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

