#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"

#include "RipplingMath.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t)
    {
    RipplingMath ripplingMath = RipplingMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREADS = Indice2D::nbThread();
    const int SIZE = w*h;

    int s = TID;
    int i =0;
    int j =0;
    while(s<SIZE){
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[
	ripplingMath.colorIJ(&ptrDevPixels[s], i, j,t);
	s+=NB_THREADS;
    }
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

