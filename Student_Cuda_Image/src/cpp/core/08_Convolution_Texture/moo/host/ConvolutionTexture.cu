#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <assert.h>

#include "ConvolutionTexture.h"
//#include "ConvolutionMathTexture.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

//extern __global__ void convolutionKernelTexture(texture<uchar4,2> tex, uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t);
extern __global__ void colorToGreyTexture(uchar4* ptrDevPixels, int w, int h);
extern __global__ void findMinMaxTexture(uchar4* ptrDevPixels, uchar* ptrDevResult,int w, int h);
extern __global__ void affineTransformTexture(uchar4* ptrDevPixels, float a, float b, int w, int h, int offset);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

texture<uchar4, 2> textureRef;

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__device__
void colorIJ(uchar4* ptrColor, float* ptrDeviceNoyau, int k, int i, int j, int s)
    {

    float sum = 0.0f;
    int ss = (int)(k*((float)k/2.0f));
    int k2 = k/2;
    for(int v = 1;v<=k2;v++)
	{
	for(int u = 1;u<=k2;u++)
	    {
	    // bas droite
	    sum+=ptrDeviceNoyau[(ss+v*k)+u]*tex2D(textureRef,j+u,i+v).x;
	    // haut droite
	    sum+=ptrDeviceNoyau[(ss-v*k)+u]*tex2D(textureRef,j+u,i-v).x;
	    // bas gauche
	    sum+=ptrDeviceNoyau[(ss+v*k)-u]*tex2D(textureRef,j-u,i+v).x;
	    // haut gauche
	    sum+=ptrDeviceNoyau[(ss-v*k)-u]*tex2D(textureRef,j-u,i-v).x;
	    }
	// bras east
	sum+=ptrDeviceNoyau[ss+v]*tex2D(textureRef,j,i+v).x;
	// bras west
	sum+=ptrDeviceNoyau[ss-v]*tex2D(textureRef,j,i-v).x;
	// bras south
	sum+=ptrDeviceNoyau[ss+v*k]*tex2D(textureRef,j+v,i).x;
	// bras north
	sum+=ptrDeviceNoyau[ss-v*k]*tex2D(textureRef,j-v,i).x;
	}
    // centre
    sum+=ptrDeviceNoyau[ss]*tex2D(textureRef,j,i).x;
    ptrColor->x = sum;
    ptrColor->y = sum;
    ptrColor->z = sum;

    ptrColor->w = 255; // opaque
    }

__global__ void convolutionKernelTexture(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t)
    {

    extern __shared__ float convSM[];// 1 instance per block !

    //ConvolutionMathTexture convMath = ConvolutionMathTexture(w, h);

    const int TID = Indice2D::tid();
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID_LOCAL;
    if(s < (k*k))
	convSM[s] = ptrDeviceNoyau[s];
    __syncthreads();

    s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)
	colorIJ(&color,convSM,k,pixelI, pixelJ, s); 	// update color
	ptrDevPixels[s] = color;
	s += NB_THREAD;
	}
    }

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

ConvolutionTexture::ConvolutionTexture()
    {
    // Inputs
    this->w = 640;
    this->h = 360;
    this->k = 9;

    // Tools
    this->dg = dim3(16, 1, 1); // disons a optimiser
    this->db = dim3(128, 1, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : Convolution CUDA";
    this->videoPath="/media/Data/Video/nasaFHD_short.avi";
    this->videoTitle="NASA FHD SHORT";

    this->videoPath="/media/Data/Video/neilPryde.avi";
    this->videoTitle="neilPryde";

    size_t sizeOctets=w*h*sizeof(uchar4);
    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostMemory), sizeOctets,hipHostMallocDefault ) );

    this->captureur = new CaptureVideo(videoPath,videoTitle,this->ptrHostMemory);

    const int N = k*k;
    sizeConvSM = N*sizeof(float);

    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostNoyau), N*sizeof(float),hipHostMallocDefault ) );

    fillDetourage(ptrHostNoyau);

    HANDLE_ERROR(hipMalloc((void **)&ptrDeviceNoyau,N*sizeof(float)));
    HANDLE_ERROR(hipMemcpy(ptrDeviceNoyau,ptrHostNoyau,N*sizeof(float),hipMemcpyHostToDevice));


    sizeSM = 2*db.x*sizeof(uchar);
    sizeResult=2*dg.x*sizeof(uchar);
    ptrHostResult = (uchar*) malloc(sizeResult);

    // first contains min of each block, then max of each block
    HANDLE_ERROR(hipMalloc((void**) &ptrDevResult, sizeResult));

    textureRef.filterMode= hipFilterModePoint;
    textureRef.normalized=false; //ou true coordonnée texture (i,j)
    textureRef.addressMode[0] = hipAddressModeClamp;
    textureRef.addressMode[1] = hipAddressModeClamp;

    pitch = w * sizeof(uchar4); //taille en octets d'une ligne
    channelDesc = hipCreateChannelDesc<uchar4>();

    //cout << endl<<"[CBI] Convolution dt =" << dt << endl;
    }

ConvolutionTexture::~ConvolutionTexture()
    {
    free(ptrHostResult);
    hipFree(ptrDevResult);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Noyau detourage 9x9
 */
void ConvolutionTexture::fillDetourage(float* ptrNoyau)
    {
    const int N = 9*9;
    // Tab auto temporaire
    float tab[N] =
	    {
		    0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.6063, 1.2765, 0.4061, -6.4829, -12.7462, -6.4829, 0.4061, 1.2765, 0.6063, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828
	    };

    for (int i = 1; i <= N; i++)
	{
	ptrNoyau[i - 1] = tab[i - 1] / 100.0f;
	}
    }

/**
 * Override
 * Call periodicly by the API
 */
void ConvolutionTexture::animationStep()
    {
    //t+=dt;
    }

/**
 * Override
 */
void ConvolutionTexture::runGPU(uchar4* ptrDevPixels)
    {
    Mat matImage = captureur->capturer();
    uchar4* image = CaptureVideo::castToUChar4(&matImage);

    HANDLE_ERROR(hipMemcpy(ptrDevPixels,image,(w*h)*sizeof(ptrDevPixels[0]),hipMemcpyHostToDevice));

    hipBindTexture2D(NULL, textureRef,ptrDevPixels,channelDesc,w,h,pitch);

    colorToGreyTexture<<<dg,db>>>(ptrDevPixels,w,h);
    HANDLE_ERROR(hipDeviceSynchronize());

    convolutionKernelTexture<<<dg,db,sizeConvSM>>>(ptrDevPixels,ptrDeviceNoyau,k,w,h,t);
    HANDLE_ERROR(hipDeviceSynchronize());

    findMinMaxTexture<<<dg,db,sizeSM>>>(ptrDevPixels,ptrDevResult,w,h);
    HANDLE_ERROR(hipMemcpy(ptrHostResult, ptrDevResult, sizeResult, hipMemcpyDeviceToHost));

    uchar max = 0;
    uchar min = 255;
    for(int i = 0;i<dg.x;i++)
	{
	uchar crt = ptrHostResult[i];
	if(crt < min)
	    min = crt;
	crt = ptrHostResult[i+dg.x];
	if(crt > max)
	    max = crt;
	}
    // affine transformation
    float a = 255.0f/(float)(max-min);
    float b = 0;
    if(min != 0)
	b = 255.0f/((-max/(float)min)+1.0f);
    affineTransformTexture<<<dg,db>>>(ptrDevPixels, a, b, w, h,0);
    //printf("min: %d, max: %d\n",min,max);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float ConvolutionTexture::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int ConvolutionTexture::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int ConvolutionTexture::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string ConvolutionTexture::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

