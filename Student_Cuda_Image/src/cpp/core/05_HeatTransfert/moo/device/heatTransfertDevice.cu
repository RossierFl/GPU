#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "HeatTransfertMath.h"
#include "CalibreurCudas.h"
#include "ColorTools_GPU.h"
#include <stdio.h>


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void heatTransfertDiffusion(float* ptrDevImageA,float* ptrDevImageB, bool isImageAInput, int w, int h,float k);
__global__ void heatTransfertEcrasement(float* ptrDevInputHeater, float* ptrDevInputImage,float* ptrDevOutput, int w, int h);
__global__ void heatTransfertConvertData(CalibreurCudas calibreurCuda,float* ptrDevInput, uchar4* ptrDevImage, int w, int h);



/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void heatTransfertDiffusion(float* ptrDevImageA,float* ptrDevImageB, bool isImageAInput, int w, int h, float k){
      HeatTransfertMath* heatTransfertMath = new HeatTransfertMath(k);
      /*if(!isImageAInput){//switch it here to avoid another if later, all thread should do the same here
	  float* tmp = ptrDevImageB;
	  ptrDevImageB = ptrDevImageA;
	  ptrDevImageA = tmp;
      }*/
      const int TID = Indice2D::tid();
      const int NB_THREAD = Indice2D::nbThread();

      const int WH=w*h;


      int pixelI;
      int pixelJ;

      int s = TID;
      while (s < WH)
  	{
	  IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)
	  if(pixelI!=0&&pixelI!=h&&pixelJ!=0&&pixelJ!=w){//not on the side of the image
		  if(!isImageAInput)heatTransfertMath->getNewPixelValue(&ptrDevImageB[s],ptrDevImageA,k,s,w); // update color
		  else heatTransfertMath->getNewPixelValue(&ptrDevImageA[s],ptrDevImageB,k,s,w); // update color
		  //ptrDevImageB[s] = newPixelValue;
	  }
	  s += NB_THREAD;
  	}
      delete heatTransfertMath;
}
__global__ void heatTransfertEcrasement(float* ptrDevInputHeater, float* ptrDevInputImage,float* ptrDevOutput, int w, int h){

      const int TID = Indice2D::tid();
      //if(TID==0)printf("start heatTransfertEcrasement\n");
      const int NB_THREAD = Indice2D::nbThread();

      const int WH=w*h;


      //int pixelI;
      //int pixelJ;

      int s = TID;
      while (s < WH)
      {
	  //IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	  // (i,j) domaine ecran
	  //float* pixelValue = 0.0;
	  ptrDevOutput[s] = ptrDevInputImage[s];
	  if(ptrDevInputHeater[s]!=0.0){//ok an if, but most of the image will be copied
	      ptrDevOutput[s]=ptrDevInputHeater[s];
	  }
	  s += NB_THREAD;
      }
      //if(TID==0)printf("end heatTransfertEcrasement\n");
}
__global__ void heatTransfertConvertData(CalibreurCudas calibreurCuda, float* ptrDevInput, uchar4* ptrDevImage, int w, int h){
	const int TID = Indice2D::tid();
        const int NB_THREAD = Indice2D::nbThread();

        const int WH=w*h;


        //int pixelI;
        //int pixelJ;

        int s = TID;
        while (s < WH)
        {
  	  //IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

  	  // (i,j) domaine ecran
  	  //float* pixelValue = 0.0;
            float hue = calibreurCuda.calibrate(ptrDevInput[s]);
            ColorTools::HSB_TO_RVB(hue, &ptrDevImage[s]);
            s += NB_THREAD;
        }
}
/*__global__ void fractaleNewton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,
	float epsilonx, float epsilonf, float epsilonxstar,float* jacobiMatrix)
    {
    NewtonMath* newtonMath = new NewtonMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	newtonMath->colorXY(&color,x, y,t,epsilonx,epsilonf,epsilonxstar,jacobiMatrix); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    delete newtonMath;

    }*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

