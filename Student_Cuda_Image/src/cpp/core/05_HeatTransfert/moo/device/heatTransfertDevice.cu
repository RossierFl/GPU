#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "HeatTransfertMath.h"
#include "CalibreurCudas.h"
#include "ColorTools_GPU.h"
#include <stdio.h>

__global__ void heatTransfertDiffusion(float* ptrDevImageA, float* ptrDevImageB, bool isImageAInput, int w, int h, float k);
__global__ void heatTransfertEcrasement(float* ptrDevInputHeater, float* ptrDevInputImage, float* ptrDevOutput, int w, int h);
__global__ void heatTransfertConvertData(CalibreurCudas calibreurCuda, float* ptrDevInput, uchar4* ptrDevImage, int w, int h);

__global__ void heatTransfertDiffusion(float* ptrDevImageA, float* ptrDevImageB, bool isImageAInput, int w, int h, float k) {
	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;

	HeatTransfertMath* heatTransfertMath = new HeatTransfertMath();

	int s = TID;
	while (s < WH) {
		int pixelI = 0;
		int pixelJ = 0;
		IndiceTools::toIJ(s, w, &pixelI, &pixelJ);

		if (pixelI != 0 && pixelI != h && pixelJ != 0 && pixelJ != w) { //not on the side of the image
			if (isImageAInput) {
				heatTransfertMath->getNewPixelValue(&ptrDevImageA[s], ptrDevImageB, k, s, w); // update color
			} else {
				heatTransfertMath->getNewPixelValue(&ptrDevImageB[s], ptrDevImageA, k, s, w); // update color
			}
		}

		s += NB_THREAD;
	}

	delete heatTransfertMath;
}
__global__ void heatTransfertEcrasement(float* ptrDevInputHeat, float* ptrDevInputImg, float* ptrDevOut, int w, int h) {
	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;

	uint s = TID;
	while (s < WH) {
		ptrDevOut[s] = ptrDevInputImg[s];

		if (ptrDevInputHeat[s] != 0.0) {	  //ok an if, but most of the image will be copied
			ptrDevOut[s] = ptrDevInputHeat[s];
		}

		s += NB_THREAD;
	}
}
__global__ void heatTransfertConvertData(CalibreurCudas calibreurCuda, float* ptrDevInput, uchar4* ptrDevImg, int w, int h) {
	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w * h;

	int s = TID;
	while (s < WH) {
		float hue = calibreurCuda.calibrate(ptrDevInput[s]);
		ColorTools::HSB_TO_RVB(hue, &ptrDevImg[s]);

		s += NB_THREAD;
	}
}
