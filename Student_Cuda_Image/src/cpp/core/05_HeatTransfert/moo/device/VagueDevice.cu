#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"

#include "VagueMath.h"

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vague(uchar4* ptrDevPixels,int w, int h,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vague(uchar4* ptrDevPixels, int w, int h, float t)
    {
    VagueMath vagueMath = VagueMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	vagueMath.colorIJ(&color,pixelI, pixelJ, t); 	// update color
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

