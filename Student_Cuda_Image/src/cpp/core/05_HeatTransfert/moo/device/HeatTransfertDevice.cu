#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "HeatTransfertMath.h"
#include "CalibreurCuda.h"
#include "ColorTools_GPU.h"
#include <stdio.h>


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void diffusion(float* ptrDevImageA,float* ptrDevImageB, int w, int h, float k, bool imgAIsInput);
__global__ void ecrasement(float* ptrDevInputHeater, float* ptrDevInputImage,float* ptrDevOutput, int w, int h);
__global__ void toScreenImageHSB(CalibreurCuda calibreurCuda,float* ptrDevInput, uchar4* ptrDevImage, int w, int h);



/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void diffusion(float* ptrDevImageA,float* ptrDevImageB, int w, int h, float k, bool imgAIsInput){
      HeatTransfertMath* math = new HeatTransfertMath(k);

      const int TID = Indice2D::tid();
      const int NB_THREAD = Indice2D::nbThread();

      const int WH=w*h;

      int pixelI;
      int pixelJ;

      int s = TID;
      while (s < WH)
  	{
	  IndiceTools::toIJ(s, w, &pixelI, &pixelJ);
	  // 1 pixel band around image not calc to avoid 9 branch
	  if(pixelI !=0 && pixelI != h && pixelJ != 0 && pixelJ != w){
		  if(!imgAIsInput)
		      math->updateHeat(&ptrDevImageB[s],ptrDevImageA,k,s,w); // update color
		  else
		      math->updateHeat(&ptrDevImageA[s],ptrDevImageB,k,s,w); // update color
	  }
	  s += NB_THREAD;
  	}
      delete math;
}
__global__ void ecrasement(float* ptrDevInputHeater, float* ptrDevInputImage,float* ptrDevOutput, int w, int h){

      const int TID = Indice2D::tid();

      const int NB_THREAD = Indice2D::nbThread();

      const int WH=w*h;

      int s = TID;
      while (s < WH)
      {
	  ptrDevOutput[s] = ptrDevInputImage[s];
	  if(ptrDevInputHeater[s]!=0.0){
	      ptrDevOutput[s]=ptrDevInputHeater[s];
	  }
	  s += NB_THREAD;
      }
}
__global__ void toScreenImageHSB(CalibreurCuda calibreurCuda, float* ptrDevInput, uchar4* ptrDevImage, int w, int h){
	const int TID = Indice2D::tid();
        const int NB_THREAD = Indice2D::nbThread();

        const int WH=w*h;

        int s = TID;
        while (s < WH)
        {
            float hue = calibreurCuda.calibrer(ptrDevInput[s]);
            ColorTools::HSB_TO_RVB(hue, &ptrDevImage[s]);
            s += NB_THREAD;
        }
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

