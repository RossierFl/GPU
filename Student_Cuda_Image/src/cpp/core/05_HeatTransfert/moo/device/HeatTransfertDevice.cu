#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"


#include "HeatTransfertMath.h"

// Attention : 	Choix du nom est impotant!
//		HeatTransfertDevice.cu et non HeatTransfert.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void heatTransfert(float* ptrDevImageAInput, float* ptrDevImageBOutput,int w, int h);
__global__ void heatEcrasement(float* ptrDevImageInput,float* ptrDevImageHeaters ,float* ptrDevImageOutput,int w,int h);
__global__ void heatToScreenImageHSB(float* ptrDevImageInput, uchar4* ptrDevImageGL, int w, int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

//Diffusion
__global__ void heatTransfert(float* ptrDevImageAInput, float* ptrDevImageBOutput,int w, int h)
    {
    HeatTransfertMath heatTransfertMath = HeatTransfertMath();

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    float color;
    float colorSud;
    float colorEst;
    float colorNord;
    float colorOuest;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)
	if(pixelI!=0 || pixelJ!=0 || pixelI<w-1 || pixelJ<h-1){
	int sS = IndiceTools::toS(w,pixelI,pixelJ-1);
	int sE = IndiceTools::toS(w,pixelI+1,pixelJ);
	int sN = IndiceTools::toS(w,pixelI,pixelJ+1);
	int sO = IndiceTools::toS(w,pixelI-1,pixelJ);
	colorSud = ptrDevImageAInput[sS];
	colorEst = ptrDevImageAInput[sE];
	colorNord= ptrDevImageAInput[sN];
	colorOuest = ptrDevImageAInput[sO];
	heatTransfertMath.calculeColorTransfert(&color,colorSud,colorEst,colorNord,colorOuest)
	heatTransfertMath.colorIJ(&color,pixelI, pixelJ, t); 	// update color
	ptrDevImageBOutput[s] = color;
	}

	s += NB_THREAD;
	}
    }

//Ecrasement entre les heater et le résulat des diffusion
__global__ void heatEcrasement(float* ptrDevImageInput,float* ptrDevImageHeaters ,float* ptrDevImageOutput,int w,int h)
    {
    HeatTransfertMath heatTransfertMath = HeatTransfertMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();


    int s = TID;
    while (s < WH)
	{
	//IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	if(ptrDevImageHeaters[s]>0){
	    ptrDevImageOutput[s] = ptrDevImageHeaters[s];
	}else{
	    ptrDevImageOutput[s] = ptrDevImageInput[s];
	}

	s += NB_THREAD;
	}
    }


//Ecrasement entre les heater et le résulat des diffusion
__global__ void heatToScreenImageHSB(float* ptrDevImageInput, uchar4* ptrDevImageGL, int w, int h)
    {
    HeatTransfertMath heatTransfertMath = HeatTransfertMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	heatTransfertMath.convertFloatToColor(ptrDevImageInput[s],&color);	// update color
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

