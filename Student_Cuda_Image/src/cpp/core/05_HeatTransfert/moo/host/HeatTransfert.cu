#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "HeatTransfert.h"
#include "Device.h"
#include "MathTools.h"

#define NB_ITERATION_AVEUGLE 2

extern __global__ void heatTransfertDiffusion(float* ptrDevImageA,float* ptrDevImageB, bool isImageAInput, int w, int h,float k);
extern __global__ void heatTransfertEcrasement(float* ptrDevInputHeater, float* ptrDevInputImage,float* ptrDevOutput, int w, int h);
extern __global__ void heatTransfertConvertData(CalibreurCudas calibreurCuda,float* ptrDevInput, uchar4* ptrDevImage, int w, int h);

HeatTransfert::HeatTransfert(int w, int h, float k) :
		calibreurCuda(0.0, 1.0, 0.7, 0.0) {
	// Inputs
	this->w = w;
	this->h = h;
	this->k = k;
	// Tools
	this->dg = dim3(8, 8, 1); // disons a optimiser
	this->db = dim3(16, 16, 1); // disons a optimiser

	//Outputs
	this->title = "[API Image] : HeatTransfert CUDA";

	// Check:
	Device::assertDim(dg, db);
	assert(w == h);
	size_t sizeImages = w * h * sizeof(float);
	//init
	//imageInit all to zero
	HANDLE_ERROR(hipMalloc((void** ) &ptrDevImageInit, sizeImages)); // Device memory allocation (*)
	HANDLE_ERROR(hipMemset(ptrDevImageInit, 0, sizeImages));

	HANDLE_ERROR(hipMalloc((void** ) &ptrDevImageA, sizeImages)); // Device memory allocation (*)
	HANDLE_ERROR(hipMemset(ptrDevImageA, 0, sizeImages));

	HANDLE_ERROR(hipMalloc((void** ) &ptrDevImageB, sizeImages)); // Device memory allocation (*)
	HANDLE_ERROR(hipMemset(ptrDevImageB, 0, sizeImages));

	//imageHeater all zero with some heater or "cooler"
	HANDLE_ERROR(hipMalloc((void** ) &ptrDevImageHeater, sizeImages)); // Device memory allocation (*)
	HANDLE_ERROR(hipMemset(ptrDevImageHeater, 0, sizeImages));

	float *imageHeaterCPU = new float[w * h];

	// ary[i][j] is then rewritten as imageHeaterCPU[i*h+j]
	for (int i = 0; i <= h; i++)    //take it to zero to avoid something strange
	        {
		for (int j = 0; j <= w; j++) {
			imageHeaterCPU[i * h + j] = 0.0;
		}
	}
	//single cooler point
	imageHeaterCPU[295 * h + 400] = 0.2;
	imageHeaterCPU[400 * h + 295] = 0.2;
	imageHeaterCPU[505 * h + 400] = 0.2;
	imageHeaterCPU[400 * h + 505] = 0.2;

	//up cooler
	for (int i = 179; i <= 195; i++) {
		for (int j = 179; j <= 195; j++) {
			imageHeaterCPU[i * h + j] = 0.2;
		}
		for (int j = 605; j <= 621; j++) {
			imageHeaterCPU[i * h + j] = 0.2;
		}
	}

	//down cooler
	for (int i = 605; i <= 621; i++) {
		for (int j = 179; j <= 195; j++) {
			imageHeaterCPU[i * h + j] = 0.2;
		}
		for (int j = 605; j <= 621; j++) {
			imageHeaterCPU[i * h + j] = 0.2;
		}
	}

	//main heater
	for (int i = 300; i <= 500; i++) {
		for (int j = 300; j <= 500; j++) {
			imageHeaterCPU[i * h + j] = 1.0;
		}
	}

	//copy this tab to GPU
	HANDLE_ERROR(hipMemcpy(ptrDevImageHeater, imageHeaterCPU, sizeImages, hipMemcpyHostToDevice));    // Host -> Device
	delete[] imageHeaterCPU;

	//ecrase A with imageHeater;

	heatTransfertEcrasement<<<dg,db>>>(ptrDevImageHeater,ptrDevImageA,ptrDevImageA, w, h);
	Device::synchronize();
	Device::checkKernelError("heatTransfertEcrasement");
}

HeatTransfert::~HeatTransfert() {
	HANDLE_ERROR(hipFree(ptrDevImageHeater));
	HANDLE_ERROR(hipFree(ptrDevImageInit));
	HANDLE_ERROR(hipFree(ptrDevImageA));
	HANDLE_ERROR(hipFree(ptrDevImageB));
}

/**
 * Override
 */
void HeatTransfert::animationStep() {
}

/**
 * Override
 */
void HeatTransfert::runGPU(uchar4* ptrDevPixels) {
	//iteration
	float* iterationOutput;
	for (int i = 0; i <= NB_ITERATION_AVEUGLE; i++) {
		heatTransfertDiffusion<<<dg,db>>>(ptrDevImageA,ptrDevImageB, (i+1)%2, w, h, k);
		Device::synchronize();

		if ((i + 1) % 2 == 1)
			iterationOutput = ptrDevImageB;
		else
			iterationOutput = ptrDevImageA;

		heatTransfertEcrasement<<<dg,db>>>(ptrDevImageHeater,iterationOutput,iterationOutput, w, h);
		Device::synchronize();
	}

	//show on screen what is going on
	heatTransfertConvertData<<<dg,db>>>(this->calibreurCuda, iterationOutput,ptrDevPixels, w, h);
	//Device::synchronize();
}

/**
 * Override
 */
float HeatTransfert::getT() {
	return 0.0;
}

/**
 * Override
 */
int HeatTransfert::getW() {
	return w;
}

/**
 * Override
 */
int HeatTransfert::getH() {
	return h;
}

/**
 * Override
 */
string HeatTransfert::getTitle() {
	return title;
}
