#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "HeatTransfert.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void heatTransfert(float* ptrDevImageAInput, float* ptrDevImageBOutput,int w, int h);
extern __global__ void heatEcrasement(float* ptrDevImageInput,float* ptrDevImageHeaters ,float* ptrDevImageOutput,int w,int h);
extern __global__ void heatToScreenImageHSB(float* ptrDevImageInput, uchar4* ptrDevImageGL, int w, int h);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

HeatTransfert::HeatTransfert(int w, int h,float dt,float k)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;
    this->k=k;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : HeatTransfert CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w== h);

    cout << endl<<"[CBI] HeatTransfert dt =" << dt;
    }

HeatTransfert::~HeatTransfert()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void HeatTransfert::animationStep()
    {
    t+=dt;
    }

/**
 * Override
 */
void HeatTransfert::runGPU(uchar4* ptrDevPixels)
    {

    heatTransfert<<<dg,db>>>(float* ptrDevImageAInput, float* ptrDevImageBOutput,int w, int h);
   heatEcrasement<<<dg,db>>>(float* ptrDevImageInput,float* ptrDevImageHeaters ,float* ptrDevImageOutput,int w,int h);
     heatToScreenImageHSB<<<dg,db>>>(float* ptrDevImageInput, uchar4* ptrDevImageGL, int w, int h);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float HeatTransfert::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int HeatTransfert::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int HeatTransfert::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string HeatTransfert::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
 
 //Init GPU Memory and copy data 
 void initGPUMemory(){
	 HANDLE_ERROR(hipMalloc(&ptrImageDeviceA,SIZE_BAND));
    HANDLE_ERROR(hipMemset(ptrImageDeviceA,0,SIZE_BAND));
	 HANDLE_ERROR(hipMalloc(&ptrImageDeviceB,SIZE_BAND));
    HANDLE_ERROR(hipMemset(ptrImageDeviceB,0,SIZE_BAND));
	 HANDLE_ERROR(hipMalloc(&prtImageHeats,SIZE_BAND));
    HANDLE_ERROR(hipMemset(prtImageHeats,0,SIZE_BAND));
 }
 
 void createDataForGPU(){
	uchar4* ptrImageDeviceA;
	uchar4* ptrImageDeviceB;
	uchar4* prtImageHeats;
 }
 
 void initGPUFirstStep(int h,int w,float k){
	heatEcrasement<<<dg,db>>>(ptrImageDeviceA,ptrDevImageHeaters ,ptrImageDeviceB,int w,int h);
	heatTransfert<<<dg,db>>>(ptrImageDeviceA,ptrImageDeviceB,int w, int h);
  
 }
 
 void freeGPUMemory(){
	HANDLE_ERROR(hipFree(ptrImageDeviceA));
	HANDLE_ERROR(hipFree(ptrImageDeviceB));
	HANDLE_ERROR(hipFree(prtImageHeats));
 }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

