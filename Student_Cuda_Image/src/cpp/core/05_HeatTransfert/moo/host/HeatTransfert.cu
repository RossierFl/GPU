#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "HeatTransfert.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void heatTransfert(float* ptrDevImageAInput, float* ptrDevImageBOutput,int w, int h);
extern __global__ void heatEcrasement(float* ptrDevImageInput,float* ptrDevImageHeaters ,float* ptrDevImageOutput,int w,int h);
extern __global__ void heatToScreenImageHSB(float* ptrDevImageInput, uchar4* ptrDevImageGL, int w, int h);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

HeatTransfert::HeatTransfert(int w, int h, float dt, float k)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt = dt;
    this->k = k;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;

    //Outputs
    this->title = "[API Image Cuda] : HeatTransfert CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);

    cout << endl << "[CBI] HeatTransfert dt =" << dt;
    }

HeatTransfert::~HeatTransfert()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void HeatTransfert::animationStep()
    {
    t += dt;
    }

/**
 * Override
 */
void HeatTransfert::runGPU(uchar4* ptrDevPixels)
    {

heatTransfert<<<dg,db>>>(float* ptrDevImageAInput, float* ptrDevImageBOutput,int w, int h);
heatEcrasement<<<dg,db>>>(float* ptrDevImageInput,float* ptrDevImageHeaters ,float* ptrDevImageOutput,int w,int h);
heatToScreenImageHSB<<<dg,db>>>(float* ptrDevImageInput, uchar4* ptrDevImageGL, int w, int h);
}

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float HeatTransfert::getT(void)
{
return t;
}

/**
 * Override
 */
int HeatTransfert::getW(void)
{
return w;
}

/**
 * Override
 */
int HeatTransfert::getH(void)
{
return h;
}

/**
 * Override
 */
string HeatTransfert::getTitle(void)
{
return title;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

    //Init GPU Memory and copy data
void initGPUMemory()
{
HANDLE_ERROR(hipMalloc(&ptrImageDeviceA, SIZE_BAND));
HANDLE_ERROR(hipMemset(ptrImageDeviceA, 0, SIZE_BAND));
HANDLE_ERROR(hipMalloc(&ptrImageDeviceB, SIZE_BAND));
HANDLE_ERROR(hipMemset(ptrImageDeviceB, 0, SIZE_BAND));
HANDLE_ERROR(hipMalloc(&prtImageHeats, SIZE_BAND));
HANDLE_ERROR(hipMemset(prtImageHeats, 0, SIZE_BAND));
}

void createDataForGPU(int h, int w)
{
float tableHostHeat[h][w];
for (int i = 0; i < h; i++)
    {
    for (int j = 0; j < w; j++)
	{
	    if(j>179 && j<195 && i>179 && i<195){		//Gros carré froids
		tableHostHeat[i][j]=0.2f;
	    }else if(j>605 && j<621 && i>179 && i<195){
		tableHostHeat[i][j]=0.2f;
	    }else if(j>179 && j<195  && i>605 && i<121){
		tableHostHeat[i][j]=0.2f;
	    }else if(j>605 && j<621  && i>605 && i<121){
		tableHostHeat[i][j]=0.2f;
	    }else if(j>295 && j<400  && i>295 && i<400){ // Petits carrés froids
		tableHostHeat[i][j]=0.2f;
	    }else if(j>400 && j<295  && i>400 && i<295){
		tableHostHeat[i][j]=0.2f;
	    }else if(j>505 && j<400  && i>505 && i<400){
		tableHostHeat[i][j]=0.2f;
	    }else if(j>400 && j<505  && i>400 && i<505){
		tableHostHeat[i][j]=0.2f;
	    }else if(j>300 && j<500  && i>300 && i<500){ //Gros carré chaud
		tableHostHeat[i][j]=1.0f;
	    } else {
		tableHostHeat[i][j]=0;
	    }
	}
    }
}

void initGPUFirstStep(int h, int w, float k)
{
heatEcrasement<<<dg,db>>>(ptrImageDeviceB,ptrDevImageHeaters ,ptrImageDeviceB, w, h);
heatTransfert<<<dg,db>>>(ptrImageDeviceA,ptrImageDeviceB, w, h);
heatEcrasement<<<dg,db>>>(ptrImageDeviceB,ptrDevImageHeaters ,ptrImageDeviceA, w, h);

}

void freeGPUMemory()
{
HANDLE_ERROR(hipFree(ptrImageDeviceA));
HANDLE_ERROR(hipFree(ptrImageDeviceB));
HANDLE_ERROR(hipFree(prtImageHeats));
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
