#include "hip/hip_runtime.h"
#include <assert.h>

#include "HeatTransfert.h"
#include "Device.h"
#include "MathTools.h"
#include <stdio.h>

#define NB_ITERATION_AVEUGLE 2

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void diffusion(float* ptrDevImageA,float* ptrDevImageB, int w, int h, float k, bool imgAIsInput);
extern __global__ void ecrasement(float* ptrDevInputHeater, float* ptrDevInputImage,float* ptrDevOutput, int w, int h);
extern __global__ void toScreenImageHSB(CalibreurCuda calibreurCuda,float* ptrDevInput, uchar4* ptrDevImage, int w, int h);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

HeatTransfert::HeatTransfert(int w, int h, float k):calibreurCuda(0.0,1.0,0.7,0.0)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->k = k;
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser


    //Outputs
    this->title = "[API Image] : HeatTransfert CUDA";

    // Check:
    Device::assertDim(dg, db);
    assert(w == h);
    size_t sizeImages = w*h*sizeof(float);
    //init
    //imageInit all to zero
    HANDLE_ERROR(hipMalloc((void**) &ptrDevImageInit, sizeImages)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevImageInit,0,sizeImages)) ;

    HANDLE_ERROR(hipMalloc((void**) &ptrDevImageA, sizeImages)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevImageA,0,sizeImages)) ;

    HANDLE_ERROR(hipMalloc((void**) &ptrDevImageB, sizeImages)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevImageB,0,sizeImages)) ;

    //imageHeater all zero with some heater or "cooler"
    HANDLE_ERROR(hipMalloc((void**) &ptrDevImageHeater, sizeImages)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevImageHeater,0,sizeImages)) ;

    float* ptrHostImageHeater = new float[w*h];
    memset(ptrHostImageHeater,0,sizeImages);

    // init
//    for(int i=0;i<=h;i++)
//	{
//	for(int j=0;j<=w;j++)
//	    {
//	    ptrHostImageHeater[i*h+j]=0.0;
//	    }
//	}
    //single cooler point
    ptrHostImageHeater[295*h+400] = 0.2;
    ptrHostImageHeater[400*h+295] = 0.2;
    ptrHostImageHeater[505*h+400] = 0.2;
    ptrHostImageHeater[400*h+505] = 0.2;

    //main heater
    for(int i=300;i<=500;i++)
	{
	for(int j=300;j<=500;j++)
	    {
	    ptrHostImageHeater[i*h+j] = 1.0;
	    }
	}

    //north cooler
    for(int i=179;i<=195;i++)
	{
	for(int j=179;j<=195;j++)
	    {
	    ptrHostImageHeater[i*h+j] = 0.2;
	    }
	for(int j=605;j<=621;j++)
	    {
	    ptrHostImageHeater[i*h+j] = 0.2;
	    }
	}

    //south cooler
    for(int i=605;i<=621;i++)
	{
	for(int j=179;j<=195;j++)
	    {
	    ptrHostImageHeater[i*h+j] = 0.2;
	    }
	for(int j=605;j<=621;j++)
	    {
	    ptrHostImageHeater[i*h+j] = 0.2;
	    }
	}

    // host --> device
    HANDLE_ERROR(hipMemcpy(ptrDevImageHeater,  ptrHostImageHeater, sizeImages, hipMemcpyHostToDevice));// Host -> Device
    delete [] ptrHostImageHeater;

    ecrasement<<<dg,db>>>(ptrDevImageHeater,ptrDevImageA,ptrDevImageA, w, h);

    Device::synchronize();
    Device::checkKernelError("heatTransfertEcrasement");

    }

HeatTransfert::~HeatTransfert()
    {
    HANDLE_ERROR(hipFree(ptrDevImageHeater));
    HANDLE_ERROR(hipFree(ptrDevImageInit));
    HANDLE_ERROR(hipFree(ptrDevImageA));
    HANDLE_ERROR(hipFree(ptrDevImageB));
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void HeatTransfert::animationStep()
    {
    }

/**
 * Override
 */
void HeatTransfert::runGPU(uchar4* ptrDevPixels)
    {

    float* iterationOutput;
    for(int i=0;i<=NB_ITERATION_AVEUGLE;i++)
	{
	diffusion<<<dg,db>>>(ptrDevImageA,ptrDevImageB, w, h, k, (i+1)%2);
	Device::synchronize();


	if((i+1)%2==1)
	    iterationOutput=ptrDevImageB;
	else
	    iterationOutput=ptrDevImageA;

	ecrasement<<<dg,db>>>(ptrDevImageHeater,iterationOutput,iterationOutput, w, h);
	Device::synchronize();
	}

    toScreenImageHSB<<<dg,db>>>(this->calibreurCuda, iterationOutput,ptrDevPixels, w, h);
    }


/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float HeatTransfert::getT(void)
    {
    return 0.0;
    }

/**
 * Override
 */
int HeatTransfert::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int HeatTransfert::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string HeatTransfert::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

