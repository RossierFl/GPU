#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "HeatTransfert.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void heatTransfert(uchar4* ptrDevPixels,int w, int h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

HeatTransfert::HeatTransfert(int w, int h,float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->dt=dt;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : HeatTransfert CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w== h);

    cout << endl<<"[CBI] HeatTransfert dt =" << dt;
    }

HeatTransfert::~HeatTransfert()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void HeatTransfert::animationStep()
    {
    t+=dt;
    }

/**
 * Override
 */
void HeatTransfert::runGPU(uchar4* ptrDevPixels)
    {
    heatTransfert<<<dg,db>>>(ptrDevPixels,w,h,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float HeatTransfert::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int HeatTransfert::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int HeatTransfert::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string HeatTransfert::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

