#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJuliaMultiGPU.h"
#include "Device.h"
#include "MathTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void mandelbrotJuliaCuMltiGPU(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath, int n,float t,bool isJulia,float cX,float cY,float offset);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJuliaMultiGPU::MandelbrotJuliaMultiGPU(int w, int h, float dt, int n,float xMin,float xMax,float yMin,float yMax,bool isJulia,float cX=0,float cY=0) :
	variateurAnimation(IntervalF(20, 200),dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;
	this->cX=cX;
    this->cY=cY;
    this->isJulia=isJulia;
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(xMin,yMin,xMax,yMax);

    //Outputs
    this->title = "[API Image Fonctionelle] : MandelbrotJuliaMultiGPU zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);

    const size_t SIZE_BAND=h/6*w*sizeof(uchar4);
    std::cout<<"--- Inits GPUS ----";
    hipSetDevice(1);
    std::cout<<"Init1 ";
    HANDLE_ERROR(hipMalloc(&ptrDevPixels1,SIZE_BAND));
    HANDLE_ERROR(hipMemset(ptrDevPixels1,0,SIZE_BAND));
    hipSetDevice(2);
    std::cout<<"Init2 ";
    HANDLE_ERROR(hipMalloc(&ptrDevPixels2,SIZE_BAND));
    HANDLE_ERROR(hipMemset(ptrDevPixels2,0,SIZE_BAND));
    hipSetDevice(3);
    std::cout<<"Init2 ";
    HANDLE_ERROR(hipMalloc(&ptrDevPixels3,SIZE_BAND));
    HANDLE_ERROR(hipMemset(ptrDevPixels3,0,SIZE_BAND));
    hipSetDevice(4);
    std::cout<<"Init3 ";
    HANDLE_ERROR(hipMalloc(&ptrDevPixels4,SIZE_BAND));
    HANDLE_ERROR(hipMemset(ptrDevPixels4,0,SIZE_BAND));
    hipSetDevice(5);
    std::cout<<"Init4 ";
    HANDLE_ERROR(hipMalloc(&ptrDevPixels5,SIZE_BAND));
    HANDLE_ERROR(hipMemset(ptrDevPixels5,0,SIZE_BAND));

    hipSetDevice(0);


    assert(w == h);
    }

MandelbrotJuliaMultiGPU::~MandelbrotJuliaMultiGPU()
    {
    hipSetDevice(1);
    HANDLE_ERROR(hipFree(ptrDevPixels1));
    hipSetDevice(2);
    HANDLE_ERROR(hipFree(ptrDevPixels2));
    hipSetDevice(3);
    HANDLE_ERROR(hipFree(ptrDevPixels3));
    hipSetDevice(4);
    HANDLE_ERROR(hipFree(ptrDevPixels4));
    hipSetDevice(5);
    HANDLE_ERROR(hipFree(ptrDevPixels5));
    hipSetDevice(0);

   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJuliaMultiGPU::animationStep()
    {


    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void MandelbrotJuliaMultiGPU::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
   // hipSetDevice(1);
   // hipSetDevice(0);
   // double x0 =domaineMath.x0;
   // double y0 =domaineMath.y0;
   // double x1 =domaineMath.x1;
    //double y1 =domaineMath.y1;
    //double dx = (x1-x0)/6;
    //double dy = (y1-y0)/6;
    const size_t SIZE_RESULT = h/6*w*(size_t)sizeof(uchar4);


    //DomaineMath dm(x0,y0,x0+dx,y0+dy);
    //INIT CUDA MEMORY
    //std::cout<<"Steps"<<std::endl;
    hipSetDevice(0);
    //std::cout<<"Steps1"<<std::endl;
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels,w,h/6,domaineMath,n,t, isJulia, cX, cY,0);
    hipSetDevice(1);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels1,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6);
    HANDLE_ERROR(hipMemcpy(ptrDevPixels+(h/6*w),ptrDevPixels1,SIZE_RESULT,hipMemcpyDeviceToDevice));//barriere implicite de sync

    //std::cout<<"Steps"<<std::endl;


   hipSetDevice(2);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels2,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*2);
    HANDLE_ERROR(hipMemcpy(ptrDevPixels+(h/6*w*2),ptrDevPixels2,SIZE_RESULT,hipMemcpyDeviceToDevice));//barriere implicite de sync
    hipSetDevice(3);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels3,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*3);
    HANDLE_ERROR(hipMemcpy(ptrDevPixels+(h/6*w*3),ptrDevPixels3,SIZE_RESULT,hipMemcpyDeviceToDevice));//barriere implicite de sync
    hipSetDevice(4);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels4,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*4);
    HANDLE_ERROR(hipMemcpy(ptrDevPixels+(h/6*w*4),ptrDevPixels4,SIZE_RESULT,hipMemcpyDeviceToDevice));//barriere implicite de sync
    hipSetDevice(5);
       mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels5,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*5);
       HANDLE_ERROR(hipMemcpy(ptrDevPixels+(h/6*w*5),ptrDevPixels5,SIZE_RESULT,hipMemcpyDeviceToDevice));//barriere implicite de sync
    hipSetDevice(0);

    }

/*--------------*\
 |*	get	 *|,
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJuliaMultiGPU::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float MandelbrotJuliaMultiGPU::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJuliaMultiGPU::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJuliaMultiGPU::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJuliaMultiGPU::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

