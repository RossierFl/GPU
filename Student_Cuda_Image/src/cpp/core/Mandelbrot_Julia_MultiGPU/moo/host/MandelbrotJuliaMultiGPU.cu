#include "hip/hip_runtime.h"
#include <assert.h>

#include "MandelbrotJuliaMultiGPU.h"
#include "Device.h"
#include "MathTools.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void mandelbrotJuliaCuMltiGPU(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath, int n,float t,bool isJulia,float cX,float cY,float offset);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJuliaMultiGPU::MandelbrotJuliaMultiGPU(int w, int h, float dt, int n,float xMin,float xMax,float yMin,float yMax,bool isJulia,float cX=0,float cY=0) :
	variateurAnimation(IntervalF(20, 200),dt)
    {
    // Inputs
    this->w = w;
    this->h = h;
    this->n = n;
	this->cX=cX;
    this->cY=cY;
    this->isJulia=isJulia;
    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit=new DomaineMath(xMin,yMin,xMax,yMax);

    //Outputs
    this->title = "[API Image Fonctionelle] : MandelbrotJuliaMultiGPU zoomable CUDA";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    assert(w == h);
    }

MandelbrotJuliaMultiGPU::~MandelbrotJuliaMultiGPU()
    {
   delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void MandelbrotJuliaMultiGPU::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 */
void MandelbrotJuliaMultiGPU::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
   // hipSetDevice(1);
   // hipSetDevice(0);
   // double x0 =domaineMath.x0;
   // double y0 =domaineMath.y0;
   // double x1 =domaineMath.x1;
    //double y1 =domaineMath.y1;
    //double dx = (x1-x0)/6;
    //double dy = (y1-y0)/6;


    //DomaineMath dm(x0,y0,x0+dx,y0+dy);

    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels,w,h/6,domaineMath,n,t, isJulia, cX, cY,0);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels,w,h/6*2,domaineMath,n,t, isJulia, cX, cY,h/6);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*2);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*3);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*4);
    mandelbrotJuliaCuMltiGPU<<<dg,db>>>(ptrDevPixels,w,h/6,domaineMath,n,t, isJulia, cX, cY,h/6*5);
    }

/*--------------*\
 |*	get	 *|,
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJuliaMultiGPU::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float MandelbrotJuliaMultiGPU::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int MandelbrotJuliaMultiGPU::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int MandelbrotJuliaMultiGPU::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string MandelbrotJuliaMultiGPU::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

