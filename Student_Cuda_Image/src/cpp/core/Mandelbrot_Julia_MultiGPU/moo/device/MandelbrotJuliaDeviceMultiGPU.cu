#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "MandelbrotJuliaMultiGPU.h"
#include "MandelbrotJuliaMathMultiGPU.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrotJuliaCuMltiGPU(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath, int n,float t,bool isJulia,float cX,float cY,float offset);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void mandelbrotJuliaCuMltiGPU(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n, float t,bool isJulia,float cX,float cY,float offset)
    {
    MandelbrotJuliaMultiGPUMath mandelbrotJuliaMath = MandelbrotJuliaMultiGPUMath(n,isJulia,cX,cY);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)
	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI+offset, pixelJ, &x, &y); //  (i,j) -> (x,y)
	
	mandelbrotJuliaMath.colorXY(&color,x, y,domaineMath,t); // update color

	int newS = IndiceTools::toS(w,pixelI+offset,pixelJ);
	ptrDevPixels[newS] = color;

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

