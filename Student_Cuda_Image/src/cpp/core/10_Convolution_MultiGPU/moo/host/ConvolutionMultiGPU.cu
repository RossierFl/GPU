#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <assert.h>

#include "ConvolutionMultiGPU.h"
//#include "ConvolutionMathMultiGPU.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "Device.h"
#include "MathTools.h"
#include "Chronos.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

//extern __global__ void convolutionKernelMultiGPU(texture<uchar4,2> tex, uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t);
extern __global__ void colorToGreyMultiGPU(uchar4* ptrDevPixels, int w, int h);
extern __global__ void findMinMaxMultiGPU(uchar4* ptrDevPixels, uchar* ptrDevResult,int w, int h);
extern __global__ void affineTransformMultiGPU(uchar4* ptrDevPixels, float a, float b, int w, int h, int offset);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

texture<uchar4, 2> textureRef;

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__device__
void colorIJMultiGPU(uchar4* ptrColor, float* ptrDeviceNoyau, int k, int i, int j, int s)
    {

    float sum = 0.0f;
    int ss = (int)(k*((float)k/2.0f));
    int k2 = k/2;
    for(int v = 1;v<=k2;v++)
	{
	for(int u = 1;u<=k2;u++)
	    {
	    // bas droite
	    sum+=ptrDeviceNoyau[(ss+v*k)+u]*tex2D(textureRef,j+u,i+v).x;
	    // haut droite
	    sum+=ptrDeviceNoyau[(ss-v*k)+u]*tex2D(textureRef,j+u,i-v).x;
	    // bas gauche
	    sum+=ptrDeviceNoyau[(ss+v*k)-u]*tex2D(textureRef,j-u,i+v).x;
	    // haut gauche
	    sum+=ptrDeviceNoyau[(ss-v*k)-u]*tex2D(textureRef,j-u,i-v).x;
	    }
	// bras east
	sum+=ptrDeviceNoyau[ss+v]*tex2D(textureRef,j,i+v).x;
	// bras west
	sum+=ptrDeviceNoyau[ss-v]*tex2D(textureRef,j,i-v).x;
	// bras south
	sum+=ptrDeviceNoyau[ss+v*k]*tex2D(textureRef,j+v,i).x;
	// bras north
	sum+=ptrDeviceNoyau[ss-v*k]*tex2D(textureRef,j-v,i).x;
	}
    // centre
    sum+=ptrDeviceNoyau[ss]*tex2D(textureRef,j,i).x;
    ptrColor->x = sum;
    ptrColor->y = sum;
    ptrColor->z = sum;

    ptrColor->w = 255; // opaque
    }

__global__ void convolutionKernelMultiGPU(uchar4* ptrDevPixels, float* ptrDeviceNoyau, int k, int w, int h, float t)
    {
    extern __shared__ float convSM[];// 1 instance per block !

    //ConvolutionMathMultiGPU convMath = ConvolutionMathMultiGPU(w, h);

    const int TID = Indice2D::tid();
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID_LOCAL;
    if(s < (k*k))
	convSM[s] = ptrDeviceNoyau[s];
    __syncthreads();

    s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)
	colorIJMultiGPU(&color,convSM,k,pixelI, pixelJ, s); 	// update color
	ptrDevPixels[s] = color;
	s += NB_THREAD;
	}
    }

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

ConvolutionMultiGPU::ConvolutionMultiGPU()
    {
    ConvolutionMultiGPU(1);
    }

ConvolutionMultiGPU::ConvolutionMultiGPU(const int NB_GPU)
    {
    // Inputs
    this->w = 640;
    this->h = 360;
    this->k = 9;
    this->NB_GPUS = NB_GPU;

    // Tools
    this->dg = dim3(32, 1, 1); // disons a optimiser
    this->db = dim3(256, 1, 1); // disons a optimiser
    this->t=0;

    //Outputs
    this->title="[API Image Cuda] : Convolution CUDA MultiGPU";
    //this->videoPath="/media/Data/Video/nasaFHD_short.avi";
    //this->videoTitle="NASA FHD SHORT";

    this->videoPath="/media/Data/Video/neilPryde.avi";
    this->videoTitle="neilPryde";

    size_t sizeOctets=w*h*sizeof(uchar4);
    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostMemory), sizeOctets,hipHostMallocDefault ) );

    this->captureur = new CaptureVideo(videoPath,videoTitle,this->ptrHostMemory);

    const int N = k*k;
    sizeConvSM = N*sizeof(float);

    HANDLE_ERROR( hipHostAlloc ((void**) &(this->ptrHostNoyau), N*sizeof(float),hipHostMallocDefault ) );

    fillDetourage(ptrHostNoyau);

    sizeSM = 2*db.x*sizeof(uchar);
    sizeResult=2*dg.x*sizeof(uchar);
    ptrHostResult = (uchar**)malloc(NB_GPU*sizeof(uchar*));

    ptrDeviceNoyau = (float**)malloc(NB_GPU*sizeof(float*));
    ptrDevResult = (uchar**)malloc(NB_GPU*sizeof(uchar*));
    for(int i = 0;i<NB_GPU;i++)
	{
	ptrHostResult[i] = (uchar*) malloc(sizeResult);
	HANDLE_ERROR(hipSetDevice(i));
	HANDLE_ERROR(hipMalloc((void**)&ptrDeviceNoyau[i],N*sizeof(float)));
	// first contains min of each block, then max of each block
	HANDLE_ERROR(hipMalloc((void**)&ptrDevResult[i], sizeResult));
	HANDLE_ERROR(hipMemcpy(ptrDeviceNoyau[i],ptrHostNoyau,N*sizeof(float),hipMemcpyHostToDevice));
	}
    HANDLE_ERROR(hipSetDevice(0));

    textureRef.filterMode= hipFilterModePoint;
    textureRef.normalized=false; //ou true coordonnée texture (i,j)
    textureRef.addressMode[0] = hipAddressModeClamp;
    textureRef.addressMode[1] = hipAddressModeClamp;

    pitch = w * sizeof(uchar4); //taille en octets d'une ligne
    channelDesc = hipCreateChannelDesc<uchar4>();

    ptrDevMultiGPUImage = (uchar4**)malloc(NB_GPU*sizeof(uchar4*));
    stream = new hipStream_t[NB_GPU];
    for(int i = 0;i<NB_GPU;i++)
	{
	HANDLE_ERROR(hipSetDevice(i));
	hipStreamCreate(&stream[i]);
	}
    HANDLE_ERROR(hipSetDevice(0));
    //Device::p2pEnableALL();
    //cout << endl<<"[CBI] Convolution dt =" << dt << endl;
    }

ConvolutionMultiGPU::~ConvolutionMultiGPU()
    {
    free(ptrHostResult);
    hipFree(ptrDevResult);
    for(int i = 0;i<NB_GPUS;i++)
	{
	hipSetDevice(i);
	hipStreamDestroy(stream[i]);
	hipDeviceReset();
	}
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Noyau detourage 9x9
 */
void ConvolutionMultiGPU::fillDetourage(float* ptrNoyau)
    {
    const int N = 9*9;
    // Tab auto temporaire
    float tab[N] =
	    {
		    0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.6063, 1.2765, 0.4061, -6.4829, -12.7462, -6.4829, 0.4061, 1.2765, 0.6063, 0.5366, 1.1794, 1.0033, -2.8306, -6.4829, -2.8306, 1.0033, 1.1794, 0.5366, 0.3705, 0.8646, 1.3475, 1.0033, 0.4061, 1.0033, 1.3475, 0.8646, 0.3705, 0.1987, 0.4746, 0.8646, 1.1794, 1.2765, 1.1794, 0.8646, 0.4746, 0.1987, 0.0828, 0.1987, 0.3705, 0.5366, 0.6063, 0.5366, 0.3705, 0.1987, 0.0828
	    };

    for (int i = 1; i <= N; i++)
	{
	ptrNoyau[i - 1] = tab[i - 1] / 100.0f;
	}
    }

/**
 * Override
 * Call periodicly by the API
 */
void ConvolutionMultiGPU::animationStep()
    {
    //t+=dt;
    }

/**
 * Override
 */
void ConvolutionMultiGPU::runGPU(uchar4* ptrDevPixels)
    {
    Chronos chrono;
    HANDLE_ERROR(hipSetDevice(0));
    Mat matImage = captureur->capturer();
    uchar4* image = CaptureVideo::castToUChar4(&matImage);

    HANDLE_ERROR(hipMemcpy(ptrDevPixels,image,(w*h)*sizeof(ptrDevPixels[0]),hipMemcpyHostToDevice));

    //printf("Image to Grey\n");
    //colorToGreyMultiGPU<<<dg,db>>>(ptrDevPixels,w,h);
    //HANDLE_ERROR(hipDeviceSynchronize());

    //printf("Convolution on grey image\n");
    //printf("Binding texture\n");
    colorToGreyMultiGPU<<<dg,db,0,stream[0]>>>(ptrDevPixels,w,h/NB_GPUS);
    hipBindTexture2D(NULL, textureRef,ptrDevPixels,channelDesc,w,(h/NB_GPUS),pitch);
    convolutionKernelMultiGPU<<<dg,db,sizeConvSM,stream[0]>>>(ptrDevPixels,ptrDeviceNoyau[0],k,w,h/NB_GPUS,t);
    findMinMaxMultiGPU<<<dg,db,sizeSM,stream[0]>>>(ptrDevPixels,ptrDevResult[0],w,h/NB_GPUS);
    HANDLE_ERROR(hipMemcpyAsync(ptrHostResult[0], ptrDevResult[0], sizeResult, hipMemcpyDeviceToHost,stream[0]));
    for(int i = 1;i<NB_GPUS;i++)
	{
	HANDLE_ERROR(hipSetDevice(i));
	if(i==NB_GPUS-1)
	    {
	    // portion of image + offset for convolution
	    hipMalloc((void**)&ptrDevMultiGPUImage[i-1],((w*h/NB_GPUS)+(k*w/2))*sizeof(uchar4));
	    hipMemcpyAsync(ptrDevMultiGPUImage[i-1],ptrDevPixels+i*(w*h/NB_GPUS)-(k*w/2),((w*h/NB_GPUS)+(k*w/2))*sizeof(uchar4),
		    hipMemcpyDeviceToDevice,stream[i]);
	    colorToGreyMultiGPU<<<dg,db,0,stream[i]>>>(ptrDevMultiGPUImage[i-1],w,((h/NB_GPUS)+(k/2)));
	    //hipStreamSynchronize(stream[i]);
	    HANDLE_ERROR(hipPeekAtLastError());
	    hipBindTexture2D(NULL, textureRef,ptrDevMultiGPUImage[i-1],channelDesc,w,((h/NB_GPUS)+(k/2)),pitch);
	    HANDLE_ERROR(hipPeekAtLastError());
	    convolutionKernelMultiGPU<<<dg,db,sizeConvSM,stream[i]>>>(ptrDevMultiGPUImage[i-1],ptrDeviceNoyau[i],k,w,((h/NB_GPUS)+(k/2)),t);
	    HANDLE_ERROR(hipPeekAtLastError());
	    findMinMaxMultiGPU<<<dg,db,sizeSM,stream[i]>>>(ptrDevMultiGPUImage[i-1],ptrDevResult[i],w,((h/NB_GPUS)+(k/2)));
	    HANDLE_ERROR(hipMemcpyAsync(ptrHostResult[i], ptrDevResult[i], sizeResult, hipMemcpyDeviceToHost,stream[i]));
	    }
	else
	    {
	    // portion of image + offset for convolution
	    hipMalloc((void**)&ptrDevMultiGPUImage[i-1],((w*h/NB_GPUS)+2*(k*w/2))*sizeof(uchar4));
	    hipMemcpyAsync(ptrDevMultiGPUImage[i-1],ptrDevPixels+i*(w*h/NB_GPUS)-(k*w/2),((w*h/NB_GPUS)+2*(k*w/2))*sizeof(uchar4),
		    hipMemcpyDeviceToDevice,stream[i]);
	    //hipStreamSynchronize(stream[i]);
	    colorToGreyMultiGPU<<<dg,db,0,stream[i]>>>(ptrDevMultiGPUImage[i-1],w,((h/NB_GPUS)+2*(k/2)));
	    HANDLE_ERROR(hipPeekAtLastError());
	    hipBindTexture2D(NULL, textureRef,ptrDevMultiGPUImage[i-1],channelDesc,w,((h/NB_GPUS)+2*(k/2)),pitch);
	    HANDLE_ERROR(hipPeekAtLastError());
	    convolutionKernelMultiGPU<<<dg,db,sizeConvSM,stream[i]>>>(ptrDevMultiGPUImage[i-1],ptrDeviceNoyau[i],k,w,((h/NB_GPUS)+2*(k/2)),t);
	    HANDLE_ERROR(hipPeekAtLastError());
	    findMinMaxMultiGPU<<<dg,db,sizeSM,stream[i]>>>(ptrDevMultiGPUImage[i-1],ptrDevResult[i],w,((h/NB_GPUS)+2*(k/2)));
	    HANDLE_ERROR(hipMemcpyAsync(ptrHostResult[i], ptrDevResult[i], sizeResult, hipMemcpyDeviceToHost,stream[i]));
	    }

	}
    uchar max = 0;
    uchar min = 255;
    float a = 0;
    float b = 0;
    for(int i = 0;i<NB_GPUS;i++)
	{
	HANDLE_ERROR(hipSetDevice(i));
	HANDLE_ERROR(hipStreamSynchronize(stream[i]));
	}
    for(int i = 0;i<NB_GPUS;i++)
	{
	for(int j = 0;j<dg.x;j++)
	    {
	    uchar crt = ptrHostResult[i][j];
	    if(crt < min)
		min = crt;
	    crt = ptrHostResult[i][j+dg.x];
	    if(crt > max)
		max = crt;
	    }
	// affine transformation
	a = 255.0f/(float)(max-min);
	b = 0;
	if(min != 0)
	    b = 255.0f/((-max/(float)min)+1.0f);
	}
    HANDLE_ERROR(hipSetDevice(0));
    affineTransformMultiGPU<<<dg,db,0,stream[0]>>>(ptrDevPixels, a, b, w, h/NB_GPUS,0);
    for(int i = 1;i<NB_GPUS;i++)
	{
	HANDLE_ERROR(hipSetDevice(i));
	if(i==NB_GPUS-1)
	    {
	    affineTransformMultiGPU<<<dg,db,0,stream[i]>>>(ptrDevMultiGPUImage[i-1], a, b, w, ((h/NB_GPUS)+(k/2)),0);
	    }
	else if(i > 0)
	    {
	    affineTransformMultiGPU<<<dg,db,0,stream[i]>>>(ptrDevMultiGPUImage[i-1], a, b, w, ((h/NB_GPUS)+2*(k/2)),0);
	    }
	}
    for(int i = 0;i<NB_GPUS;i++)
	{
	HANDLE_ERROR(hipSetDevice(i));
	HANDLE_ERROR(hipStreamSynchronize(stream[i]));
	if(i>0)
	    hipMemcpyAsync(ptrDevPixels+i*(w*h/NB_GPUS),ptrDevMultiGPUImage[i-1]+(k*w/2),((w*h/NB_GPUS))*sizeof(uchar4),hipMemcpyDeviceToDevice,
		    stream[i]);
	}
    for(int i = 0;i<NB_GPUS;i++)
	{
	HANDLE_ERROR(hipSetDevice(i));
	HANDLE_ERROR(hipStreamSynchronize(stream[i]));
	}
    //HANDLE_ERROR(hipStreamSynchronize(stream[0]));
    HANDLE_ERROR(hipSetDevice(0));

    //printf("Rescaling colors\n");
    //    findMinMaxMultiGPU<<<dg,db,sizeSM>>>(ptrDevPixels,ptrDevResult,w,h);
    //    HANDLE_ERROR(hipMemcpy(ptrHostResult, ptrDevResult, sizeResult, hipMemcpyDeviceToHost));
    //printf("min: %d, max: %d\n",min,max);
    chrono.stop();
    cout << "ElapseTime:  " << chrono.getDeltaTime()<< " (s)" << endl;
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float ConvolutionMultiGPU::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int ConvolutionMultiGPU::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int ConvolutionMultiGPU::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string ConvolutionMultiGPU::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

