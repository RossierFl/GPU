#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"

#include "Vague0Math.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __global__ void vague0(uchar4* ptrDevPixels,int w, int h,float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

void launchKernelVague0(uchar4* ptrDevPixels, int w, int h, float t)
    {
    dim3 dg = dim3(8, 8, 1); // disons, a optimiser !!
    dim3 db = dim3(16, 16, 1); // disons, a optimiser !!

    //Device::print(dg, db);
     Device::checkDimError(dg,db);

     vague0<<<dg,db>>>(ptrDevPixels,w,h,t);
    Device::checkKernelError("vague0");
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void vague0(uchar4* ptrDevPixels, int w, int h, float t)
    {
    Vague0Math vague0Math = Vague0Math(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH=w*h;

    uchar4 color;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	vague0Math.colorIJ(&color,pixelI, pixelJ, t); 	// update color
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }


/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

