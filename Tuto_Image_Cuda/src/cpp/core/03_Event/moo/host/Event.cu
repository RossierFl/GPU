#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Event.h"
#include "Device.h"
#include "MathTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void event(uchar4* ptrDevPixels,int w, int h,DomaineMath domaineMath,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Event::Event(int w, int h, float dt)
    {
    // Inputs
    this->w = w;
    this->h = h;

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->t = 0;
    ptrDomaineMathInit = new DomaineMath(0, 0, 10, 10);

    //Outputs
    this->title = "[API Image Cuda] : Event Window, try to clik on the window, and look console";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);

    }

Event::~Event()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 */
void Event::animationStep()
    {
    // rien
    }

/**
 * Override
 */
void Event::runGPU(uchar4* ptrDevPixels, const DomaineMath& domaineMath)
    {
    event<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,t);
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/


/**
 * Override
 */
DomaineMath* Event::getDomaineMathInit(void)
    {
    return ptrDomaineMathInit;
    }

/**
 * Override
 */
float Event::getT(void)
    {
    return t;
    }

/**
 * Override
 */
int Event::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Event::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string Event::getTitle(void)
    {
    return title;
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

