#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdio>

#include "cuda_utils.h"
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isSaucisson_Ok(int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ static float computeSaucisson(int n);
__global__ static void computeSaucissonGPU(int n,float* ptrDevResult, int length);
__device__ void reduceIntraThreadSaucisson(float* tabSM, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isSaucisson_Ok(int n)
    {
    cout << endl << "[Saucisson Cuda]" << endl;
    float saucissonGPU = computeSaucisson(n);
    printf("[GPU SAUCISSON] : %f\n",saucissonGPU);
    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ float computeSaucisson(int n)
    {
    float res = 0.0;
    float* ptrHostResult=&res;
    float* ptrDevResult;

    dim3 dg = dim3(32,1,1);
    dim3 db = dim3(64,1,1);

    Device::checkDimError(dg,db);
    size_t size=sizeof(float);

    HANDLE_ERROR(hipMalloc((void**) &ptrDevResult, size));
    HANDLE_ERROR(hipMemset(ptrDevResult,0,size)) ;
    HANDLE_ERROR(hipMemcpy(ptrHostResult, ptrDevResult, size, hipMemcpyDeviceToHost));// Device -> Host

    int nbThreads = db.x;
    size_t sizeSM = sizeof(float)*nbThreads;

    computeSaucissonGPU<<<dg,db,sizeSM>>>(n, ptrDevResult,nbThreads);


    hipDeviceSynchronize();

    HANDLE_ERROR(hipMemcpy(ptrHostResult, ptrDevResult, size, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(ptrDevResult));

    double DX = 1.0/(double)n;
    return res*DX;
    }

__global__ void computeSaucissonGPU(int n, float* ptrDevResult,int length)
    {
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    extern __shared__ float tabSM[];

    reduceIntraThreadSaucisson(tabSM, n);
    __syncthreads();
    CUDAUtils::reduceIntraBlock(tabSM);
    CUDAUtils::reduceInterBlock(tabSM,ptrDevResult);
    }

__device__ float f(float x)
    {
    return 4.0 / (1.0 + x * x); // see pdf
    }

__device__ void reduceIntraThreadSaucisson(float* tabSM, int n)
    {
    const int NBTHREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocalBlock();

    const double DX = 1.0/(double)n;
    int s = TID;
    float sumCrtThread = 0.0;
    float xs = 0.0;

    while(s<n)
	{
	xs = s * DX;
	sumCrtThread += f(xs);
	s+=NBTHREAD;
	}

    tabSM[TID_LOCAL] = sumCrtThread;
    }



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

