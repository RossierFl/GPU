#include "hip/hip_runtime.h"

#include "AleaTools.h"
#include "Device.h"
#include <stdio.h>
#include <limits.h>
#include "cudaTools.h"
#include "Indice1D.h"
#include "reduction.h"

#define WIDTH 20
#define HEIGHT 20
#define SIZE (WIDTH * HEIGHT)
#define MIN 0
#define MAX 255

#define DEBUG 1

__host__ void swap(uint* image, const uint IMAGE_SIZE);
__host__ void fillImage(uint* image, const uint IMAGE_SIZE);
__host__ void useHistogramme();
__global__ void hist(uint* ptrImageDevGM, size_t sizeImage, uint* ptrHistogrammeDevGM, size_t sizeHistogramme);

__host__ void swap(uint* image, const uint IMAGE_SIZE)
{
	AleaTools r;
	uint a = r.uniformeAB(0, IMAGE_SIZE - 1);
	uint b = r.uniformeAB(0, IMAGE_SIZE - 1);

	uint temp = image[a];
	image[a] = image[b];
	image[b] = temp;
}

__host__ void fillImage(uint* image, const uint IMAGE_SIZE)
{
	// rempli
	for(int i = 0; i < IMAGE_SIZE; i++)
	{
		*image = i % (MAX + 1);
		image++;
	}

	// mélange les données
	for(int i = 0; i < SIZE * 10; i++)
	{
		//swap(image, IMAGE_SIZE);
	}
}

__host__ void useHistogramme()
{
	// Parameters
	uint nThreadPerBlock = 4;
	uint nBlockPerMP = 2;
	dim3 dg(nBlockPerMP, 1, 1); // TODO to optimize
	dim3 db(nThreadPerBlock, 1, 1); // TODO to optimize
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#ifdef DEBUG
	printf("Parameters\n");
#endif

	// Image
	size_t sizeImage = sizeof(uint) * SIZE;
	uint* ptrImageRAM = new uint[SIZE];
	fillImage(ptrImageRAM, SIZE);
#ifdef DEBUG
	printf("Fill image\n");
#endif

	// Image en GRAM
	uint* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrImageDevGM, sizeImage));
	HANDLE_ERROR(hipMemcpy(ptrImageDevGM, ptrImageRAM, sizeImage, hipMemcpyHostToDevice));
#ifdef DEBUG
	printf("Image en GRAM\n");
#endif

	// Histogramme en RAM
	uint* ptrHistogrammeRAM = new uint[MAX - MIN + 1];
#ifdef DEBUG
	printf("histogramme en RAM\n");
#endif

	// Histogramme en GRAM
	uint* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(uint) * (MAX - MIN + 1);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));
#ifdef DEBUG
	printf("histogramme en GRAM\n");
#endif

	// call kernel
	hist<<<dg,db,sizeHistogramme>>>(ptrImageDevGM, sizeImage, ptrHistogrammeDevGM, sizeHistogramme);
	Device::checkKernelError("Kernel error: hist");
#ifdef DEBUG
	printf("Kernel finished\n");
#endif

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(ptrHistogrammeRAM, ptrHistogrammeDevGM, sizeHistogramme, hipMemcpyDeviceToHost)); // barrière de synchronisation
#ifdef DEBUG
	printf("final histogramme copied\n");
#endif

	// Affichage du résultat
	uint sum = 0;
	for(uint i = MIN; i <= MAX; i++)
	{
		sum += ptrHistogrammeRAM[i];
		printf("hist(%d) = %d\n", i, ptrHistogrammeRAM[i]);
	}
	if(sum == SIZE) printf("OK, sum == SIZE\n");
	else printf("ERROR !!!!!!!!!!, sum != SIZE\n");
}

__global__ void hist(uint* ptrImageDevGM, size_t sizeImageByte, uint* ptrHistogrammeDevGM, size_t sizeHistogrammeByte)
{
	// @formatter:off
	extern __shared__ float tabSM[]; // 1 instance per block !
	// @formatter:on

	initTabSM(tabSM, MAX + 1, 0);

	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocal();
	const uint NB_THREAD = Indice1D::nbThread();

	uint s = TID;
	while(s < SIZE)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&tabSM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}

	// synchronization entre les threads du meme bloc
	__syncthreads();

	// merge local hist with global hist
	s = TID;
	while(s < MAX + 1)
	{
		// work >>>>
		atomicAdd(&ptrHistogrammeDevGM[s], tabSM[s]);
		// <<<< end work
		s += NB_THREAD;
	}
}
