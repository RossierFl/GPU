#include "hip/hip_runtime.h"
#include "AleaTools.h"
#include "Device.h"
#include <stdio.h>
#include <limits.h>
#include "cudaTools.h"
#include "Indice1D.h"
#include "reduction.h"
#include <assert.h>

#define WIDTH 2000
#define HEIGHT 2000
#define SIZE (WIDTH * HEIGHT)
#define MIN 0 /* SHOULD BE 0 ! */
#define MAX 255
#define NB_POSSIBLE_VALUE (MAX - MIN + 1)

// #define DEBUG 1

static __host__ void swap(uint* image, const uint IMAGE_SIZE);
static __host__ void fillImage(uint* image, const uint IMAGE_SIZE);
__host__ bool useHistogramme();
static __host__ bool checkHistogramme(uint* image, const uint IMAGE_SIZE, uint* hist);
static __global__ void hist(uint* ptrImageDevGM, size_t sizeImage, uint* ptrHistogrammeDevGM, size_t sizeHistogramme);

static __host__ void swap(uint* image, const uint IMAGE_SIZE)
{
	AleaTools r;
	for(int i = 0; i < IMAGE_SIZE * 10; i++)
	{
		int a = r.uniformeAB(0, IMAGE_SIZE - 1);
		int b = r.uniformeAB(0, IMAGE_SIZE - 1);

		uint tempA = image[a];
		uint tempB = image[b];

		image[a] = tempB;
		image[b] = tempA;
	}
}

static __host__ void fillImage(uint* image, const uint IMAGE_SIZE)
{
	// rempli
	uint* itr = image;
	for(uint i = 0; i < IMAGE_SIZE; i++)
	{
		*itr = (i * 11) % (NB_POSSIBLE_VALUE);
		itr++;
	}

	swap(image, IMAGE_SIZE);
}

static __global__ void hist(uint* ptrImageDevGM, size_t sizeImageByte, uint* ptrHistogrammeDevGM, size_t sizeHistogrammeByte)
{
	// @formatter:off
	extern __shared__ uint tabSM[];// 1 instance per block !
	// @formatter:on

	initTabSM(tabSM, NB_POSSIBLE_VALUE, 0);

	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocal();
	const uint NB_THREAD = Indice1D::nbThread();
	const uint NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

	uint s = TID;
	while(s < SIZE)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&tabSM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}

	// synchronization entre les threads du meme bloc
	__syncthreads();

	// merge local hist with global hist
	s = TID_LOCAL;
	while(s < NB_POSSIBLE_VALUE)
	{
		// work >>>>
		atomicAdd(&ptrHistogrammeDevGM[s], tabSM[s]);
		// <<<< end work
		s += NB_THREAD_LOCAL;
	}
}

static __host__ bool checkHistogramme(uint* image, const uint IMAGE_SIZE, uint* hist)
{
	bool valid = true;

	// init empty histogramme
	uint counts[NB_POSSIBLE_VALUE];
	for(int i = 0; i < NB_POSSIBLE_VALUE; i++)
	{
		counts[i] = 0;
	}

	// construct histogramme on CPU
	for(int i = 0; i < SIZE; i++)
	{
		uint v = *(image + i);
		counts[v]++;
	}

	// compare results
	uint sum2 = 0;
	for(int i = 0; i < NB_POSSIBLE_VALUE; i++)
	{
		sum2 += counts[i];
		if(counts[i] != hist[i])
		{
			valid = false;
			printf("ERROR ! : counts[%d] = %d but hist[%d] = %d\n", i, counts[i], i, hist[i]);
		}
	}

	if(sum2 != IMAGE_SIZE) {
		valid = false;
		printf("sum2 = %d\n", sum2);
	}

	return valid;
}

__host__ bool useHistogramme() {

	printf("\n[Histogramme]\n");
	printf("Image size: %dx%d (%d samples)\n", WIDTH, HEIGHT, SIZE);
	printf("Min value: %d\n", MIN);
	printf("Max value: %d\n", MAX);

	// Parameters
	uint nThreadPerBlock = 16;
	uint nBlockPerMP = 32;
	dim3 dg(nBlockPerMP, 1, 1); // TODO to optimize
	dim3 db(nThreadPerBlock, 1, 1); // TODO to optimize
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// Image
	size_t sizeImage = sizeof(uint) * SIZE;
	uint* ptrImageRAM = new uint[SIZE];
	fillImage(ptrImageRAM, SIZE);

	// Image en GRAM
	uint* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrImageDevGM, sizeImage));
	HANDLE_ERROR(hipMemcpy(ptrImageDevGM, ptrImageRAM, sizeImage, hipMemcpyHostToDevice));

	// Histogramme en RAM
	uint* ptrHistogrammeRAM = new uint[NB_POSSIBLE_VALUE];

	// Histogramme en GRAM
	uint* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(uint) * (NB_POSSIBLE_VALUE);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));

	// call kernel
	hist<<<dg,db,sizeHistogramme>>>(ptrImageDevGM, sizeImage, ptrHistogrammeDevGM, sizeHistogramme);
	Device::checkKernelError("Kernel error: hist");
	Device::synchronize();

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(ptrHistogrammeRAM, ptrHistogrammeDevGM, sizeHistogramme, hipMemcpyDeviceToHost)); // barrière de synchronisation

	// Libération de la mémoire
	HANDLE_ERROR(hipFree(ptrHistogrammeDevGM));

	// Affichage du résultat
	uint sum = 0;
	for (uint i = MIN; i <= MAX; i++) {
		sum += ptrHistogrammeRAM[i];
#ifdef DEBUG
		printf("hist(%d) = %d\n", i, ptrHistogrammeRAM[i]);
#endif
	}

	bool isOk = checkHistogramme(ptrImageRAM, SIZE, ptrHistogrammeRAM);
	printf("isOk = %d\n", isOk);

	return isOk;
}
