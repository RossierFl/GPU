#include "hip/hip_runtime.h"
//#include <curandTools.h>
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include <assert.h>
#include <math.h>
#include <limits.h>
#include "Device.h"

#include <hiprand/hiprand_kernel.h>
#include "montecarloDeviceMulti.h"

__device__ float uniformMulti( float min, float max, hiprandState& localState);
__device__ float fMulti(float x);
__global__ void computeMontecarloGPUMulti(float* ptrDevGM, int nTabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom);
__device__ void reduce_intra_thread_montecarloMulti(float* tabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom);
__host__ bool isMontecarlo_OkMulti(long n, float xMin, float xMax, int m);
__global__ void setup_kernel_rand_montecarloMulti(hiprandState* ptrDevTabGeneratorThread, int deviceId);

/*--------------------------------------------
 * Reductions
 -------------------------------------------------*/

 __device__ static void init_tabSMMulti(float * tabSM,int nTabSM){
     const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();
     int s = Indice1D::tidLocal();
     while(s<nTabSM){
 	 tabSM[s]=0;//TID local
 	 s+=NB_THREAD_LOCAL;
     }

     __syncthreads;


 }

 static __device__ void ecrasementMulti(float * tabSM,int moitier){

     const int NBR_THREAD = Indice1D::nbThreadBlock();
     const int TID_LOCAL = Indice1D::tidLocalBlock();
     int saut = moitier;
     int s = TID_LOCAL;
     while(s<moitier){
 	tabSM[s]+=tabSM[s+saut];
 	s+=NBR_THREAD;

     }


 }

 static __device__ void reduction_intra_blockMulti(float * tabSM,int n){
     const int NBR_THREAD = Indice1D::nbThread();

             int moitier = n/2;

             while(moitier>=1){
         	ecrasementMulti(tabSM,moitier);

                 moitier/=2;
                 __syncthreads;//Faire attention
             }




 }

 static __device__ void reduction_inter_blockMulti(float * tabSM,float * ptrDevResult,int nTabSM){
     const int TID_LOCAL = Indice1D::tidLocalBlock();
     const int NBR_THREAD_LOCAL = Indice1D::nbThreadBlock();
     int s = TID_LOCAL;

     while(s<=nTabSM){
 	//Lock lock(&mutex);
 	//lock.lock();
 	//*ptrDevResult+=tabSM[0];
 	atomicAdd(&ptrDevResult[s],tabSM[s]);
 	s+=NBR_THREAD_LOCAL;
 	//lock.unlock();

     }

 }


/*---------------------------------------------------------------------------*
 * Lancement
 *---------------------------------------------------------------------------*/

/*__host__ bool isMontecarlo_Ok(long n, float xMin, float xMax, int m) {
	printf("Montecarlo[n=%d, min=%f, max=%f, m=%d]\n", n, xMin, xMax, m);
	float resultHost = 0;
	float* ptrDevGM = NULL;
	size_t size = sizeof(float);
	int nThreadPerBlock = 4;
	size_t sizeTabSM = sizeof(float) * nThreadPerBlock;
	HANDLE_ERROR(hipMalloc(&ptrDevGM, size));
	HANDLE_ERROR(hipMemset(ptrDevGM, 0, size));

	dim3 dg(1, 1, 1); // TODO to optimize
	dim3 db(nThreadPerBlock, 1, 1); // TODO to optimize
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);

	//init random generator
	hiprandState* ptrDevRandom;
	size_t sizeRandom = sizeof(hiprandState) * nThreadPerBlock;
	HANDLE_ERROR(hipMalloc((void**) &ptrDevRandom, sizeRandom));
	HANDLE_ERROR(hipMemset(ptrDevRandom, 0, sizeRandom));
	setup_kernel_rand_montecarlo<<<dg,db>>>(ptrDevRandom,Device::getDeviceId());
	Device::checkKernelError("setup_kernel_rand");

	computeMontecarloGPU<<<dg,db,sizeTabSM>>>(ptrDevGM, sizeTabSM, xMin, xMax, m, n, ptrDevRandom);
	Device::checkKernelError("computeMontecarloGPU");

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(&resultHost, ptrDevGM, size, hipMemcpyDeviceToHost)); // barrière de synchronisation

	// Affichage du résultat
	printf("Montecarlo[n=%d, min=%f, max=%f, m=%d] = %f\n", n, xMin, xMax, m, resultHost);

	return true;
}*/



__device__ float uniformMulti(const float MIN, const float MAX, hiprandState& localState) {
	float r = hiprand_uniform(&localState);
	return MIN + r * (MAX - MIN);
}

/*---------------------------------------------------------------------------*
 * Montecarlo
 *---------------------------------------------------------------------------*/

__device__ float fMulti(float x) {
	return 0.3;
}

__global__ void computeMontecarloGPUMulti(float* ptrDevGM, int nTabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom){
	extern __shared__ float tabSM[];// 1 instance per block !
	const int TID = Indice1D::tid();
	init_tabSMMulti(tabSM, nTabSM);

	__syncthreads();
	reduce_intra_thread_montecarloMulti(tabSM, xMin, xMax, m, n, ptrDevRandom);
	__syncthreads();
	reduction_intra_blockMulti(tabSM,nTabSM);
	__syncthreads();
	reduction_inter_blockMulti(tabSM, ptrDevGM,nTabSM);
	if(TID == 0) *ptrDevGM = *ptrDevGM / n * (xMax - xMin) * m;
}

__device__ void reduce_intra_thread_montecarloMulti(float* tabSM, float xMin, float xMax, int m, long n,
        hiprandState* tabGeneratorThread) {
	const int TID = Indice1D::tid();
	const int TID_LOCAL = Indice1D::tidLocal();
	const int NB_THREAD = Indice1D::nbThread();

	hiprandState localState = tabGeneratorThread[TID];
	float intraThreadSum = 0;
	int s = TID;
	while (s < n) {
		float xAlea = uniformMulti(xMin, xMax, localState);
		float yAlea = uniformMulti(0, m, localState);
		if (yAlea < fMulti(xAlea)) intraThreadSum++;
		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = intraThreadSum;
}

/*
 * I have to used this hack to avoid multiple definition
 */
__global__ void setup_kernel_rand_montecarloMulti(hiprandState* ptrDevTabGeneratorThread, int deviceId)
    {
    int tid = Indice1D::tid();

    // Customisation du generator: Proposition (au lecteur de faire mieux)
    // Contrainte : Doit etre diff�rent d'un GPU � l'autre
    int deltaSeed = deviceId * INT_MAX;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;

    int seed = 1234 + deltaSeed;    // deviceId+tid;
    int sequenceNumber = tid + deltaSequence;    // + tid;
    int offset = deltaOffset;

    //Each thread gets same seed , a different sequence number , no offset
    hiprand_init(seed, sequenceNumber, offset, &ptrDevTabGeneratorThread[tid]);
    }

void MontecarloDeviceMulti::runMontecarloMulti(hiprandState* ptrDevRandom,float* ptrDevGM,float  xMin,float xMax,int m,long n,size_t sizeTabSM,dim3 dg,dim3 db){
    setup_kernel_rand_montecarloMulti<<<dg,db>>>(ptrDevRandom,Device::getDeviceId());
    Device::checkKernelError("setup_kernel_rand");

    computeMontecarloGPUMulti<<<dg,db,sizeTabSM>>>(ptrDevGM, sizeTabSM, xMin, xMax, m, n, ptrDevRandom);
    Device::checkKernelError("computeMontecarloGPU");
}

