#include "hip/hip_runtime.h"
//#include <curandTools.h>
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include <assert.h>
#include <math.h>
#include <limits.h>

#include <hiprand/hiprand_kernel.h>
#include "UtilsCuda.h"
#include <omp.h>

__device__ float uniformMulti( float min, float max, hiprandState& localState);
__device__ float fMulti(float x);
__global__ void computeMontecarloGPUMulti(float* ptrDevGM, int nTabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom);
__device__ void reduce_intra_thread_montecarloMulti(float* tabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom);
__host__ bool isMontecarlo_OkMulti(long n, float xMin, float xMax, int m);
__global__ void setup_kernel_rand_montecarloMulti(hiprandState* ptrDevTabGeneratorThread, int deviceId);


/*---------------------------------------------------------------------------*
 * Lancement
 *---------------------------------------------------------------------------*/

__host__ bool isMontecarloMulti_Ok(long n, float xMin, float xMax, int m) {
	printf("MontecarloMulti[n=%d, min=%f, max=%f, m=%d]\n", n, xMin, xMax, m);
	float* resultHost = new float[6];
	assert(n%6==0);//could be more generic
	int nbNPerGPU = n/6;
	int nThreadPerBlock = 4;
	dim3 dg(1, 1, 1); // TODO to optimize
	dim3 db(nThreadPerBlock, 1, 1); // TODO to optimize
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);

	Device::checkKernelError("setup_kernel_rand_montecarloMulti");
	int nbGPU = 6; //could be more generic
	omp_set_num_threads(nbGPU);
#pragma omp parralel for
	for(int i=0;i<nbGPU;i++)
	  {
	    hipSetDevice(i);
	   // printf("i:%d\n",i);

	    //init random generator
	    hiprandState* ptrDevRandom;
	    size_t sizeRandom = sizeof(hiprandState) * nThreadPerBlock;
	    HANDLE_ERROR(hipMalloc((void**) &ptrDevRandom, sizeRandom));
	    HANDLE_ERROR(hipMemset(ptrDevRandom, 0, sizeRandom));
	    setup_kernel_rand_montecarloMulti<<<dg,db>>>(ptrDevRandom,Device::getDeviceId());

	    float* ptrDevGM = NULL;
	    size_t size = sizeof(float);
	    size_t sizeTabSM = sizeof(float) * nThreadPerBlock;
	    HANDLE_ERROR(hipMalloc(&ptrDevGM, size));
	    HANDLE_ERROR(hipMemset(ptrDevGM, 0, size));

	    computeMontecarloGPUMulti<<<dg,db,sizeTabSM>>>(ptrDevGM, sizeTabSM, xMin, xMax, m, nbNPerGPU, ptrDevRandom);
	    Device::checkKernelError("computeMontecarloGPUMulti");

	    // Récupération du résultat
	    HANDLE_ERROR(hipMemcpy(&resultHost[i], ptrDevGM, size, hipMemcpyDeviceToHost)); // barrière de synchronisation
	  }
	float result = 0.0;
	for(int i=0;i<nbGPU;i++)//could be more generic
	  {
	      result += resultHost[i];
	  }
	result /= nbGPU;


	// Affichage du résultat
	printf("MontecarloMulti[n=%d, min=%f, max=%f, m=%d] = %f\n", n, xMin, xMax, m, result);

	return true;
}



__device__ float uniformMulti(const float MIN, const float MAX, hiprandState& localState) {
	float r = hiprand_uniform(&localState);
	return MIN + r * (MAX - MIN);
}

/*---------------------------------------------------------------------------*
 * Montecarlo
 *---------------------------------------------------------------------------*/

__device__ float fMulti(float x) {
	return 0.3;
}

__global__ void computeMontecarloGPUMulti(float* ptrDevGM, int nTabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom){
	extern __shared__ float tabSM[];// 1 instance per block !
	const int TID = Indice1D::tid();
	UtilsCUDA::initialise(tabSM, nTabSM);

	__syncthreads();
	reduce_intra_thread_montecarloMulti(tabSM, xMin, xMax, m, n, ptrDevRandom);
	__syncthreads();
	UtilsCUDA::reduce_intra_block(tabSM);
	__syncthreads();
	UtilsCUDA::reduce_inter_block(tabSM, ptrDevGM);
	if(TID == 0) *ptrDevGM = *ptrDevGM / n * (xMax - xMin) * m;
}

__device__ void reduce_intra_thread_montecarloMulti(float* tabSM, float xMin, float xMax, int m, long n,
        hiprandState* tabGeneratorThread) {
	const int TID = Indice1D::tid();
	const int TID_LOCAL = Indice1D::tidLocal();
	const int NB_THREAD = Indice1D::nbThread();

	hiprandState localState = tabGeneratorThread[TID];
	float intraThreadSum = 0;
	int s = TID;
	while (s < n) {
		float xAlea = uniformMulti(xMin, xMax, localState);
		float yAlea = uniformMulti(0, m, localState);
		if (yAlea < fMulti(xAlea)) intraThreadSum++;
		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = intraThreadSum;
}

/*
 * I have to used this hack to avoid multiple definition
 */
__global__ void setup_kernel_rand_montecarloMulti(hiprandState* ptrDevTabGeneratorThread, int deviceId)
    {
    int tid = Indice1D::tid();

    // Customisation du generator: Proposition (au lecteur de faire mieux)
    // Contrainte : Doit etre diff�rent d'un GPU � l'autre
    int deltaSeed = deviceId * INT_MAX;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;

    int seed = 1234 + deltaSeed;    // deviceId+tid;
    int sequenceNumber = tid + deltaSequence;    // + tid;
    int offset = deltaOffset;

    //Each thread gets same seed , a different sequence number , no offset
    hiprand_init(seed, sequenceNumber, offset, &ptrDevTabGeneratorThread[tid]);
    }


