#include "hip/hip_runtime.h"
// Attention : Extension .cu
#include <curandTools.h>
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "UtilsCuda.h"
#include <assert.h>
#include <math.h>
#include <limits.h>

//#include <hiprand/hiprand_kernel.h>
#include "Lock.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  bool isHistogram_Ok(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__  static bool computeHistogram(int n);
__global__ static void computeHistogramGPU(int n,bool* ptrDevRes, int ptrDevTabData[], float* ptrDevTabHisto,int lenght);
__global__ static void initTabData(hiprandState* tabGeneratorThread,int n,int ptrDevTabData[], int lenght);
__device__ void reduce_intra_thread_histogram(int ptrDevTabData[], float* tabSM, int n);
__device__ bool checkHistogram(float* ptrDevRes,int lenght,int n);
__device__ static void reduce_inter_block_histogram(float* tabSM, float* tabHistogramGM);
#define HISTOGRAM_SIZE 256


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  bool isHistogram_Ok(int n)
    {
    cout << endl << "[Histogram Cuda 2]" << endl;


    bool histogramGPU = computeHistogram(n);

    //cout <<"\n[GPUPI] "<< sumPiGPU << endl;
    //printf("[GPU_Histogram] %f\n",histogramGPU);

    return histogramGPU;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ bool computeHistogram(int n)
    {
    assert(n%HISTOGRAM_SIZE==0&&"n mut be a multiple of HISTOGRAM_SIZE");
    bool res = false;
    bool* ptrRes=&res;	// on host (CPU)
    bool* ptrDevRes; 	// on device (GPU)
    // Specifier nb thread : ici 1 thread au total !
    //dim3 dg = dim3(2,1,1);
    //dim3 db = dim3(4,1,1);
    dim3 dg = dim3(2,1,1);
    dim3 db = dim3(4,1,1);

    // Debug
    Device::checkDimError(dg,db);

    //init result
    size_t size_res=sizeof(bool);
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size_res)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size_res)) ;
    //HANDLE_ERROR(hipMemcpy(ptrRes, ptrDevRes, size_res, hipMemcpyDeviceToHost));// Device -> Host

    //init tab data
    int* ptrDevTabData;
    size_t sizeTabData = sizeof(int)*n;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevTabData, sizeTabData)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevTabData,0,sizeTabData)) ;

    //init tab histogram final
    float* ptrDevTabHisto;
    size_t sizeTabHisto = sizeof(float)*HISTOGRAM_SIZE;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevTabHisto, sizeTabHisto)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevTabHisto,0,sizeTabHisto));

    //init random generator
    hiprandState* ptrDevRandom;
    size_t sizeRandom = sizeof(hiprandState);
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRandom, sizeRandom)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRandom,0,sizeRandom));
    setup_kernel_rand<<<dg,db>>>(ptrDevRandom,Device::getDeviceId());
    Device::checkKernelError("setup_kernel_rand");

    //init tabData
    initTabData<<<dg,db>>>(ptrDevRandom,n,ptrDevTabData,HISTOGRAM_SIZE);
    Device::checkKernelError("initTabData"); // facultatif


    int length = HISTOGRAM_SIZE;
    size_t sizeTabSM = sizeof(float)*length;
    assert((length&(length-1))==0&&"Whoops, length must be a power of two !");
    //int n,bool* ptrRes,float* ptrDevRes, float* ptrDevTabData,int lenght
    //computeHistogramGPU<<<dg,db,sizeTabSM>>>(n,ptrDevRes, ptrDevTabData,ptrDevTabHisto,HISTOGRAM_SIZE); // asynchrone !!
    Device::checkKernelError("computeHistogramGPU"); // facultatif

    //v1
    Device::synchronize();// Pour printf sur  GPU

    // memoryManagement => barrier de synchronisation
    HANDLE_ERROR(hipMemcpy(ptrRes, ptrDevRes, size_res, hipMemcpyDeviceToHost));// Device -> Host
    HANDLE_ERROR(hipFree(ptrDevRes)); // device dispose memory in (*)
    HANDLE_ERROR(hipFree(ptrDevTabData)); // device dispose memory in (*)
    HANDLE_ERROR(hipFree(ptrDevTabHisto)); // device dispose memory in (*)
    HANDLE_ERROR(hipFree(ptrDevRandom)); // device dispose memory in (*)
    //HANDLE_ERROR(hipFree(ptrDevTabData)); // device dispose memory in (*)
    return res;
    }


__global__ static void initTabData(hiprandState* ptrDevTabGeneratorThread,int n,int ptrDevTabData[], int lenght){
	  //init hiprand
	  /*int tid = Indice1D::tid();
	  // Global Memory -> Register
	  hiprandState localState = ptrDevTabGeneratorThread[tid]; //Optimisation
	  const int NBTHREAD = Indice1D::nbThread();
          const int TID = Indice1D::tid();

          int s = TID;
          while(s<n){
              ptrDevTabData[s]=s%HISTOGRAM_SIZE;
              s+=NBTHREAD;
          }
          __syncthreads();
          if(TID==0){
              for(int i = 0; i<n;i++)printf("value before swap %i\n",ptrDevTabData[i]);
          }
          s = TID;
          //xAlea = hiprand_uniform(&generatorThread);

          //assert(idx1==idx2);



	  while(s<n){
	      float idx1 = hiprand_uniform(&localState);
	      float idx2 = hiprand_uniform(&localState);
	      int id1 = (int)(idx1 * ((n) + 1));
	      int id2 = (int)(idx2 * ((n) + 1));
	      //lock.lock();
	      int tmp1 = ptrDevTabData[id1];
	      int tmp2 = ptrDevTabData[id2];
	      Lock lock1=Lock(&id1);
	      Lock lock2=Lock(&id2);
	      lock1.lock();
	      lock2.lock();
	      ptrDevTabData[id1] = tmp2;
	      ptrDevTabData[id2] = tmp1;
	      lock1.unlock();
	      lock2.unlock();
	      assert(tmp1==ptrDevTabData[id2]);
	      assert(tmp2==ptrDevTabData[id1]);
	      s+=NBTHREAD;
	      //lock.unlock();
	  }
	  if(TID==0)for(int i = 0; i<n;i++)printf("value after swap %i\n",ptrDevTabData[i]);*/

}

__global__ void computeHistogramGPU(int n,bool* ptrDevRes, int ptrDevTabData[], float* ptrDevTabHisto,int lenght)
    {

    //une instance par block
      //init tab
      const int TID_LOCAL = Indice1D::tidLocalBlock();
      extern __shared__ float tabSM[];


      UtilsCUDA::initialise(tabSM,lenght);
      UtilsCUDA::initialise(ptrDevTabHisto,lenght);
      reduce_intra_thread_histogram(ptrDevTabData,tabSM, n);
      reduce_inter_block_histogram(tabSM,ptrDevTabHisto);
      *ptrDevRes = checkHistogram(ptrDevTabHisto,lenght,n);
  }

__device__ void reduce_inter_block_histogram(float* tabSM, float* tabHistogramGM){
	  const int NBTHREAD = Indice1D::nbThread();
          const int TID = Indice1D::tid();
          const int TID_LOCAL = Indice1D::tidLocalBlock();

          int s = TID;
          while(s<HISTOGRAM_SIZE){
              atomicAdd(&tabHistogramGM[s],tabSM[s]);
              //atomicAdd(&tabSM[0],1);
              s+=NBTHREAD;
          }

     }

__device__ bool checkHistogram(float* ptrDevRes,int lenght,int n){
	const int NBTHREAD = Indice1D::nbThread();
        const int TID = Indice1D::tid();
        const int TID_LOCAL = Indice1D::tidLocalBlock();

        int s = TID;
        bool isOK = true;
        while(isOK&&s<n){
            //sumCurrentThread += v(s)*w(s);
  	  if(isOK){
  	      if(ptrDevRes[s]!=n/HISTOGRAM_SIZE)isOK=false;
  	  }
            s+=NBTHREAD;
        }
        return isOK;
}


__device__ void reduce_intra_thread_histogram(int ptrDevTabData[], float* tabSM, int n){
      const int NBTHREAD = Indice1D::nbThread();
      const int TID = Indice1D::tid();
      const int TID_LOCAL = Indice1D::tidLocalBlock();

      int s = TID;
      while(s<n){
	  atomicAdd(&tabSM[(int)ptrDevTabData[s]],1);
          s+=NBTHREAD;
      }
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

