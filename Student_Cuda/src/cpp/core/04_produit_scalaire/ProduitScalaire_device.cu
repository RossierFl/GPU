#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "ProduitScalaireDevice.h"
#include "Lock.h"
#include <iostream>
#include "Device.h"
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
#define M_W 50
#define M_V 50
#define VI 1.442249570307408
#define WI 0.7390851332151607

__device__ int mutex=0;
static __global__ void calculeProduitScalaire(int n,int nTabSM,double* ptrDevResult);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/


static __device__ double w(long i){
    double x = abs(cos((double)i));
    for(long j =1;j<=M_W;j++){
	x=x-(cos(x)-x)/(-sin(x)-1);

    }
    return (x/WI)*sqrt((double)i);
   // return sqrt((double)i);

}


static __device__ double v(long i){
    double x =1.5 + abs(cos((double) i));
    for(long j = 1;j < M_V;j++){
	double xCarre=x*x;
	x = x-(xCarre * x-3)/(3*xCarre);
    }
    return (x/VI)*sqrt((double) i);
  //  return sqrt((double) i);
}


static __device__ void ecrasement(double * tabSM,int moitier){

    const int NBR_THREAD = Indice1D::nbThreadBlock();
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    int saut = moitier;
    int s = TID_LOCAL;
    while(s<moitier){
	tabSM[s]+=tabSM[s+saut];
	s+=NBR_THREAD;

    }


}




static __device__ void reduction_intra_block(double * tabSM,int n){
    const int NBR_THREAD = Indice1D::nbThread();

            int moitier = n/2;

            while(moitier>=1){
        	ecrasement(tabSM,moitier);

                moitier/=2;
                __syncthreads;//Faire attention
            }




}

static __device__ void reduction_inter_block(double * tabSM,double * ptrDevResult){
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    if(threadIdx.x==0){
	Lock lock(&mutex);
	lock.lock();
	*ptrDevResult+=tabSM[0];
	//atomicAdd(ptrDevResult,tabSM[0]);
	lock.unlock();

    }

}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __device__ float prodScalaire(long i)
    {
	return w(i)*v(i);
    }

static __device__ void reduction_intra_thread(double * tabSM,long vectorLenght){
    //Executer par thread
       const int NBR_THREAD = Indice1D::nbThread();
       const int NBR_THREAD_LOCAL = Indice1D::nbThreadBlock();
           const int TID = Indice1D::tid();
           const int TID_LOCAL = Indice1D::tidLocalBlock();
           long s = TID;
           double sommeThread=0;
           while(s<vectorLenght){

               sommeThread+=prodScalaire(s);
               s+=NBR_THREAD;

           }


           tabSM[TID_LOCAL]=sommeThread;
           __syncthreads;
}
__device__ static void init_tabSM(double * tabSM,int nTabSM){
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();
    int s = Indice1D::tidLocal();
    while(s<nTabSM){
	 tabSM[s]=0;//TID local
	 s+=NB_THREAD_LOCAL;
    }

    __syncthreads;


}

static __global__ void calculeProduitScalaire(int n, int nTabSM,double* ptrDevResult){
   //Une instance par bloc
     extern __shared__ double tabSM[];

    init_tabSM(tabSM,nTabSM);
    reduction_intra_thread(tabSM,n);
    reduction_intra_block(tabSM,nTabSM);
    reduction_inter_block(tabSM,ptrDevResult);


}


 void ProduitScalaireDevice::runProduitScalaire(int n,int  nTabSM,double * ptrDevResult,dim3 dg,dim3 db){
     size_t size = nTabSM*sizeof(double);
     calculeProduitScalaire<<<dg,db,size>>>(n,nTabSM,ptrDevResult);//asynchronous
     Device::checkKernelError("calculeProduitScalaire");

}

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

