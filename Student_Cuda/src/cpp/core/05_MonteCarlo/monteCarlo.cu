#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdio>
#include <hiprand/hiprand_kernel.h>

#include "cuda_utils.h"
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isMonteCarlo_Ok(long n, float xMin, float xMax, int m);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ static float computeMonteCarlo(int n);
__global__ static void computeMonteCarloGPU(float* ptrDevGM, int nTabSM, float xMin, float xMax,
	int m, long n, hiprandState* ptrDevRandom);
__device__ static void reduceIntraThreadMonteCarlo(float* tabSM, float xMin, float xMax, int m, long n,
	hiprandState* generator);
__device__ static float uniform( float min, float max, hiprandState &localState);
__global__ static void setup_kernel_rand(hiprandState* tabGeneratorThread,int deviceId);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isMonteCarlo_Ok(long n, float xMin, float xMax, int m)
    {
    float resultHost = 0;
    float* ptrDevResult = NULL;
    size_t size = sizeof(float);
    const int nbThreadBlock = 16;
    size_t sizeSM = sizeof(float) * nbThreadBlock;

    HANDLE_ERROR(hipMalloc(&ptrDevResult, size));
    HANDLE_ERROR(hipMemset(ptrDevResult, 0, size));

    dim3 dg(1, 1, 1);
    dim3 db(nbThreadBlock, 1, 1);

    //init random generator
    hiprandState* ptrDevRandomGen;
    size_t sizeRandom = sizeof(hiprandState) * nbThreadBlock;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRandomGen, sizeRandom));
    HANDLE_ERROR(hipMemset(ptrDevRandomGen, 0, sizeRandom));

    setup_kernel_rand<<<dg,db>>>(ptrDevRandomGen,Device::getDeviceId());

    //Device::checkKernelError("setup_kernel_rand");

    // carefull using int not size_t
    computeMonteCarloGPU<<<dg,db,sizeSM>>>(ptrDevResult, nbThreadBlock, xMin, xMax, m, n, ptrDevRandomGen);

    //Device::checkKernelError("computeMonteCarloGPU");

    HANDLE_ERROR(hipMemcpy(&resultHost, ptrDevResult, size, hipMemcpyDeviceToHost));
    // no need of cudaDevice because hipMemcpy

    printf("Montecarlo[n: %ld, min: %f, max: %f, m: %d] : %f\n", n, xMin, xMax, m, resultHost);
    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ float computeMonteCarlo(int n)
    {
	return 0.0;
    }

__global__ void computeMonteCarloGPU(float* ptrDevResult, int sizeTabSM, float xMin, float xMax,
	int m, long n, hiprandState* ptrDevRandom)
    {
    extern __shared__ float tabSM[];// 1 instance per block !
    const int TID = Indice1D::tid();
    CUDAUtils::initialize(tabSM, sizeTabSM);
    __syncthreads();
    reduceIntraThreadMonteCarlo(tabSM, xMin, xMax, m, n, ptrDevRandom);
    __syncthreads();
    CUDAUtils::reduceIntraBlock(tabSM);
    __syncthreads();
    CUDAUtils::reduceInterBlock(tabSM,ptrDevResult);
    if(TID == 0)
	*ptrDevResult = *ptrDevResult / n * (xMax - xMin) * m;
    }

__device__ float uniform(const float MIN, const float MAX, hiprandState& localState)
    {
    float r = hiprand_uniform(&localState);
    return MIN + r * (MAX - MIN);
    }

__device__ void reduceIntraThreadMonteCarlo(float* tabSM, float xMin, float xMax, int m, long n,
	hiprandState* generator)
    {
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD = Indice1D::nbThread();
    const float probability = 0.3;
    hiprandState state = generator[TID];
    float intraThreadSum = 0;
    int s = TID;
    while (s < n)
	{
	float xAlea = uniform(xMin, xMax, state);
	float yAlea = uniform(0, m, state);
	if (yAlea < probability)
	    intraThreadSum++;
	s += NB_THREAD;
	}
    tabSM[TID_LOCAL] = intraThreadSum;
    }

//Nvidia Doc : Each thread gets same seed , a different sequence number , no offset
__global__ void setup_kernel_rand(hiprandState* tabGeneratorThread,int deviceId)
    {
    int tid = Indice1D::tid();
    //Customisation du generator: Proposition (au lecteur de faire mieux)
    // Contrainte : Doit etre différent d'un GPU à l'autre
    int deltaSeed=deviceId* INT_MAX;
    int deltaSequence=deviceId *100;
    int deltaOffset=deviceId *100;
    int seed=1234+deltaSeed;
    int sequenceNumber=tid+deltaSequence;
    int offset=deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabGeneratorThread[tid]);
    }



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

