#include "hip/hip_runtime.h"

#include <iostream>
#include <stdlib.h>
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

// #define DEBUG 1

__host__ void useAdd();
static __global__ void add(float* ptrDevV1, float* ptrDevV2, int n, float* ptrDevResult);
static __device__ float work(float v1, float v2);
static __host__ void fillArray(float* ptrArray, int n, int start, int pas);

__host__ void useAdd() {
	int n = 10;

	float* v1 = new float[n];
	float* v2 = new float[n];
	float* vResult = new float[n];
	fillArray(v1, n, 1, 1);
	fillArray(v2, n, 2, 2);

	size_t size = sizeof(float) * n;
	float* ptrDevV1 = NULL;
	float* ptrDevV2 = NULL;
	float* ptrDevResult = NULL;
	HANDLE_ERROR( hipMalloc(&ptrDevV1, size) );
	HANDLE_ERROR( hipMalloc(&ptrDevV2, size) );
	HANDLE_ERROR( hipMalloc(&ptrDevResult, size) );
	//
	HANDLE_ERROR( hipMemset(ptrDevV1, 0, size) );
	HANDLE_ERROR( hipMemset(ptrDevV2, 0, size) );
	HANDLE_ERROR( hipMemset(ptrDevResult, 0, size) );
	//
	HANDLE_ERROR( hipMemcpy(ptrDevV1, v1, size, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(ptrDevV2, v2, size, hipMemcpyHostToDevice) );

	dim3 dg(4, 4, 1);
	dim3 db(4, 8, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	add<<<dg,db>>>(ptrDevV1, ptrDevV2, n, ptrDevResult); // asynchronous
	Device::checkKernelError("erreur kernel");

	HANDLE_ERROR( hipMemcpy(vResult, ptrDevResult, size, hipMemcpyDeviceToHost) ); // barrière de synchronisation
	for(int i = 0; i < n; i++) {
		std::cout << vResult[i] << "|";
	}
	std::cout << std::endl;
}

__host__ void fillArray(float* ptrArray, int n, int start, int pas) {
	for(int i = 0; i < n; i++) {
		ptrArray[i] = start;
		start += pas;
	}
}

__global__ void add(float* ptrDevV1, float* ptrDevV2, int n, float* ptrDevResult) {
	const int NB_THREAD = Indice2D::nbThread();
	const int TID = Indice2D::tid();

	int s = TID;
	while(s < n) {
		ptrDevResult[s] = work(ptrDevV1[s], ptrDevV2[s]);
		s += NB_THREAD;
	}
}

__device__ float work(float v1, float v2) {
	return v1 + v2;
}
