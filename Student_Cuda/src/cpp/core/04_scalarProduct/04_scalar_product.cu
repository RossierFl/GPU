#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "UtilsCuda.h"
#include <assert.h>
#include <math.h>

using std::cout;
using std::endl;

#define M_W 100
#define M_V 100
//#define VI 1.4422495703074083
#define VI 1.4422495703074083017725115
//#define WI 0.7390850782394409
#define WI 0.7390851332151606722931092

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  bool isScalarProduct_Ok(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__  static double computeScalarProduct(int n);
__host__ double theoricalResult(int n);
__global__ static void computeScalarProductGPU(int n,double* ptrDevResult, int lenght);
__device__ void reduce_intra_thread_scalar_product(double* tabSM, int n);
__device__ double v(int i);
__device__ double w(int i);


/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  bool isScalarProduct_Ok(int n)
    {
    cout << endl << "[ScalarProduct Cuda 2]" << endl;


    double scalarProductGPU = computeScalarProduct(n);

    //cout <<"\n[GPUPI] "<< sumPiGPU << endl;
    printf("[GPU_SCALAR_PRODUCT] %f\n",scalarProductGPU);

    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ double computeScalarProduct(int n)
    {
    double res = 0.0;
    double* ptrRes=&res;	// on host (CPU)
    double* ptrDevRes; 	// on device (GPU)
    // Specifier nb thread : ici 1 thread au total !
    dim3 dg = dim3(16,1,1);
    dim3 db = dim3(32,1,1);

    // Debug
     Device::checkDimError(dg,db);

    size_t size=sizeof(double);
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size)) ;
    HANDLE_ERROR(hipMemcpy(ptrRes, ptrDevRes, size, hipMemcpyDeviceToHost));// Device -> Host
    int length = db.x;
    size_t sizeTabSM = sizeof(double)*length;
    assert((length&(length-1))==0&&"Whoops, length must be a power of two !");
    computeScalarProductGPU<<<dg,db,sizeTabSM>>>(n, ptrDevRes,length); // asynchrone !!
    Device::checkKernelError("computeScalarProductGPU"); // facultatif

    //v1
    Device::synchronize();// Pour printf sur  GPU

    // memoryManagement => barrier de synchronisation
    HANDLE_ERROR(hipMemcpy(ptrRes, ptrDevRes, size, hipMemcpyDeviceToHost));// Device -> Host
    HANDLE_ERROR(hipFree(ptrDevRes)); // device dispose memory in (*)
    printf("Result GPU = %f and theorical result is %f\n",res,theoricalResult(n));
    return res;
    }

__host__ double theoricalResult(int n){
    n--;
    return (n/(double)2)*(n+1);
}



__global__ void computeScalarProductGPU(int n, double* ptrDevRes,int lenght)
    {

    //une instance par block
      //init tab
      const int TID_LOCAL = Indice1D::tidLocalBlock();
      extern __shared__ double tabSM[];
      UtilsCUDA::initialise(tabSM,lenght);


      reduce_intra_thread_scalar_product(tabSM, n);
      UtilsCUDA::reduce_intra_block(tabSM);
      UtilsCUDA::reduce_inter_block(tabSM,ptrDevRes);
  }

__device__ void reduce_intra_thread_scalar_product(double* tabSM, int n){
  const int NBTHREAD = Indice1D::nbThread();
      const int TID = Indice1D::tid();
      const int TID_LOCAL = Indice1D::tidLocalBlock();

      int s = TID;
      double sumCurrentThread = 0.0;
      while(s<n){
          sumCurrentThread += v(s)*w(s);
          s+=NBTHREAD;
      }
      //printf("currentSum=%f PI=%f\n in reduce intrathread", sumCurrentThread,sumCurrentThread*DX);
      //cout<<"currentSum"<<sumCurrentThread<<endl;
      tabSM[TID_LOCAL] = sumCurrentThread;
}


__device__ double v(int i){
    const int TID = Indice1D::tid();
    double x = 1.5+abs(cos((double)i));

    for(int j = 1; j<=M_V; j++){
	double xCarre=x*x;
	x = x-(xCarre * x -3)/(3*xCarre);
    }
    //if(TID==0)printf("xv = %.25f",x);
    return (x/VI)*sqrt((double)i);

}

__device__ double w(int i){
  const int TID = Indice1D::tid();
    double x = abs(cos((double)i));
    for(int j = 1; j<=M_W;j++){
	x = x-(cos(x)-x)/(-sin(x)-1);
    }
    //if(TID==0)printf("xw = %.25f",x);
    return (x/WI)*sqrt((double)i);
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

