#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdio>

#include "cuda_utils.h"
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"

#define M_W 100
#define M_V 100
#define VI 1.4422495703074083017725115
#define WI 0.7390851332151606722931092

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isProduitScalaire_Ok(int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ static double computeProduitScalaire(int n);
__host__ double result(int n);
__global__ static void computeProduitScalaireGPU(int n,double* ptrDevResult, int length);
__device__ void reduceIntraThreadProduitScalaire(double* tabSM, int n);
__device__ double v(int i);
__device__ double w(int i);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isProduitScalaire_Ok(int n)
    {
    cout << endl << "[Produit Scalaire Cuda]" << endl;
    double scalarProductGPU = computeProduitScalaire(n);
    printf("[GPU PRODUIT SCALAIRE] : %f\n",scalarProductGPU);
    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ double computeProduitScalaire(int n)
    {
    double res = 0.0;
    double* ptrHostRes=&res;
    double* ptrDevRes;

    dim3 dg = dim3(16,1,1); // block
    dim3 db = dim3(32,1,1); // thread/block

    Device::checkDimError(dg,db);
    size_t size=sizeof(double);
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size));
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size)) ;
    HANDLE_ERROR(hipMemcpy(ptrHostRes, ptrDevRes, size, hipMemcpyDeviceToHost));// Device -> Host
    int length = db.x;
    size_t sizeTabSM = sizeof(double)*length;

    computeProduitScalaireGPU<<<dg,db,sizeTabSM>>>(n, ptrDevRes,length);
    Device::checkKernelError("computeScalarProductGPU");
    //v1
    Device::synchronize();
    HANDLE_ERROR(hipMemcpy(ptrHostRes, ptrDevRes, size, hipMemcpyDeviceToHost));// Device -> Host
    HANDLE_ERROR(hipFree(ptrDevRes));
    printf("Result GPU = %f\n",res);
    printf("Expected result is: %f\n",result(n));
    return res;
    }

__host__ double result(int n)
    {
    n--;
    return (n/2.0)*(n+1);
    }

__global__ void computeProduitScalaireGPU(int n, double* ptrDevResult,int length)
    {
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    extern __shared__ double tabSM[];
    CUDAUtils::initialize(tabSM,length);
    reduceIntraThreadProduitScalaire(tabSM, n);
    CUDAUtils::reduceIntraBlock(tabSM);
    CUDAUtils::reduceInterBlock(tabSM,ptrDevResult);
    }

// add suffix to avoid multiple definition error
__device__ void reduceIntraThreadProduitScalaire(double* tabSM, int n)
    {
    const int NBTHREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    int s = TID;
    double sumCurrentThread = 0.0;
    while(s<n){
	sumCurrentThread += v(s)*w(s);
	s+=NBTHREAD;
    }

    tabSM[TID_LOCAL] = sumCurrentThread;
    }

__device__ double v(int i)
    {
    const int TID = Indice1D::tid();
    double x = 1.5+abs(cos((double)i));
    for(int j = 1; j<=M_V; j++){
	double xCarre=x*x;
	x = x-(xCarre * x -3)/(3*xCarre);
    }

    return (x/VI)*sqrt((double)i);
    }

__device__ double w(int i)
    {
    const int TID = Indice1D::tid();
    double x = abs(cos((double)i));
    for(int j = 1; j<=M_W;j++)
	{
	x = x-(cos(x)-x)/(-sin(x)-1);
	}

    return (x/WI)*sqrt((double)i);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

