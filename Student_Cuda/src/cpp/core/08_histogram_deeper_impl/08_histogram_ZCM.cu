#include "hip/hip_runtime.h"
// Attention : Extension .cu
//#include <curandTools.h>
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "UtilsCuda.h"
#include <assert.h>
#include <math.h>
#include <limits.h>
#include "AleaTools.h"
#include "Chronos.h"
#include "hip/hip_runtime.h"

//#include <hiprand/hiprand_kernel.h>
#include "Lock.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

//__host__  bool isHistogramSM_Ok(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__  void computeHistogram_ZCM(int* dataImg, int dataLength, int* resultHisto, int lengthHisto);
__global__ static void computeHistogramGPU_ZCM(int n,int* ptrDevRes, int* ptrDevTabData,int lenght);
__device__ void reduce_intra_thread_histogram_ZCM(int* ptrDevTabData, int* tabSM, int n);
__device__ static void reduce_inter_block_histogram_ZCM(int* tabSM, int* tabHistogramGM, int histogramSize);
/*__host__ bool checkHistogram_host(int* ptrDevRes,int lenght,int n);
__global__ void checkHistogram_GPU(int* ptrDevRes,int lenght,int n, bool* ptrDevBoolResult);*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ void computeHistogram_ZCM(int* dataImg, int dataLength, int* resultHisto, int lengthHisto)
    {
    //Device::loadCudaDriverAll(true);
    cout << endl << "[Histogram Cuda 2 Zero Copy Memory]" << endl;
    printf("tets\n");

    int* ptrDevRes; 	// on device (GPU)
    dim3 dg = dim3(4,1,1);
    dim3 db = dim3(8,1,1);

    // Debug
    Device::checkDimError(dg,db);
    printf("tetsfadsf\n");
    Chronos chrono;
    chrono.start();

    //HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
    //copy tab data
    size_t size_res=sizeof(int)*lengthHisto;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size_res)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size_res)) ;


    //tab data zerocopy
    printf("tetsfads\n");
    int* dataImgZCM;
    size_t dataSize = dataLength*sizeof(int);
    HANDLE_ERROR(hipHostAlloc((void**)&dataImgZCM,dataSize,hipHostMallocMapped));
    HANDLE_ERROR(hipHostAlloc((void**)&dataImgZCM,dataSize,hipHostMallocMapped|hipHostMallocWriteCombined));
    int flagInutialiser=0;
    int* ptrDevTabData;
    //size_t dataSize = dataLength*sizeof(int);
    memcpy(dataImgZCM, dataImg, dataSize);
    //HANDLE_ERROR(hipMemcpy(dataImgZCM,dataImg,dataSize,hipMemcpyHostToHost));
    HANDLE_ERROR(hipHostGetDevicePointer(&ptrDevTabData,dataImgZCM,flagInutialiser));
    //HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size_res)); // Device memory allocation (*)
    //HANDLE_ERROR(hipMemset(ptrDevRes,0,size_res)) ;
    printf("tets\n");



    size_t sizeTabSM = sizeof(int)*lengthHisto;
    assert((lengthHisto&(lengthHisto-1))==0&&"Whoops, length must be a power of two !");
    chrono.stop();
    chrono.print("MM In : ");
    chrono.start();
    computeHistogramGPU_ZCM<<<dg,db,sizeTabSM>>>(dataLength,ptrDevRes, ptrDevTabData,lengthHisto); // asynchrone !!
    Device::checkKernelError("computeHistogramGPU_ZCM"); // facultatif
    Device::synchronize();// Pour printf sur  GPU
    chrono.stop();
    chrono.print("Computation In : ");
    chrono.start();
    // memoryManagement => barrier de synchronisation
    HANDLE_ERROR(hipMemcpy(resultHisto, ptrDevRes, size_res, hipMemcpyDeviceToHost));// Device -> Host

    HANDLE_ERROR(hipFree(ptrDevRes)); // device dispose memory in (*)
    //HANDLE_ERROR(hipFree(ptrDevTabData)); // device dispose memory in (*)*/
    HANDLE_ERROR(hipHostFree(dataImgZCM));
    //HANDLE_ERROR(hipFree(ptrDevRes));
    chrono.stop();
    chrono.print("MM Out : ");
    }

__global__ void computeHistogramGPU_ZCM(int n,int* ptrDevRes, int* ptrDevTabData,int lenght)
    {

    //une instance par block
      //init tab
      const int TID_LOCAL = Indice1D::tidLocalBlock();
      extern __shared__ int tabSM[];


      UtilsCUDA::initialise(tabSM,lenght);
      //UtilsCUDA::initialise(ptrDevRes,lenght);
      reduce_intra_thread_histogram_ZCM(ptrDevTabData,tabSM, n);
      __syncthreads();
      reduce_inter_block_histogram_ZCM(tabSM,ptrDevRes, lenght);
  }

__device__ void reduce_inter_block_histogram_ZCM(int* tabSM, int* tabHistogramGM, int histogramSize){
	  const int NBTHREAD = Indice1D::nbThread();
	  const int NBTHREADBLOCK = Indice1D::nbThreadBlock();
          const int TID = Indice1D::tid();
          const int TID_LOCAL = Indice1D::tidLocal();

          int s = TID_LOCAL;
          while(s<histogramSize){
              atomicAdd(&tabHistogramGM[s],tabSM[s]);
              s+=NBTHREADBLOCK;
          }

     }


__device__ void reduce_intra_thread_histogram_ZCM(int* ptrDevTabData, int* tabSM, int n){
      const int NBTHREAD = Indice1D::nbThread();
      const int TID = Indice1D::tid();
      const int TID_LOCAL = Indice1D::tidLocalBlock();

      int s = TID;
      while(s<n){
	  int cValue= ptrDevTabData[s];

	  atomicAdd(&tabSM[cValue],1);
          s+=NBTHREAD;
      }
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

