#include "hip/hip_runtime.h"
// Attention : Extension .cu
//#include <curandTools.h>
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "UtilsCuda.h"
#include <assert.h>
#include <math.h>
#include <limits.h>
#include "AleaTools.h"
#include "Chronos.h"

//#include <hiprand/hiprand_kernel.h>
#include "Lock.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

//__host__  bool isHistogramSM_Ok(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__  void computeHistogram_GM(int* dataImg, int dataLength, int* resultHisto, int lengthHisto);
__global__ static void computeHistogramGPU_GM(int n,int* ptrDevRes, int* ptrDevTabData,int lenght);
__device__ void reduce_intra_thread_histogram_GM(int* ptrDevTabData, int* ptrDevRes, int n);
/*__host__ bool checkHistogram_host(int* ptrDevRes,int lenght,int n);
__global__ void checkHistogram_GPU(int* ptrDevRes,int lenght,int n, bool* ptrDevBoolResult);*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ void computeHistogram_GM(int* dataImg, int dataLength, int* resultHisto, int lengthHisto)
    {
    //Device::loadCudaDriverAll(false);
    cout << endl << "[Histogram Cuda 2 Global Memory]" << endl;

    int* ptrDevRes; 	// on device (GPU)
    dim3 dg = dim3(4,1,1);
    dim3 db = dim3(8,1,1);

    // Debug
    Device::checkDimError(dg,db);

    Chronos chrono;
    chrono.start();
    //init result
    size_t size_res=sizeof(int)*lengthHisto;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size_res)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size_res)) ;

    //copy tab data
    int* ptrDevTabData;
    size_t sizeTabData = sizeof(int)*dataLength;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevTabData, sizeTabData)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemcpy(ptrDevTabData,dataImg,sizeTabData,hipMemcpyHostToDevice));



    //size_t sizeTabSM = sizeof(int)*lengthHisto;
    //assert((lengthHisto&(lengthHisto-1))==0&&"Whoops, length must be a power of two !");
    chrono.stop();
    chrono.print("MM In : ");
    chrono.start();
    computeHistogramGPU_GM<<<dg,db>>>(dataLength,ptrDevRes, ptrDevTabData,lengthHisto); // asynchrone !!
    Device::checkKernelError("computeHistogramGPU_GM"); // facultatif

    //v1
    Device::synchronize();// Pour printf sur  GPU
    chrono.stop();
    chrono.print("Computation In : ");
    chrono.start();
    // memoryManagement => barrier de synchronisation
    HANDLE_ERROR(hipMemcpy(resultHisto, ptrDevRes, size_res, hipMemcpyDeviceToHost));// Device -> Host

    HANDLE_ERROR(hipFree(ptrDevRes)); // device dispose memory in (*)
    HANDLE_ERROR(hipFree(ptrDevTabData)); // device dispose memory in (*)
    chrono.stop();
    chrono.print("MM Out : ");
    }

__global__ void computeHistogramGPU_GM(int n,int* ptrDevRes, int* ptrDevTabData,int lenght)
    {

    //une instance par block
      //init tab
      const int TID_LOCAL = Indice1D::tidLocalBlock();
      //extern __shared__ int tabSM[];


      //UtilsCUDA::initialise(tabSM,lenght);
      UtilsCUDA::initialise(ptrDevRes,lenght);
      reduce_intra_thread_histogram_GM(ptrDevTabData,ptrDevRes, n);
      //__syncthreads();
      //reduce_inter_block_histogram_GM(tabSM,ptrDevRes, lenght); useless
  }


__device__ void reduce_intra_thread_histogram_GM(int* ptrDevTabData, int* ptrDevRes, int n){
      const int NBTHREAD = Indice1D::nbThread();
      const int TID = Indice1D::tid();
      const int TID_LOCAL = Indice1D::tidLocalBlock();

      int s = TID;
      while(s<n){
	  int cValue= ptrDevTabData[s];

	  atomicAdd(&ptrDevRes[cValue],1);
          s+=NBTHREAD;
      }
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

