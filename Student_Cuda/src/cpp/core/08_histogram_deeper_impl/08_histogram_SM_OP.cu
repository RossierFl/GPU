#include "hip/hip_runtime.h"
// Attention : Extension .cu
//#include <curandTools.h>
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "UtilsCuda.h"
#include <assert.h>
#include <math.h>
#include <limits.h>
#include "AleaTools.h"
#include "Chronos.h"

//#include <hiprand/hiprand_kernel.h>
#include "Lock.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

//__host__  bool isHistogramSM_Ok(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__  void computeHistogram_SM_OP(int* dataImg, int dataLength, int* resultHisto, int lengthHisto);
__global__ static void computeHistogramGPU_SM_OP(int n,int* ptrDevRes, int* ptrDevTabData,int lenght);
__device__ void reduce_intra_thread_histogram_SM_OP(int* ptrDevTabData, int* tabSM, int n);
__device__ static void reduce_inter_block_histogram_SM_OP(int* tabSM, int* tabHistogramGM, int histogramSize);
/*__host__ bool checkHistogram_host(int* ptrDevRes,int lenght,int n);
__global__ void checkHistogram_GPU(int* ptrDevRes,int lenght,int n, bool* ptrDevBoolResult);*/



/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ void computeHistogram_SM_OP(int* dataImg, int dataLength, int* resultHisto, int lengthHisto)
    {
    //Device::loadCudaDriverAll(false);
    cout << endl << "[Histogram Cuda 2 Shared Memory OP]" << endl;

    int* ptrDevRes; 	// on device (GPU)
    dim3 dg = dim3(32,1,1);
    dim3 db = dim3(192,1,1);

    // Debug
    Device::checkDimError(dg,db);

    Chronos chrono;
    chrono.start();
    //init result
    size_t size_res=sizeof(int)*lengthHisto;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size_res)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size_res)) ;

    //copy tab data
    int* ptrDevTabData;
    size_t sizeTabData = sizeof(int)*dataLength;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevTabData, sizeTabData)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemcpy(ptrDevTabData,dataImg,sizeTabData,hipMemcpyHostToDevice)) ;


    size_t sizeTabSM = sizeof(int)*lengthHisto;
    //assert((lengthHisto&(lengthHisto-1))==0&&"Whoops, length must be a power of two !"); //try to comment this to avoid useles computation
    chrono.stop();
    chrono.print("MM In : ");
    chrono.start();
    computeHistogramGPU_SM_OP<<<dg,db,sizeTabSM>>>(dataLength,ptrDevRes, ptrDevTabData,lengthHisto); // asynchrone !!
    Device::checkKernelError("computeHistogramGPU_SM"); // facultatif
    Device::synchronize();// Pour printf sur  GPU
    chrono.stop();
    chrono.print("Computation In : ");
    chrono.start();
    // memoryManagement => barrier de synchronisation
    HANDLE_ERROR(hipMemcpy(resultHisto, ptrDevRes, size_res, hipMemcpyDeviceToHost));// Device -> Host

    HANDLE_ERROR(hipFree(ptrDevRes)); // device dispose memory in (*)
    HANDLE_ERROR(hipFree(ptrDevTabData)); // device dispose memory in (*)
    chrono.stop();
    chrono.print("MM Out : ");
    }

__global__ void computeHistogramGPU_SM_OP(int n,int* ptrDevRes, int* ptrDevTabData,int lenght)
    {

    //une instance par block
      //init tab
      const int TID_LOCAL = Indice1D::tidLocalBlock();
      extern __shared__ int tabSM[];


      UtilsCUDA::initialise(tabSM,lenght);
      //UtilsCUDA::initialise(ptrDevRes,lenght);
      reduce_intra_thread_histogram_SM_OP(ptrDevTabData,tabSM, n);
      __syncthreads();
      reduce_inter_block_histogram_SM_OP(tabSM,ptrDevRes, lenght);
  }

__device__ void reduce_inter_block_histogram_SM_OP(int* tabSM, int* tabHistogramGM, int histogramSize){
	  const int NBTHREAD = Indice1D::nbThread();
	  const int NBTHREADBLOCK = Indice1D::nbThreadBlock();
          const int TID = Indice1D::tid();
          const int TID_LOCAL = Indice1D::tidLocal();

          int s = TID_LOCAL;
          while(s<histogramSize){
              atomicAdd(&tabHistogramGM[s],tabSM[s]);
              s+=NBTHREADBLOCK;
          }

     }

/*__global__ void checkHistogram_GPU(int* ptrDevRes,int lenght,int n, bool* ptrDevBoolResult){
	extern __shared__ int tabSM[];
	const int NBTHREAD = Indice1D::nbThread();
        const int TID = Indice1D::tid();
        const int TID_LOCAL = Indice1D::tidLocalBlock();

        int s = TID;
        bool isOK = true;
        int expectedResult = n/lenght;
        while(isOK&&s<lenght){
	    if(ptrDevRes[s]!=expectedResult)tabSM[s]=false;
	    else tabSM[s]=true;
            s+=NBTHREAD;
        }
        if(TID_LOCAL)
        __syncthreads();
        *ptrDevBoolResult = isOK;
}
__host__ bool checkHistogram_host(int* ptrDevRes,int lenght,int n){

    dim3 dg = dim3(16,1,1);
    dim3 db = dim3(32,1,1);

    //copy tab data
    int* ptrDevTabData;
    size_t sizeTabData = sizeof(int)*lenght;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevTabData, sizeTabData)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemcpy(ptrDevTabData,ptrDevRes,sizeTabData,hipMemcpyHostToDevice)) ;

    //result
    bool res = false;
    bool* ptrHostRes = &res;
    bool* ptrDevResBool;
    size_t sizeRes = sizeof(bool);
    HANDLE_ERROR(hipMalloc((void**) &ptrDevResBool, sizeRes)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevResBool,0,sizeRes)) ;

    size_t sizeTabSM = sizeof(bool)*lengthHisto;
    checkHistogram_GPU<<<dg,db>>>(ptrDevTabData,lenght,n, ptrDevResBool); // asynchrone !!

    Device::synchronize();

    HANDLE_ERROR(hipMemcpy(ptrHostRes, ptrDevResBool, sizeRes, hipMemcpyDeviceToHost));// Device -> Host

    HANDLE_ERROR(hipFree(ptrDevResBool));
    HANDLE_ERROR(hipFree(ptrDevTabData));
}*/


__device__ void reduce_intra_thread_histogram_SM_OP(int* ptrDevTabData, int* tabSM, int n){
      const int NBTHREAD = Indice1D::nbThread();
      const int TID = Indice1D::tid();
      const int TID_LOCAL = Indice1D::tidLocalBlock();

      int s = TID;
      while(s<n){
	  int cValue= ptrDevTabData[s];

	  atomicAdd(&tabSM[cValue],1);
          s+=NBTHREAD;
      }
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

