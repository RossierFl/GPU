#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include <iostream>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__host__ void useAdd();

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __global__ void add(float* ptrDevV1, float* ptrDevV2, int n, float* ptrDevResult);
static __device__ float work(float v1, float v2);
static __host__ void fillArray(float* floatArray, int n, int k);
/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__host__ void useAdd()
    {
    int n = 3;

    float v1[n];
    fillArray(v1,n,1);

    float v2[n];
    fillArray(v2,n,2);

    float vResult[n];
    size_t size = n*sizeof(float);
    float* ptrDevV1 = NULL;
    float* ptrDevV2 = NULL;
    float* ptrDevResult = NULL;

    HANDLE_ERROR(hipMalloc(&ptrDevV1,size));
    HANDLE_ERROR(hipMalloc(&ptrDevV2,size));
    HANDLE_ERROR(hipMalloc(&ptrDevResult,size));
    //
    HANDLE_ERROR(hipMemset(ptrDevV1,0,size));
    HANDLE_ERROR(hipMemset(ptrDevV2,0,size));
    HANDLE_ERROR(hipMemset(ptrDevResult,0,size));
    //memCpy(src, dst, size, flag-copy)
    HANDLE_ERROR(hipMemcpy(ptrDevV1, v1, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(ptrDevV2, v2, size, hipMemcpyHostToDevice));

    dim3 dg = dim3(2,2,1);
    dim3 db = dim3(2,2,1);
    Device::checkDimError(dg,db);
    add<<<dg,db>>>(ptrDevV1, ptrDevV2, 3, ptrDevResult);//asynchronous
    Device::checkKernelError("addVector");
    HANDLE_ERROR(hipMemcpy(vResult, ptrDevResult, size, hipMemcpyDeviceToHost));//barriere implicite de sync
    std::cout << vResult[0] << "," << vResult[1] << "," << vResult[2] << std::endl;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__global__ void add(float* ptrDevV1, float* ptrDevV2, int n, float* ptrDevResult)
    {
	const int NB_THREADS = Indice2D::nbThread();
	const int TID = Indice2D::tid();

	int s = TID;

	while(s < n)
	    {
	    ptrDevResult[s] = work(ptrDevV1[s], ptrDevV2[s]);
	    s+=NB_THREADS;
	    }
    }

__device__ float work(float v1, float v2)
    {
	return v1 + v2;
    }

__host__ void fillArray(float* floatArray, int n, int k)
    {
	for(int i = 0;i<n;i++)
	    {
	    floatArray[i] = (i+1)*k;
	    }
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

