#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include <stdio.h>
#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __global__ void add(float* ptrDevV1,float* ptrDevV2, int n, float* ptrDevResult);
static __device__ float work(float v1,float v2);
void fillArray(float* tab,int n,int power);
/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

void use(){
  int n = 3;
  float v1[n];
  fillArray(v1,n,1);
  float v2[n];
  fillArray(v2,n,2);

  float vResult[3];
  float* ptrDevV1 = NULL;
  float* ptrDevV2 = NULL;
  float* ptrDevResult =NULL;
  size_t size = 3*sizeof(float);
  HANDLE_ERROR(hipMalloc(&ptrDevV1,size));
  HANDLE_ERROR(hipMalloc(&ptrDevV2,size));
  HANDLE_ERROR(hipMalloc(&ptrDevResult,size));

  HANDLE_ERROR(hipMemset(ptrDevV1,0,size));
  HANDLE_ERROR(hipMemset(ptrDevV2,0,size));
  HANDLE_ERROR(hipMemset(ptrDevResult,0,size));
  //memcpy(src,dst,size,flag-copy)
  HANDLE_ERROR(hipMemcpy(ptrDevV1,v1,size,hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(ptrDevV2,v2,size,hipMemcpyHostToDevice));


  dim3 dg = dim3(2,2,1);
  dim3 db = dim3(2,2,1);
  add<<<dg,db>>>(ptrDevV1,ptrDevV2,3,ptrDevResult);//asynchrone
  HANDLE_ERROR(hipMemcpy(vResult,ptrDevResult,size,hipMemcpyDeviceToHost)); //hipMemcpy = barrière de synchronisation
  Device::checkKernelError("addvector");
  printf("%f,%f,%f",vResult[0],vResult[1],vResult[2]);
}

void fillArray(float* tab,int n,int power){
  for(int i=1;i<n+1;i++){
      tab[i]=i*power;
  }
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void add(float* ptrDevV1,float* ptrDevV2, int n, float* ptrDevResult)
{
  const int NBTHREAD = Indice2D::nbThread();
  const int TID = Indice2D::tid();

  int s = TID;

  while(s<n){
      ptrDevResult[s] = work(ptrDevV1[s],ptrDevV2[s]);
      s+=NBTHREAD;
  }
}

__device__ float work(float v1,float v2){
    return v1+v2;
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

