#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "HistogrammeDevice.h"
#include "Lock.h"
#include <iostream>
#include "Device.h"
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
#define M_W 50
#define M_V 50
#define VI 1.442249570307408
#define WI 0.7390851332151607

__device__ int mutex=0;
static __global__ void calculeHistogramme(int n,int nTabSM,int* ptrDevResult);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/





static __device__ void ecrasement(int * tabSM,int moitier){

    const int NBR_THREAD = Indice1D::nbThreadBlock();
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    int saut = moitier;
    int s = TID_LOCAL;
    while(s<moitier){
	tabSM[s]+=tabSM[s+saut];
	s+=NBR_THREAD;

    }


}




static __device__ void reduction_intra_block(int * tabSM,int n){
    const int NBR_THREAD = Indice1D::nbThread();

            int moitier = n/2;

            while(moitier>=1){
        	ecrasement(tabSM,moitier);

                moitier/=2;
                __syncthreads;//Faire attention
            }




}

static __device__ void reduction_inter_block(int * tabSM,int * ptrDevResult,int nTabSM){
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    const int NBR_THREAD_LOCAL = Indice1D::nbThreadBlock();
    int s = TID_LOCAL;

    while(s<nTabSM){
	//Lock lock(&mutex);
	//lock.lock();
	//*ptrDevResult+=tabSM[0];
	atomicAdd(&ptrDevResult[s],tabSM[s]);
	s+=NBR_THREAD_LOCAL;
	//lock.unlock();

    }

}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


static __device__ void reduction_intra_thread(int * tabSM,int nTabSM,int * tabEntrees,int nTabEntrees){
    //Executer par thread
       const int NBR_THREAD = Indice2D::nbThread();
       //const int NBR_THREAD_LOCAL = Indice1D::nbThreadBlock();
           const int TID = Indice2D::tid();
          // const int TID_LOCAL = Ind::tidLocalBlock();
           long s = TID;
     //      float sommeThread=0;
           while(s<nTabEntrees){

             //  sommeThread+=prodScalaire(s);
              atomicAdd(&tabSM[tabEntrees[s]],1);

               s+=NBR_THREAD;

           }


           //tabSM[TID_LOCAL]=sommeThread;
           __syncthreads;
}
__device__ static void init_tabSM(int * tabSM,int nTabSM){
    const int NB_THREAD_LOCAL = Indice1D::nbThreadBlock();
    int s = Indice1D::tidLocal();
    while(s<nTabSM){
	 tabSM[s]=0;//TID local
	 s+=NB_THREAD_LOCAL;
    }

    __syncthreads;


}

static __global__ void calculeHistogramme(int nEntrees, int nTabSM,int* ptrDevResult,int * tabEntrees){
   //Une instance par bloc
     extern __shared__ int tabSM[];

    init_tabSM(tabSM,nTabSM);
    reduction_intra_thread(tabSM,nTabSM,tabEntrees,nEntrees);
    //reduction_intra_block(tabSM,nTabSM);
    reduction_inter_block(tabSM,ptrDevResult,nTabSM);


}


 void HistogrammeDevice::runHistogramme(int nEntrees,int  nTabSM,int * ptrDevResult,int * tabEntrees,dim3 dg,dim3 db){
     size_t size = nTabSM*sizeof(int);
     calculeHistogramme<<<dg,db,size>>>(nEntrees,nTabSM,ptrDevResult,tabEntrees);//asynchronous
     Device::checkKernelError("calculeHistogramme");

}

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

