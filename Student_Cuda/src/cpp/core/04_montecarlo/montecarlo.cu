#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include "cudaTools.h"
#include "hiprand/hiprand_kernel.h"
#include "Indice1D.h"
#include "Device.h"
#include "reduction.h"

// #define DEBUG 1

__device__ float uniform(const float MIN, const float MAX, hiprandState& localState);
__global__ void setup_kernel_rand(hiprandState* tabGeneratorThread, const int DEVICE_ID);
__device__ float f(float x);
__global__ void kernel(float* ptrDevGM, const int nTabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long long N,
        hiprandState* tabGeneratorThread);
__device__ void reduceIntraThread(float* tabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long long N, hiprandState* tabGeneratorThread);
__host__ bool useMontecarlo();

/*---------------------------------------------------------------------------*
 * Nombres aléatoires
 *---------------------------------------------------------------------------*/

__device__ float uniform(const float MIN, const float MAX, hiprandState& localState) {
	float r = hiprand_uniform(&localState);
	return MIN + r * (MAX - MIN);
}

__global__ void setup_kernel_rand(hiprandState* tabGeneratorThread, const int DEVICE_ID) {
	const int TID = Indice1D::tid();

	int deltaSeed = DEVICE_ID * INT_MAX;
	int deltaSequence = DEVICE_ID * 100;
	int deltaOffset = DEVICE_ID * 100;

	int seed = 1234 + deltaSeed;
	int sequenceNumber = TID + deltaSequence;
	int offset = deltaOffset;
	hiprand_init(seed, sequenceNumber, offset, &tabGeneratorThread[TID]);
}

/*---------------------------------------------------------------------------*
 * Montecarlo
 *---------------------------------------------------------------------------*/

__device__ float f(float x) {
	return 0.3;
}

__global__ void kernel(float* ptrDevGM, const int nTabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long long N,
        hiprandState* tabGeneratorThread) {
	// @formatter:off
	extern __shared__ float tabSM[];// 1 instance per block !
	// @formatter:on

	// TID
	const int TID = Indice1D::tid();

	// Init
	initTabSM(tabSM, nTabSM, 0);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After init");
#endif

	__syncthreads(); // TODO useless?

	// reduceIntraThread
	reduceIntraThread(tabSM, X_MIN, X_MAX, M, N, tabGeneratorThread);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After reduceIntraThread");
#endif

	__syncthreads(); // TODO is really necessary ??

	// reduceIntraBlock
	reduceIntraBlock(tabSM, nTabSM);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After reduceIntraBlock");
#endif

	__syncthreads(); // TODO is really necessary ??

	// reduceInterBlock
	reduceInterBlock(tabSM, ptrDevGM);
	if(TID == 0) {
		*ptrDevGM = *ptrDevGM / N * (X_MAX - X_MIN) * M;
	}
#ifdef DEBUG
	debugTabGM(ptrDevGM, 1, "Result");
#endif
}

__device__ void reduceIntraThread(float* tabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long long N,
        hiprandState* tabGeneratorThread) {
	const int TID = Indice1D::tid();
	const int TID_LOCAL = Indice1D::tidLocal();
	const int NB_THREAD = Indice1D::nbThread();

	hiprandState localState = tabGeneratorThread[TID];
	float intraThreadSum = 0;
	int s = TID;
	while (s < N) {
		const float X_ALEA = uniform(X_MIN, X_MAX, localState);
		const float Y_ALEA = uniform(0, M, localState);
		if (Y_ALEA < f(X_ALEA)) {
			intraThreadSum++;
		}

		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = intraThreadSum;
}

/*---------------------------------------------------------------------------*
 * Lancement
 *---------------------------------------------------------------------------*/

__host__ bool useMontecarlo() {

	// Paramètres de l'algorithme
	const unsigned long long N = 5000000;
	const float X_MIN = 0;
	const float X_MAX = 2;
	const int M = 1;

	// résultat final en RAM
	float resultRAM = 0;

	// résultat final sur le GPU après réduction
	float* ptrDevGM = NULL;
	size_t size = sizeof(float);
	int nThreadPerBlock = 4;
	size_t sizeTabSM = sizeof(float) * nThreadPerBlock;
	HANDLE_ERROR(hipMalloc(&ptrDevGM, size));
	HANDLE_ERROR(hipMemset(ptrDevGM, 0, size));

	// Paramètres du GPU
	dim3 dg(1, 1, 1); // TODO to optimize
	dim3 db(nThreadPerBlock, 1, 1); // TODO to optimize
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
	const int DEVICE_ID = 1;

	// générateurs de nombres aléatoires sur le GPU
	hiprandState* ptrDevTabGeneratorThread = NULL;
	size_t sizeTabGeneratorThread = sizeof(hiprandState) * nThreadPerBlock;
	HANDLE_ERROR(hipMalloc(&ptrDevTabGeneratorThread, sizeTabGeneratorThread));
	HANDLE_ERROR(hipMemset(ptrDevTabGeneratorThread, 0, sizeTabGeneratorThread));

	// Lancement du kernel 1: initialisation des générateurs de nombres aléatoires
	setup_kernel_rand<<<dg,db,sizeTabSM>>>(ptrDevTabGeneratorThread, DEVICE_ID);
	Device::checkKernelError("Kernel error: setup_kernel_rand");

	// Lancement du kernel 2: calcul de montecarlo
	kernel<<<dg,db,sizeTabSM>>>(ptrDevGM, sizeTabSM, X_MIN, X_MAX, M, N, ptrDevTabGeneratorThread);
	Device::checkKernelError("Kernel error: montecarlo");

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(&resultRAM, ptrDevGM, size, hipMemcpyDeviceToHost)); // barrière de synchronisation

	// Affichage du résultat
	printf("Montecarlo[n=%d, min=%f, max=%f, m=%d] = %f\n", N, X_MIN, X_MAX, M, resultRAM);

	return true;
}

