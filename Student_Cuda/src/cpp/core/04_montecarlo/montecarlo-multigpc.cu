#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include "cudaTools.h"
#include "hiprand/hiprand_kernel.h"
#include "Indice1D.h"
#include "Device.h"
#include "reduction.h"

// #define DEBUG 1

static __device__ float uniform(const float MIN, const float MAX, hiprandState& localState);
static __global__ void setup_kernel_rand(hiprandState* tabGeneratorThread, const int DEVICE_ID);
static __device__ float f(float x);
static __global__ void kernel(float* ptrDevGM, const int nTabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long N,
        hiprandState* tabGeneratorThread);
static __device__ void reduceIntraThread(float* tabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long N, hiprandState* tabGeneratorThread);
__host__ bool useMontecarloMultiGPU();

/*---------------------------------------------------------------------------*
 * Nombres al������atoires
 *---------------------------------------------------------------------------*/

static __device__ float uniform(const float MIN, const float MAX, hiprandState& localState) {
	float r = hiprand_uniform(&localState);
	return MIN + r * (MAX - MIN);
}

static __global__ void setup_kernel_rand(hiprandState* tabGeneratorThread, const int DEVICE_ID) {
	const int TID = Indice1D::tid();

	int deltaSeed = DEVICE_ID * INT_MAX;
	int deltaSequence = DEVICE_ID * 100;
	int deltaOffset = DEVICE_ID * 100;

	int seed = 1234 + deltaSeed;
	int sequenceNumber = TID + deltaSequence;
	int offset = deltaOffset;
	hiprand_init(seed, sequenceNumber, offset, &tabGeneratorThread[TID]);
}

/*---------------------------------------------------------------------------*
 * Montecarlo
 *---------------------------------------------------------------------------*/

static __device__ float f(float x) {
	return 0.3;
}

static __global__ void kernel(float* ptrDevGM, const int nTabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long N,
        hiprandState* tabGeneratorThread) {
	// @formatter:off
	extern __shared__ float tabSM[];// 1 instance per block !
	// @formatter:on

	// TID
	const int TID = Indice1D::tid();

	// Init
	initTabSM(tabSM, nTabSM, 0);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After init");
#endif

	__syncthreads(); // TODO useless?

	// reduceIntraThread
	reduceIntraThread(tabSM, X_MIN, X_MAX, M, N, tabGeneratorThread);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After reduceIntraThread");
#endif

	__syncthreads(); // TODO is really necessary ??

	// reduceIntraBlock
	reduceIntraBlock(tabSM, nTabSM);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After reduceIntraBlock");
#endif

	__syncthreads(); // TODO is really necessary ??

	// reduceInterBlock
	reduceInterBlock(tabSM, ptrDevGM);
#ifdef DEBUG
	debugTabGM(ptrDevGM, 1, "Result");
#endif
}

static __device__ void reduceIntraThread(float* tabSM, const float X_MIN, const float X_MAX, const int M, const unsigned long N,
        hiprandState* tabGeneratorThread) {
	const int TID = Indice1D::tid();
	const int TID_LOCAL = Indice1D::tidLocal();
	const int NB_THREAD = Indice1D::nbThread();

	hiprandState localState = tabGeneratorThread[TID];
	float intraThreadSum = 0;
	int s = TID;
	while (s < N) {
		const float X_ALEA = uniform(X_MIN, X_MAX, localState);
		const float Y_ALEA = uniform(0, M, localState);
		if (Y_ALEA < f(X_ALEA)) {
			intraThreadSum++;
		}

		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = intraThreadSum;
}

/*---------------------------------------------------------------------------*
 * Lancement
 *---------------------------------------------------------------------------*/

static __host__ float runOnDevice(const uint DEVICE, const unsigned long N, const float X_MIN, const float X_MAX, const int M, const uint N_THREAD_PER_BLOCK) {
	// Use GPU
	hipSetDevice(DEVICE);

	// r��sultat final en RAM
	float resultRAM = 0;

	// r��sultat final sur le GPU apr��s r��duction
	float* ptrDevGM = NULL;
	size_t size = sizeof(float);
	size_t sizeTabSM = sizeof(float) * N_THREAD_PER_BLOCK;
	HANDLE_ERROR(hipMalloc(&ptrDevGM, size));
	HANDLE_ERROR(hipMemset(ptrDevGM, 0, size));

	// Param��tres du GPU
	dim3 dg(1, 1, 1); // TODO to optimize
	dim3 db(N_THREAD_PER_BLOCK, 1, 1); // TODO to optimize
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// G��n��rateurs de nombres al��atoires sur le GPU
	hiprandState* ptrDevTabGeneratorThread = NULL;
	size_t sizeTabGeneratorThread = sizeof(hiprandState) * N_THREAD_PER_BLOCK;
	HANDLE_ERROR(hipMalloc(&ptrDevTabGeneratorThread, sizeTabGeneratorThread));
	HANDLE_ERROR(hipMemset(ptrDevTabGeneratorThread, 0, sizeTabGeneratorThread));

	// Lancement du kernel 1: initialisation des g��n��rateurs de nombres al��atoires
	setup_kernel_rand<<<dg,db,sizeTabSM>>>(ptrDevTabGeneratorThread, DEVICE);
	Device::checkKernelError("Kernel error: setup_kernel_rand");
	Device::synchronize();

	// Lancement du kernel 2: calcul de montecarlo
	kernel<<<dg,db,sizeTabSM>>>(ptrDevGM, sizeTabSM, X_MIN, X_MAX, M, N, ptrDevTabGeneratorThread);
	Device::checkKernelError("Kernel error: montecarlo");
	Device::synchronize();

	// R��cup��ration du r��sultat
	HANDLE_ERROR(hipMemcpy(&resultRAM, ptrDevGM, size, hipMemcpyDeviceToHost)); // barri��re de synchronisation

	// Lib��ration de la m��moire
	hipFree(ptrDevGM);

	return resultRAM;
}

__host__ bool useMontecarloMultiGPU() {
	// Parametres de l'algorithme
	const uint NB_GPU = Device::getDeviceCount();
	unsigned long N = 5000000;
	const unsigned long N_PER_GPU = N / NB_GPU;
	const float X_MIN = 0;
	const float X_MAX = 2;
	const int M = 1;
	const uint N_THREAD_PER_BLOCK = 1;

	double sum = 0;

#pragma omp parallel for reduction(+:sum)
	for(uint device = 0; device < NB_GPU; device++) {
		sum += runOnDevice(device, N_PER_GPU, X_MIN, X_MAX, M, N_THREAD_PER_BLOCK);
	}

	// Moyenne
	sum = sum / N * (X_MAX - X_MIN) * M;

	// Affichage du r��sultat
	printf("Montecarlo multigpu[n=%lu, min=%f, max=%f, m=%d] = %f\n", N, X_MIN, X_MAX, M, sum);

	return true;
}

