#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdio>
#include <hiprand/hiprand_kernel.h>

#include "cuda_utils.h"
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"

#include <omp.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isMonteCarloMGPU_Ok(long n, float xMin, float xMax, int m);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ static float computeMonteCarloMGPU(int n);
__global__ static void computeMonteCarloGPUMGPU(float* ptrDevGM, int nTabSM, float xMin, float xMax,
	int m, long n, hiprandState* ptrDevRandom);
__device__ static void reduceIntraThreadMonteCarloMGPU(float* tabSM, float xMin, float xMax, int m, long n,
	hiprandState* generator);
__device__ static float uniformMGPU( float min, float max, hiprandState &localState);
__global__ static void setup_kernel_randMGPU(hiprandState* tabGeneratorThread,int deviceId);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isMonteCarloMGPU_Ok(long n, float xMin, float xMax, int m)
    {
    const int nbGPU = 4;
    const int newN = n/nbGPU;
    float* resultHost = new float[nbGPU];

    size_t size = sizeof(float);
    const int nbThreadBlock = 16;
    size_t sizeSM = sizeof(float) * nbThreadBlock;



    dim3 dg(1, 1, 1);
    dim3 db(nbThreadBlock, 1, 1);


    omp_set_num_threads(nbGPU);
    // use OMP to simplify monogpu to multigpu dev
#pragma omp parralel for
    for(int i=0;i<nbGPU;i++)
	{
	hipSetDevice(i);

	// move to inside for loop because array is bound to device ID #i
	float* ptrDevResult = NULL;
	HANDLE_ERROR(hipMalloc(&ptrDevResult, size));
	HANDLE_ERROR(hipMemset(ptrDevResult, 0, size));
	//init random generator
	hiprandState* ptrDevRandomGen;
	size_t sizeRandom = sizeof(hiprandState) * nbThreadBlock;
	HANDLE_ERROR(hipMalloc((void**) &ptrDevRandomGen, sizeRandom));
	HANDLE_ERROR(hipMemset(ptrDevRandomGen, 0, sizeRandom));

	setup_kernel_randMGPU<<<dg,db>>>(ptrDevRandomGen,Device::getDeviceId());

	//Device::checkKernelError("setup_kernel_rand");

	// carefull using int not size_t
	computeMonteCarloGPUMGPU<<<dg,db,sizeSM>>>(ptrDevResult, nbThreadBlock, xMin, xMax, m, newN, ptrDevRandomGen);

	//Device::checkKernelError("computeMonteCarloGPU");

	HANDLE_ERROR(hipMemcpy(&resultHost[i], ptrDevResult, size, hipMemcpyDeviceToHost));
	// no need of cudaDevice because hipMemcpy

	}
    float resultMGPU = 0.0;
    for(int i=0;i<nbGPU;i++)//could be more generic
	{
	resultMGPU += resultHost[i];
	}
    resultMGPU /= nbGPU;

    printf("Montecarlo[n: %ld, min: %f, max: %f, m: %d] : %f\n", n, xMin, xMax, m, resultMGPU);
    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ float computeMonteCarloMGPU(int n)
    {
    return 0.0;
    }

__global__ void computeMonteCarloGPUMGPU(float* ptrDevResult, int sizeTabSM, float xMin, float xMax,
	int m, long n, hiprandState* ptrDevRandom)
    {
    extern __shared__ float tabSM[];
    const int TID = Indice1D::tid();
    CUDAUtils::initialize(tabSM, sizeTabSM);
    __syncthreads();
    reduceIntraThreadMonteCarloMGPU(tabSM, xMin, xMax, m, n, ptrDevRandom);
    __syncthreads();
    CUDAUtils::reduceIntraBlock(tabSM);
    __syncthreads();
    CUDAUtils::reduceInterBlock(tabSM,ptrDevResult);
    if(TID == 0)
	*ptrDevResult = *ptrDevResult / n * (xMax - xMin) * m;
    }

__device__ float uniformMGPU(const float MIN, const float MAX, hiprandState& localState)
    {
    float r = hiprand_uniform(&localState);
    return MIN + r * (MAX - MIN);
    }

__device__ void reduceIntraThreadMonteCarloMGPU(float* tabSM, float xMin, float xMax, int m, long n,
	hiprandState* generator)
    {
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();
    const int NB_THREAD = Indice1D::nbThread();
    const float probability = 0.3;
    hiprandState state = generator[TID];
    float intraThreadSum = 0;
    int s = TID;
    while (s < n)
	{
	float xAlea = uniformMGPU(xMin, xMax, state);
	float yAlea = uniformMGPU(0, m, state);
	if (yAlea < probability)
	    intraThreadSum++;
	s += NB_THREAD;
	}
    tabSM[TID_LOCAL] = intraThreadSum;
    }

//Nvidia Doc : Each thread gets same seed , a different sequence number , no offset
__global__ void setup_kernel_randMGPU(hiprandState* tabGeneratorThread,int deviceId)
    {
    int tid = Indice1D::tid();
    //Customisation du generator: Proposition (au lecteur de faire mieux)
    // Contrainte : Doit etre différent d'un GPU à l'autre
    int deltaSeed=deviceId* INT_MAX;
    int deltaSequence=deviceId *100;
    int deltaOffset=deviceId *100;
    int seed=1234+deltaSeed;
    int sequenceNumber=tid+deltaSequence;
    int offset=deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabGeneratorThread[tid]);
    }



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

