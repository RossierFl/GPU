#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include <assert.h>
#include <math.h>
#include "reduction.h"

using std::cout;
using std::endl;

#define M_W 100
#define M_V 100
#define VI 1.4422495703074083017725115
#define WI 0.7390851332151606722931092

__host__ double theoricalResult(uint n);
__global__ void scalarProduct(const uint N, float* ptrDevResultGM, const uint nTabSM);
__device__ void reduceIntraThread(float* tabSM, uint n);
__device__ float v(int i);
__device__ float w(int i);
__host__ bool useScalarProduct();

__host__ double theoricalResult(uint n) {
	n--;
	return (n / (double) 2) * (n + 1);
}

__global__ void scalarProduct(const uint N, float* ptrDevResultGM, const uint N_TAB_SM) {
	/* Shared memory */
	extern __shared__ float tabSM[];

	const uint TID_LOCAL = Indice1D::tidLocalBlock();
	initTabSM(tabSM, N_TAB_SM, 0);

	// reduceIntraThread
	reduceIntraThread(tabSM, N_TAB_SM);

	__syncthreads(); // TODO is really necessary ??

	// reduceIntraBlock
	reduceIntraBlock(tabSM, N_TAB_SM);

	// reduceIntraBlock
	reduceInterBlock(tabSM, ptrDevResultGM);
}

__device__ void reduceIntraThread(float* tabSM, uint n) {
	const uint NB_THREAD = Indice1D::nbThread();
	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocalBlock();

	uint s = TID;
	float sum = 0;
	while (s < n) {
		sum += v(s) * w(s);
		s += NB_THREAD;
	}

	tabSM[TID_LOCAL] = sum;
}

__device__ float v(int i) {
	float x = 1.5 + abs(cos((float) i));
	for (int j = 1; j <= M_V; j++) {
		float xCarre = x * x;
		x = x - (xCarre * x - 3) / (3 * xCarre);
	}
	return (x / VI) * sqrt((float) i);
}

__device__ float w(int i) {
	float x = abs(cos((float) i));
	for (int j = 1; j <= M_W; j++) {
		x = x - (cos(x) - x) / (-sin(x) - 1);
	}
	return (x / WI) * sqrt((float) i);
}

__host__ bool useScalarProduct() {
	const uint N = 100000;
	printf("[Scalar Product]\n");

	// Paramètres du GPU
	const uint NB_THREAD = 16;
	assert(NB_THREAD % 2 == 0);
	dim3 dg = dim3(NB_THREAD, 1, 1);
	dim3 db = dim3(32, 1, 1);
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);

	/* Result on CPU */
	float scalarProductResult = 0;

	/* Result on GPU */
	float* ptrScalarProductResultDevGRAM;
	HANDLE_ERROR(hipMalloc(&ptrScalarProductResultDevGRAM, sizeof(float)));
	HANDLE_ERROR(hipMemset(ptrScalarProductResultDevGRAM, 0, sizeof(float)));

	/* Launch kernel */
	size_t sizeTabSMByte = sizeof(float) * NB_THREAD;
	scalarProduct<<<dg,db,sizeTabSMByte>>>(N, ptrScalarProductResultDevGRAM, NB_THREAD);
	Device::checkKernelError("Kernel error: scalarProduct");
	Device::synchronize(); // Display printf

	/* Fetch result */
	HANDLE_ERROR(hipMemcpy(&scalarProductResult, ptrScalarProductResultDevGRAM, sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(ptrScalarProductResultDevGRAM));
	printf("Result GPU = %f - CPU = %f\n", scalarProductResult, theoricalResult(N));

	return true;
}

