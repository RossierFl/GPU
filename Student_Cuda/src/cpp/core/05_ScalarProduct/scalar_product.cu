#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include <assert.h>
#include <math.h>
#include "reduction.h"

#define DEBUG 1

#define M_W 100
#define M_V 100
#define VI 1.4422495703074083017725115
#define WI 0.7390851332151606722931092

__host__ double theoricalResult(uint n);
__global__ void scalarProduct(const uint N, double* ptrDevResultGM, const uint nTabSM);
__device__ void reduceIntraThread(double* tabSM, uint n);
__device__ double v(int i);
__device__ double w(int i);
__host__ bool useScalarProduct();

__host__ double theoricalResult(uint n) {
	n--;
	return (n / 2.0) * (n + 1);
}

__global__ void scalarProduct(const uint N, double* ptrDevResultGM, const uint N_TAB_SM) {
	/* Shared memory */
	extern __shared__ double tabSM[];

	const uint TID_LOCAL = Indice1D::tidLocalBlock();
	initTabSM(tabSM, N_TAB_SM, 0);

	// reduceIntraThread
	reduceIntraThread(tabSM, N_TAB_SM);

	__syncthreads(); // TODO is really necessary ??

#ifdef DEBUG
	if(TID_LOCAL == 0)
	debugTabSM(tabSM, N_TAB_SM, "after intra thread");
#endif

	// reduceIntraBlock
	reduceIntraBlock(tabSM, N_TAB_SM);

	__syncthreads(); // TODO is really necessary ??

	// reduceIntraBlock
	reduceInterBlock(tabSM, ptrDevResultGM);
}

__device__ void reduceIntraThread(double* tabSM, uint n) {
	const uint NB_THREAD = Indice1D::nbThread();
	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocalBlock();

	uint s = TID;
	double sum = 0;
	while (s < n) {
		sum += v(s) * w(s);
		s += NB_THREAD;
	}

	tabSM[TID_LOCAL] = sum;
}

__device__ double v(int i) {
	double x = 1.5 + abs(cos((double) i));
	for (int j = 1; j <= M_V; j++) {
		double xCarre = x * x;
		x = x - (xCarre * x - 3) / (3 * xCarre);
	}
	return (x / VI) * sqrt((double) i);
}

__device__ double w(int i) {
	double x = abs(cos((double) i));
	for (int j = 1; j <= M_W; j++) {
		x = x - (cos(x) - x) / (-sin(x) - 1);
	}
	return (x / WI) * sqrt((double) i);
}

__host__ bool useScalarProduct() {
	const uint N = 100000;
	printf("[Scalar Product]\n");

	// Paramètres du GPU
	const uint NB_THREAD = 32;
	assert(NB_THREAD % 2 == 0);
	dim3 dg = dim3(16, 1, 1);
	dim3 db = dim3(NB_THREAD, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	/* Result on CPU */
	double scalarProductResult = 0;

	/* Result on GPU */
	double* ptrScalarProductResultDevGRAM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrScalarProductResultDevGRAM, sizeof(double)));
	HANDLE_ERROR(hipMemset(ptrScalarProductResultDevGRAM, 0, sizeof(double)));

	/* Launch kernel */
	size_t sizeTabSMByte = sizeof(double) * NB_THREAD;
	scalarProduct<<<dg,db,sizeTabSMByte>>>(N, ptrScalarProductResultDevGRAM, NB_THREAD);
	Device::checkKernelError("Kernel error: scalarProduct");
	Device::synchronize(); // Display printf

	/* Fetch result */
	HANDLE_ERROR(hipMemcpy(&scalarProductResult, ptrScalarProductResultDevGRAM, sizeof(double), hipMemcpyDeviceToHost));
	printf("Result GPU = %f - CPU = %f\n", scalarProductResult, theoricalResult(N));

	/* Free memory */
	HANDLE_ERROR(hipFree(ptrScalarProductResultDevGRAM));

	return true;
}

