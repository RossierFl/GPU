#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include <assert.h>
#include <math.h>
#include "reduction.h"

#define DEBUG 1

#define M_W 50
#define M_V 50
#define VI 1.4422495703074083017725115
#define WI 0.7390851332151606722931092

__host__ double checkScalarProduct(uint n);
__global__ void scalarProduct(const uint N, double* ptrDevResultGM, const uint nTabSM);
__device__ void reduceIntraThread(double* tabSM, uint n);
__device__ double v(int i);
__device__ double w(int i);
__host__ bool useScalarProduct();

__host__ double checkScalarProduct(uint n) {
	n--;
	return (n / 2.0) * (n + 1);
}

__global__ void scalarProduct(const uint N, double* ptrDevResultGM, const uint N_TAB_SM) {
	/* Shared memory */
	extern __shared__ double tabSM[];

	const uint TID_LOCAL = Indice1D::tidLocalBlock();
	initTabSM(tabSM, N_TAB_SM, 0);

	// reduceIntraThread
	reduceIntraThread(tabSM, N_TAB_SM);

	__syncthreads(); // TODO is really necessary ??

#ifdef DEBUG
	if(TID_LOCAL == 0)
	debugTabSM(tabSM, N_TAB_SM, "after intra thread");
#endif

	// reduceIntraBlock
	reduceIntraBlock(tabSM, N_TAB_SM);

	__syncthreads(); // TODO is really necessary ??

	// reduceIntraBlock
	reduceInterBlock(tabSM, ptrDevResultGM);
}

__device__ void reduceIntraThread(double* tabSM, uint n) {
	const int NB_THREAD = Indice1D::nbThread();
	const int TID = Indice1D::tid();
	const int TID_LOCAL = Indice1D::tidLocalBlock();

	int s = TID;
	double sum = 0;
	while (s < n) {
		sum += v(s) * w(s);
		s += NB_THREAD;
	}

	tabSM[TID_LOCAL] = sum;
}

__device__ double v(int i) {
	double x = 1.5 + abs(cos((double) i));
	for (int j = 1; j <= M_V; j++) {
		double xCarre = x * x;
		x = x - (xCarre * x - 3) / (3 * xCarre);
	}
	return (x / VI) * sqrt((double) i);
}

__device__ double w(int i) {
	double x = abs(cos((double) i));
	for (int j = 1; j <= M_W; j++) {
		x = x - (cos(x) - x) / (-sin(x) - 1);
	}
	return (x / WI) * sqrt((double) i);
}

__host__ bool useScalarProduct() {
	printf("\n[Scalar Product]\n");

	const uint N = 100000;

	// Paramètres du GPU
	const uint NB_THREAD = 32;
	assert(NB_THREAD % 2 == 0);
	dim3 dg = dim3(16, 1, 1);
	dim3 db = dim3(NB_THREAD, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	/* Result on CPU */
	double scalarProductResult = 0;

	/* Result on GPU */
	double* ptrScalarProductResultDevGRAM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrScalarProductResultDevGRAM, sizeof(double)));
	HANDLE_ERROR(hipMemset(ptrScalarProductResultDevGRAM, 0, sizeof(double)));

	/* Launch kernel */
	size_t sizeTabSMByte = sizeof(double) * NB_THREAD;
	scalarProduct<<<dg,db,sizeTabSMByte>>>(N, ptrScalarProductResultDevGRAM, NB_THREAD);
	Device::checkKernelError("Kernel error: scalarProduct");
	Device::synchronize(); // Display printf

	/* Fetch result */
	HANDLE_ERROR(hipMemcpy(&scalarProductResult, ptrScalarProductResultDevGRAM, sizeof(double), hipMemcpyDeviceToHost));
	printf("GPU = %f\n", scalarProductResult);
	printf("CPU = %f\n", checkScalarProduct(N));

	/* Free memory */
	HANDLE_ERROR(hipFree(ptrScalarProductResultDevGRAM));

	return true;
}

