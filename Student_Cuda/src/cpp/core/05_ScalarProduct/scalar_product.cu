#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "reduction.h"
#include <assert.h>
#include <math.h>
#include "MathTools.h"

// #define DEBUG 1

#define M_W 100
#define M_V 100
#define VI 1.4422495703074083017725115
#define WI 0.7390851332151606722931092

__host__ bool useScalarProduct();
static __host__ double theoricalResult(int n);
static __global__ void scalarProduct(const uint N, double* ptrDevResult, const uint NB_THREAD);
static __device__ void reduceIntraThread(double* tabSM, int n);
static __device__ double v(int i);
static __device__ double w(int i);

static __host__ double theoricalResult(int n) {
	n--;
	return (n / (double) 2) * (n + 1);
}

static __global__ void scalarProduct(const uint N, double* ptrDevRes, const uint NB_THREAD) {
	/* Shared memory */
	extern __shared__ double tabSM[];
	initTabSM(tabSM, NB_THREAD, 0);

	// Reduce intra thread
	reduceIntraThread(tabSM, N);

	__syncthreads(); // TODO is really necessary ??

	// Reduce intra block
	reduceIntraBlock(tabSM, NB_THREAD);

	__syncthreads(); // TODO is really necessary ??

	// Reduce inter block
	reduceInterBlock(tabSM, ptrDevRes);
}

static __device__ void reduceIntraThread(double* tabSM, int n) {
	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocalBlock();
	const uint NB_THREAD = Indice1D::nbThread();

	int s = TID;
	double sum = 0.0;
	while (s < n) {
		sum += v(s) * w(s);
		s += NB_THREAD;
	}

	tabSM[TID_LOCAL] = sum;
}

static __device__ double v(int i) {
	const uint TID = Indice1D::tid();
	double x = 1.5 + abs(cos((double) i));

	for (int j = 1; j <= M_V; j++) {
		double xCarre = x * x;
		x = x - (xCarre * x - 3) / (3 * xCarre);
	}

	return (x / VI) * sqrt((double) i);
}

static __device__ double w(int i) {
	const uint TID = Indice1D::tid();
	double x = abs(cos((double) i));

	for (int j = 1; j <= M_W; j++) {
		x = x - (cos(x) - x) / (-sin(x) - 1);
	}

	return (x / WI) * sqrt((double) i);
}

__host__ bool useScalarProduct() {

	printf("\n[Scalar Product]\n");

	const uint N = 50000;

	/* Scalar product in RAM */
	double scalarProductRAM = 0.0;
	double* ptrScalarProductRAM = &scalarProductRAM;

	/* Scalar product in GRAM */
	double* ptrDevScalarProductGRAM = NULL;
	size_t size = sizeof(double);
	HANDLE_ERROR(hipMalloc(&ptrDevScalarProductGRAM, size));
	HANDLE_ERROR(hipMemset(ptrDevScalarProductGRAM, 0, size));

	/* Parameters */
	const uint NB_THREAD = 32;
	assert((NB_THREAD & (NB_THREAD - 1)) == 0); // should be 2^xy
	dim3 dg = dim3(16, 1, 1);
	dim3 db = dim3(NB_THREAD, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	/* Processing */
	size_t sizeTabSM = sizeof(double) * NB_THREAD;
	scalarProduct<<<dg,db,sizeTabSM>>>(N, ptrDevScalarProductGRAM, NB_THREAD); // asynchrone
	Device::checkKernelError("kernel error: scalarProduct");
	Device::synchronize(); // printf

	/* Fetch result */
	HANDLE_ERROR(hipMemcpy(ptrScalarProductRAM, ptrDevScalarProductGRAM, size, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(ptrDevScalarProductGRAM));

	/* Control result */
	double referenceValue = theoricalResult(N);
	printf("Result GPU = %f\n", scalarProductRAM);
	printf("Result CPU = %f\n", referenceValue);
	bool isOk = MathTools::isEquals(scalarProductRAM, referenceValue, 1);
	printf("isOk = %d\n", isOk);

	return isOk;
}
