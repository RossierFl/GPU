#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "Device.h"
#include "Indice1D.h"
#include "reduction.h"
#include "cudaTools.h"

static __global__ void kernel(uint* ptrImageDevGM, size_t sizeImage, uint* ptrHistogrammeDevGM, size_t sizeHistogramme)
{
	const uint TID = Indice1D::tid();
	const uint NB_THREAD = Indice1D::nbThread();

	uint s = TID;
	while (s < sizeImage)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&ptrHistogrammeDevGM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}
}

__host__ void hist_cuda_gm(int* data, int* hist, const uint DATA_SIZE, const int MIN_VALUE, const int MAX_VALUE, const int DG, const int DB)
{
	// Image
	const size_t DATA_SIZE_BYTE = sizeof(int) * DATA_SIZE;
	const uint HIST_SIZE = MAX_VALUE - MIN_VALUE + 1;
	const size_t HIST_SIZE_BYTE = sizeof(int) * HIST_SIZE;

	// Parameters
	uint nThreadPerBlock = DB; // TODO to optimize
	uint nBlockPerMP = DG; // TODO to optimize
	dim3 dg(nBlockPerMP, 1, 1);
	dim3 db(nThreadPerBlock, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// Image en GRAM
	uint* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrImageDevGM, DATA_SIZE_BYTE));
	HANDLE_ERROR(hipMemcpy(ptrImageDevGM, data, DATA_SIZE_BYTE, hipMemcpyHostToDevice));

	// Histogramme en GRAM
	uint* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(int) * (HIST_SIZE);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));

	// call kernel
	kernel<<<dg,db>>>(ptrImageDevGM, DATA_SIZE, ptrHistogrammeDevGM, HIST_SIZE);
	Device::checkKernelError("Kernel error: kernel cuda GM");
	Device::synchronize();

	// R��cup��ration du r��sultat
	HANDLE_ERROR(hipMemcpy(hist, ptrHistogrammeDevGM, HIST_SIZE_BYTE, hipMemcpyDeviceToHost)); // barri��re de synchronisation

	// Lib��ration de la m��moire
	HANDLE_ERROR(hipFree(ptrImageDevGM));
	HANDLE_ERROR(hipFree(ptrHistogrammeDevGM));
}
