#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "Device.h"
#include "Indice1D.h"
#include "reduction.h"
#include "cudaTools.h"

static __global__ void kernel(uint* ptrImageDevGM, size_t sizeImage, uint* ptrHistogrammeDevGM, size_t sizeHistogramme)
{
	const uint TID = Indice1D::tid();
	const uint NB_THREAD = Indice1D::nbThread();

	uint s = TID;
	while (s < sizeImage)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&ptrHistogrammeDevGM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}
}

__host__ void hist_cuda_gm(int* data, int* hist, const uint DATA_SIZE, const int MIN_VALUE, const int MAX_VALUE)
{
	// Image
	const size_t DATA_SIZE_BYTE = sizeof(int) * DATA_SIZE;
	const uint NB_POSSIBLE_VALUE = MAX_VALUE - MIN_VALUE + 1;
	const size_t HIST_SIZE_BYTE = sizeof(int) * NB_POSSIBLE_VALUE;

	// Parameters
	uint nThreadPerBlock = 32; // TODO to optimize
	uint nBlockPerMP = 32; // TODO to optimize
	dim3 dg(nBlockPerMP, 1, 1);
	dim3 db(nThreadPerBlock, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// Image en GRAM
	uint* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrImageDevGM, DATA_SIZE_BYTE));
	HANDLE_ERROR(hipMemcpy(ptrImageDevGM, data, DATA_SIZE_BYTE, hipMemcpyHostToDevice));

	// Histogramme en GRAM
	uint* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(uint) * (NB_POSSIBLE_VALUE);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));

	// call kernel
	kernel<<<dg,db>>>(ptrImageDevGM, DATA_SIZE, ptrHistogrammeDevGM, NB_POSSIBLE_VALUE);
	Device::checkKernelError("Kernel error: kernel cuda GM");
	Device::synchronize();

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(hist, ptrHistogrammeDevGM, HIST_SIZE_BYTE, hipMemcpyDeviceToHost)); // barrière de synchronisation

	// Libération de la mémoire
	HANDLE_ERROR(hipFree(ptrImageDevGM));
	HANDLE_ERROR(hipFree(ptrHistogrammeDevGM));
}
