#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "Device.h"
#include "Indice1D.h"
#include "reduction.h"
#include "cudaTools.h"

static __global__ void kernel(int* ptrImageDevGM, size_t sizeImage, int* ptrHistogrammeDevGM, size_t sizeHistogramme)
{
	// @formatter:off
	extern __shared__ uint tabSM[];// 1 instance per block !
	// @formatter:on

	initTabSM(tabSM, sizeHistogramme, 0);

	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocal();
	const uint NB_THREAD = Indice1D::nbThread();
	const uint NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

	uint s = TID;
	while(s < sizeImage)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&tabSM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}

	// synchronization entre les threads du meme bloc
	__syncthreads();

	// merge local hist with global hist
	s = TID_LOCAL;
	while(s < sizeHistogramme)
	{
		// work >>>>
		atomicAdd(&ptrHistogrammeDevGM[s], tabSM[s]);
		// <<<< end work
		s += NB_THREAD_LOCAL;
	}
}

void run_on_device(int* data, int* hist, const uint DATA_SIZE, const int MIN_VALUE, const int MAX_VALUE, const int DG, const int DB) {
	// Image
	const size_t DATA_SIZE_BYTE = sizeof(int) * DATA_SIZE;
	const uint HIST_SIZE = MAX_VALUE - MIN_VALUE + 1;
	const size_t HIST_SIZE_BYTE = sizeof(int) * HIST_SIZE;

	// Parameters
	uint nThreadPerBlock = DB; // TODO to optimize
	uint nBlockPerMP = DG; // TODO to optimize
	dim3 dg(nBlockPerMP, 1, 1);
	dim3 db(nThreadPerBlock, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// Image en GRAM
	int* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrImageDevGM, DATA_SIZE_BYTE));
	HANDLE_ERROR(hipMemcpy(ptrImageDevGM, data, DATA_SIZE_BYTE, hipMemcpyHostToDevice));

	// Histogramme en GRAM
	int* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(int) * (HIST_SIZE);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));

	// call kernel
	kernel<<<dg,db,HIST_SIZE_BYTE>>>(ptrImageDevGM, DATA_SIZE, ptrHistogrammeDevGM, HIST_SIZE);
	Device::checkKernelError("Kernel error: kernel cuda MultiGPU");
	Device::synchronize();

	// Recuperation du resultat
	HANDLE_ERROR(hipMemcpy(hist, ptrHistogrammeDevGM, HIST_SIZE_BYTE, hipMemcpyDeviceToHost)); // barriere de synchronisation

	// Liberation de la memoire
	HANDLE_ERROR(hipFree(ptrImageDevGM));
	HANDLE_ERROR(hipFree(ptrHistogrammeDevGM));
}

void hist_cuda_multigpu(int* data, int* hist, const uint DATA_SIZE, const int MIN_VALUE, const int MAX_VALUE, const int DG, const int DB) {

	const uint NB_GPU = Device::getDeviceCount();
	const uint HIST_SIZE = MAX_VALUE - MIN_VALUE + 1;
	const uint HIST_SIZE_BYTES = HIST_SIZE * sizeof(int);

	/* Compute steps for each GPU */
	uint remainingH = DATA_SIZE;
	uint start = 0;
	const uint stepH = DATA_SIZE / NB_GPU;
	uint steps[NB_GPU];
	uint starts[NB_GPU];
	for (int device = 0; device < NB_GPU; device++) {
		starts[device] = start;
		if (device == NB_GPU - 1) {
			steps[device] = remainingH;
		} else {
			steps[device] = stepH;
		}
		remainingH -= steps[device];
		start += steps[device];
	}

	/* Processing */
#pragma omp parallel for
	for (int device = 0; device < NB_GPU; device++) {
		// set current device
		hipSetDevice(device);

		int* local_hist = new int[HIST_SIZE];
		memset(local_hist, 0, HIST_SIZE_BYTES);
		run_on_device(data + starts[device], local_hist, steps[device], MIN_VALUE, MAX_VALUE, DG, DB);

#pragma omp critical(section)
		{
			for(int i = 0; i < HIST_SIZE; i++) {
				hist[i] += local_hist[i];
			}
		}
	}

	// reset current device
	hipSetDevice(0);
}
