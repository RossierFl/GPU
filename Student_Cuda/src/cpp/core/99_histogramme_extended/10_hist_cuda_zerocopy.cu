#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "Device.h"
#include "Indice1D.h"
#include "reduction.h"
#include "cudaTools.h"

static __global__ void kernel(int* ptrImageDevGM, size_t sizeImage, int* ptrHistogrammeDevGM, size_t sizeHistogramme)
{
	// @formatter:off
	extern __shared__ uint tabSM[];// 1 instance per block !
	// @formatter:on

	initTabSM(tabSM, sizeHistogramme, 0);

	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocal();
	const uint NB_THREAD = Indice1D::nbThread();
	const uint NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

	uint s = TID;
	while(s < sizeImage)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&tabSM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}

	// synchronization entre les threads du meme bloc
	__syncthreads();

	// merge local hist with global hist
	s = TID_LOCAL;
	while(s < sizeHistogramme)
	{
		// work >>>>
		atomicAdd(&ptrHistogrammeDevGM[s], tabSM[s]);
		// <<<< end work
		s += NB_THREAD_LOCAL;
	}
}

__host__ void hist_cuda_zerocopy(int* data, int* hist, const uint DATA_SIZE, const int MIN_VALUE, const int MAX_VALUE, const int DG, const int DB)
{
	//HANDLE_ERROR(hipSetDevice(0));
	//HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
	//HANDLE_ERROR(hipSetDevice(0));

	// Image
	const size_t DATA_SIZE_BYTE = sizeof(int) * DATA_SIZE;
	const uint HIST_SIZE = MAX_VALUE - MIN_VALUE + 1;
	const size_t HIST_SIZE_BYTE = sizeof(int) * HIST_SIZE;

	// Parameters
	uint nThreadPerBlock = DB; // TODO to optimize
	uint nBlockPerMP = DG; // TODO to optimize
	dim3 dg(nBlockPerMP, 1, 1);
	dim3 db(nThreadPerBlock, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// Image en GRAM
	int* ptrDataZeroCopyMem = NULL;
	HANDLE_ERROR(hipHostAlloc(&ptrDataZeroCopyMem, DATA_SIZE_BYTE, hipHostMallocDefault));
	memcpy(ptrDataZeroCopyMem, data, DATA_SIZE_BYTE);
	int* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipHostGetDevicePointer(&ptrImageDevGM, ptrDataZeroCopyMem, 0));

	// Histogramme en GRAM
	int* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(int) * (HIST_SIZE);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));

	// call kernel
	kernel<<<dg,db,HIST_SIZE_BYTE>>>(ptrImageDevGM, DATA_SIZE, ptrHistogrammeDevGM, HIST_SIZE);
	Device::checkKernelError("Kernel error: kernel cuda ZCMEM");
	Device::synchronize();

	// Recuperation du resultat
	HANDLE_ERROR(hipMemcpy(hist, ptrHistogrammeDevGM, HIST_SIZE_BYTE, hipMemcpyDeviceToHost)); // barri������re de synchronisation

	// Liberation de la memoire
	HANDLE_ERROR(hipHostFree(ptrDataZeroCopyMem));
	//HANDLE_ERROR(hipFree(ptrImageDevGM));
	HANDLE_ERROR(hipFree(ptrHistogrammeDevGM));
}
