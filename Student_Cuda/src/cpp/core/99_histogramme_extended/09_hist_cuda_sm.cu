#include "hip/hip_runtime.h"
#include "AleaTools.h"
#include "Device.h"
#include <stdio.h>
#include "cudaTools.h"
#include "Indice1D.h"
#include "reduction.h"

static __global__ void kernel(uint* ptrImageDevGM, size_t sizeImageByte, uint* ptrHistogrammeDevGM, size_t sizeHistogrammeByte)
{
	// @formatter:off
	extern __shared__ uint tabSM[];// 1 instance per block !
	// @formatter:on

	initTabSM(tabSM, NB_POSSIBLE_VALUE, 0);

	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocal();
	const uint NB_THREAD = Indice1D::nbThread();
	const uint NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

	uint s = TID;
	while(s < SIZE)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&tabSM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}

	// synchronization entre les threads du meme bloc
	__syncthreads();

	// merge local hist with global hist
	s = TID_LOCAL;
	while(s < NB_POSSIBLE_VALUE)
	{
		// work >>>>
		atomicAdd(&ptrHistogrammeDevGM[s], tabSM[s]);
		// <<<< end work
		s += NB_THREAD_LOCAL;
	}
}

__host__ void hist_cuda_sm(int* data, int* hist, const uint DATA_SIZE, const int MIN_VALUE, const int MAX_VALUE)
{
	printf("\n[Histogramme cuda SM]\n");

	// Image
	const size_t DATA_SIZE_BYTE = sizeof(int) * DATA_SIZE;
	const uint NB_POSSIBLE_VALUE = MAX_VALUE - MIN_VALUE + 1;
	const size_t HIST_SIZE_BYTE = sizeof(int) * NB_POSSIBLE_VALUE;

	// Parameters
	uint nThreadPerBlock = 16; // TODO to optimize
	uint nBlockPerMP = 32; // TODO to optimize
	dim3 dg(nBlockPerMP, 1, 1);
	dim3 db(nThreadPerBlock, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// Image en GRAM
	uint* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrImageDevGM, DATA_SIZE_BYTE));
	HANDLE_ERROR(hipMemcpy(ptrImageDevGM, data, DATA_SIZE_BYTE, hipMemcpyHostToDevice));

	// Histogramme en GRAM
	uint* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(uint) * (NB_POSSIBLE_VALUE);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));

	// call kernel
	kernel<<<dg,db,HIST_SIZE_BYTE>>>(ptrImageDevGM, DATA_SIZE, ptrHistogrammeDevGM, NB_POSSIBLE_VALUE);
	Device::checkKernelError("Kernel error: kernel cuda SM");
	Device::synchronize();

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(hist, ptrHistogrammeDevGM, HIST_SIZE_BYTE, hipMemcpyDeviceToHost)); // barrière de synchronisation

	// Libération de la mémoire
	HANDLE_ERROR(hipFree(ptrImageDevGM));
	HANDLE_ERROR(hipFree(ptrHistogrammeDevGM));
}