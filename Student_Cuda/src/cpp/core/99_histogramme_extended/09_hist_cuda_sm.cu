#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "Device.h"
#include "Indice1D.h"
#include "reduction.h"
#include "cudaTools.h"

static __global__ void kernel(uint* ptrImageDevGM, size_t sizeImage, uint* ptrHistogrammeDevGM, size_t sizeHistogramme)
{
	// @formatter:off
	extern __shared__ uint tabSM[];// 1 instance per block !
	// @formatter:on

	initTabSM(tabSM, sizeHistogramme, 0);

	const uint TID = Indice1D::tid();
	const uint TID_LOCAL = Indice1D::tidLocal();
	const uint NB_THREAD = Indice1D::nbThread();
	const uint NB_THREAD_LOCAL = Indice1D::nbThreadBlock();

	uint s = TID;
	while(s < sizeImage)
	{
		// work >>>>
		uint value = ptrImageDevGM[s];
		atomicAdd(&tabSM[value], 1);
		// <<<< end work

		s += NB_THREAD;
	}

	// synchronization entre les threads du meme bloc
	__syncthreads();

	// merge local hist with global hist
	s = TID_LOCAL;
	while(s < sizeHistogramme)
	{
		// work >>>>
		atomicAdd(&ptrHistogrammeDevGM[s], tabSM[s]);
		// <<<< end work
		s += NB_THREAD_LOCAL;
	}
}

__host__ void hist_cuda_sm(int* data, int* hist, const uint DATA_SIZE, const int MIN_VALUE, const int MAX_VALUE, const int DG, const int DB)
{
	// Image
	const size_t DATA_SIZE_BYTE = sizeof(int) * DATA_SIZE;
	const uint HIST_SIZE = MAX_VALUE - MIN_VALUE + 1;
	const size_t HIST_SIZE_BYTE = sizeof(int) * HIST_SIZE;

	// Parameters
	uint nThreadPerBlock = DB; // TODO to optimize
	uint nBlockPerMP = DG; // TODO to optimize
	dim3 dg(nBlockPerMP, 1, 1);
	dim3 db(nThreadPerBlock, 1, 1);
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif

	// Image en GRAM
	uint* ptrImageDevGM = NULL;
	HANDLE_ERROR(hipMalloc(&ptrImageDevGM, DATA_SIZE_BYTE));
	HANDLE_ERROR(hipMemcpy(ptrImageDevGM, data, DATA_SIZE_BYTE, hipMemcpyHostToDevice));

	// Histogramme en GRAM
	uint* ptrHistogrammeDevGM = NULL;
	size_t sizeHistogramme = sizeof(int) * (HIST_SIZE);
	HANDLE_ERROR(hipMalloc(&ptrHistogrammeDevGM, sizeHistogramme));
	HANDLE_ERROR(hipMemset(ptrHistogrammeDevGM, 0, sizeHistogramme));

	// call kernel
	kernel<<<dg,db,HIST_SIZE_BYTE>>>(ptrImageDevGM, DATA_SIZE, ptrHistogrammeDevGM, HIST_SIZE);
	Device::checkKernelError("Kernel error: kernel cuda SM");
	Device::synchronize();

	// R��cup��ration du r��sultat
	HANDLE_ERROR(hipMemcpy(hist, ptrHistogrammeDevGM, HIST_SIZE_BYTE, hipMemcpyDeviceToHost)); // barri��re de synchronisation

	// Lib��ration de la m��moire
	HANDLE_ERROR(hipFree(ptrImageDevGM));
	HANDLE_ERROR(hipFree(ptrHistogrammeDevGM));
}
