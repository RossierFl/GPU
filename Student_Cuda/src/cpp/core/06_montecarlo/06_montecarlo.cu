#include "hip/hip_runtime.h"
//#include <curandTools.h>
#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include <assert.h>
#include <math.h>
#include <limits.h>

#include <hiprand/hiprand_kernel.h>
#include "UtilsCuda.h"

__device__ float uniform( float min, float max, hiprandState& localState);
__device__ float f(float x);
__global__ void computeMontecarloGPU(float* ptrDevGM, int nTabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom);
__device__ void reduce_intra_thread_montecarlo(float* tabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom);
__host__ bool isMontecarlo_Ok(long n, float xMin, float xMax, int m);
__global__ void setup_kernel_rand_montecarlo(hiprandState* ptrDevTabGeneratorThread, int deviceId);


/*---------------------------------------------------------------------------*
 * Lancement
 *---------------------------------------------------------------------------*/

__host__ bool isMontecarlo_Ok(long n, float xMin, float xMax, int m) {
	printf("Montecarlo[n=%d, min=%f, max=%f, m=%d]\n", n, xMin, xMax, m);
	float resultHost = 0;
	float* ptrDevGM = NULL;
	size_t size = sizeof(float);
	int nThreadPerBlock = 4;
	size_t sizeTabSM = sizeof(float) * nThreadPerBlock;
	HANDLE_ERROR(hipMalloc(&ptrDevGM, size));
	HANDLE_ERROR(hipMemset(ptrDevGM, 0, size));

	dim3 dg(1, 1, 1); // TODO to optimize
	dim3 db(nThreadPerBlock, 1, 1); // TODO to optimize
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);

	//init random generator
	hiprandState* ptrDevRandom;
	size_t sizeRandom = sizeof(hiprandState) * nThreadPerBlock;
	HANDLE_ERROR(hipMalloc((void**) &ptrDevRandom, sizeRandom));
	HANDLE_ERROR(hipMemset(ptrDevRandom, 0, sizeRandom));
	setup_kernel_rand_montecarlo<<<dg,db>>>(ptrDevRandom,Device::getDeviceId());
	Device::checkKernelError("setup_kernel_rand");

	computeMontecarloGPU<<<dg,db,sizeTabSM>>>(ptrDevGM, sizeTabSM, xMin, xMax, m, n, ptrDevRandom);
	Device::checkKernelError("computeMontecarloGPU");

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(&resultHost, ptrDevGM, size, hipMemcpyDeviceToHost)); // barrière de synchronisation

	// Affichage du résultat
	printf("Montecarlo[n=%d, min=%f, max=%f, m=%d] = %f\n", n, xMin, xMax, m, resultHost);

	return true;
}



__device__ float uniform(const float MIN, const float MAX, hiprandState& localState) {
	float r = hiprand_uniform(&localState);
	return MIN + r * (MAX - MIN);
}

/*---------------------------------------------------------------------------*
 * Montecarlo
 *---------------------------------------------------------------------------*/

__device__ float f(float x) {
	return 0.3;
}

__global__ void computeMontecarloGPU(float* ptrDevGM, int nTabSM, float xMin, float xMax, int m, long n, hiprandState* ptrDevRandom){
	extern __shared__ float tabSM[];// 1 instance per block !
	const int TID = Indice1D::tid();
	UtilsCUDA::initialise(tabSM, nTabSM);

	__syncthreads();
	reduce_intra_thread_montecarlo(tabSM, xMin, xMax, m, n, ptrDevRandom);
	__syncthreads();
	UtilsCUDA::reduce_intra_block(tabSM);
	__syncthreads();
	UtilsCUDA::reduce_inter_block(tabSM, ptrDevGM);
	if(TID == 0) *ptrDevGM = *ptrDevGM / n * (xMax - xMin) * m;
}

__device__ void reduce_intra_thread_montecarlo(float* tabSM, float xMin, float xMax, int m, long n,
        hiprandState* tabGeneratorThread) {
	const int TID = Indice1D::tid();
	const int TID_LOCAL = Indice1D::tidLocal();
	const int NB_THREAD = Indice1D::nbThread();

	hiprandState localState = tabGeneratorThread[TID];
	float intraThreadSum = 0;
	int s = TID;
	while (s < n) {
		float xAlea = uniform(xMin, xMax, localState);
		float yAlea = uniform(0, m, localState);
		if (yAlea < f(xAlea)) intraThreadSum++;
		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = intraThreadSum;
}

/*
 * I have to used this hack to avoid multiple definition
 */
__global__ void setup_kernel_rand_montecarlo(hiprandState* ptrDevTabGeneratorThread, int deviceId)
    {
    int tid = Indice1D::tid();

    // Customisation du generator: Proposition (au lecteur de faire mieux)
    // Contrainte : Doit etre diff�rent d'un GPU � l'autre
    int deltaSeed = deviceId * INT_MAX;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;

    int seed = 1234 + deltaSeed;    // deviceId+tid;
    int sequenceNumber = tid + deltaSequence;    // + tid;
    int offset = deltaOffset;

    //Each thread gets same seed , a different sequence number , no offset
    hiprand_init(seed, sequenceNumber, offset, &ptrDevTabGeneratorThread[tid]);
    }


