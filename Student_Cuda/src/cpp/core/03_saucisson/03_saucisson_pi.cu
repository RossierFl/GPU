#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "UtilsCuda.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  bool isPiGPU_Ok(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__  static float computePi(int n);
__global__ static void computePiGPU(int n,float* ptrDevResult, int lenght);
__device__ void reduce_intra_thread(float* tabSM, int n);
//__device__ void reduce_inter_block(float* tabSM, float* ptrDevResult);
//__device__ void reduce_intra_block(float* tabSM, int length);
//__device__ void ecrasement(float* tabSM, int half);
__device__ float fpi(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  bool isPiGPU_Ok(int n)
    {
    cout << endl << "[PI Cuda 2]" << endl;


    float sumPiGPU = computePi(n);

    //cout <<"\n[GPUPI] "<< sumPiGPU << endl;
    printf("[GPUPI] %.10f\n",sumPiGPU);

    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ float computePi(int n)
    {
    float res = 0.0;
    float* ptrRes=&res;	// on host (CPU)
    float* ptrDevRes; 	// on device (GPU)
    // Specifier nb thread : ici 1 thread au total !
    dim3 dg = dim3(32,1,1);
    dim3 db = dim3(64,1,1);

    // Debug
     Device::checkDimError(dg,db);

    size_t size=sizeof(float);
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size)) ;
    HANDLE_ERROR(hipMemcpy(ptrRes, ptrDevRes, size, hipMemcpyDeviceToHost));// Device -> Host
    int length = db.x;
    size_t sizeTabSM = sizeof(float)*length;
    assert((length&(length-1))==0&&"Whoops, length must be a power of two !");
    computePiGPU<<<dg,db,sizeTabSM>>>(n, ptrDevRes,length); // asynchrone !!
    Device::checkKernelError("computePiGPU"); // facultatif

    //v1
    Device::synchronize();// Pour printf sur  GPU

    //v2
   // hipDeviceSynchronize(); // Pour printf sur  GPU

    // memoryManagement => barrier de synchronisation
    HANDLE_ERROR(hipMemcpy(ptrRes, ptrDevRes, size, hipMemcpyDeviceToHost));// Device -> Host
    HANDLE_ERROR(hipFree(ptrDevRes)); // device dispose memory in (*)
    double DX = 1.0/(double)n;
    return res*DX;
    }



__global__ void computePiGPU(int n, float* ptrDevRes,int lenght)
    {

    //une instance par block
      //init tab
      const int TID_LOCAL = Indice1D::tidLocalBlock();
      extern __shared__ float tabSM[];

      //UtilsCUDA::initialise(tabSM,lenght); //pas vraiment utile ici mais a garder

      reduce_intra_thread(tabSM, n);
      __syncthreads;
      UtilsCUDA::reduce_intra_block(tabSM);
      UtilsCUDA::reduce_inter_block(tabSM,ptrDevRes);
  }

__device__ void reduce_intra_thread(float* tabSM, int n){
  const int NBTHREAD = Indice1D::nbThread();
      const int TID = Indice1D::tid();
      const int TID_LOCAL = Indice1D::tidLocalBlock();
      const double DX = 1.0/(double)n;

      int s = TID;
      float sumCurrentThread = 0.0;
      float xs = 0.0;
      while(s<n){
	  xs = s * DX;
          sumCurrentThread += fpi(xs);
          s+=NBTHREAD;
      }
      //printf("currentSum=%f PI=%f\n in reduce intrathread", sumCurrentThread,sumCurrentThread*DX);
      //cout<<"currentSum"<<sumCurrentThread<<endl;
      tabSM[TID_LOCAL] = sumCurrentThread;
}





  __device__ float fpi(float x)
      {
      return 4.0 / (1.0 + x * x);
      }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

