#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdio.h>

#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "00_pi_tools.h"
#include "reduction.h"

// #define DEBUG 1

static __global__ void kernel(float* ptrDevPiGM, const int n, const int nTabSM);
static __device__ void reduceIntraThread(float* tabSM, int const n);
static __device__ float fpi(float x);
static double piCuda(const int n);
bool usePI();

/**
 * Kernel used to compute PI
 *
 * @param ptrDevPiGM location to save the final result of PI
 * @param n number of iterations
 */
__global__ void kernel(float* ptrDevPiGM, const int n, const int nTabSM) {
	// @formatter:off
	extern __shared__ float tabSM[];// 1 instance per block !
	// @formatter:on

	// Init
	initTabSM(tabSM, nTabSM, 0);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After init");
#endif

	__syncthreads(); // TODO useless?

	// reduceIntraThread
	reduceIntraThread(tabSM, n);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After reduceIntraThread");
#endif

	__syncthreads(); // TODO is really necessary ??

	// reduceIntraBlock
	reduceIntraBlock(tabSM, nTabSM);
#ifdef DEBUG
	debugTabSM(tabSM, nTabSM, "After reduceIntraBlock");
#endif

	__syncthreads(); // TODO is really necessary ??

	// reduceInterBlock
	reduceInterBlock(tabSM, ptrDevPiGM);
#ifdef DEBUG
	debugTabGM(ptrDevPiGM, 1, "PI");
#endif
}

__device__ void reduceIntraThread(float* tabSM, const int N) {
	const int TID = Indice1D::tid();
	const int TID_LOCAL = Indice1D::tidLocal();
	const int NB_THREAD = Indice1D::nbThread();

	const float DX = 1.0 / N;
	float intraThreadSum = 0;
	int s = TID;
	while (s < N) {
		float xi = s * DX;

		intraThreadSum += fpi(xi);

		s += NB_THREAD;
	}
	tabSM[TID_LOCAL] = intraThreadSum * DX;
}

__device__ float fpi(float x) {
	return 4 / (1 + x * x);
}

double piCuda(const int n) {
	// résultat final de Pi en RAM
	float resultRAM = 0;

	// résultat final de Pi sur le GPU après réduction
	float* ptrDevPiGM = NULL;
	size_t size = sizeof(float);
	int nThreadPerBlock = 4;
	size_t sizeTabSM = sizeof(float) * nThreadPerBlock;
	HANDLE_ERROR(hipMalloc(&ptrDevPiGM, size));
	HANDLE_ERROR(hipMemset(ptrDevPiGM, 0, size));

	// Lancement du kernel
	dim3 dg(1, 1, 1); // TODO to optimize
	dim3 db(nThreadPerBlock, 1, 1); // TODO to optimize
#ifdef DEBUG
	Device::checkDimError(dg, db);
	Device::checkDimOptimiser(dg, db);
#endif
	kernel<<<dg,db,sizeTabSM>>>(ptrDevPiGM, n, nThreadPerBlock); // asynchronous
	Device::checkKernelError("Kernel error");

	// Récupération du résultat
	HANDLE_ERROR(hipMemcpy(&resultRAM, ptrDevPiGM, size, hipMemcpyDeviceToHost)); // barrière de synchronisation

	return (double) resultRAM;
}

bool usePI() {

	printf("\n[PI]\n");

	const int N = 2000000;
	return isAlgoPI_OK(piCuda, N, "Pi cuda");
}
