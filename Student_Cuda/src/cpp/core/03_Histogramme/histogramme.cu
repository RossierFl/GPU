#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdio>
#include <curandTools.h>

#include "cuda_utils.h"
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "AleaTools.h"

#define HISTOGRAMME_SIZE 512

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isHistogramme_Ok(int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ static bool computeHistogramme(int n);
__global__ static void computeHistogrammeGPU(int n,bool* ptrDevResult, int* ptrDevData, int* ptrDevHistogramme,int length);
__device__ void reduceIntraThreadHistogramme(int* ptrDevData, int* tabSM, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isHistogramme_Ok(int n)
    {
    cout << endl << "[Histogramme Cuda]" << endl;
    int histogrammeGPU = computeHistogramme(n);
    printf("[GPU HISTOGRAMME] : %d\n",histogrammeGPU);
    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ void swap(int* ptrHostData, int size)
    {
    //fill the data
    for(int i=0;i<size;i++)
	{
	ptrHostData[i]=i%HISTOGRAMME_SIZE;
	}
    //swap
    AleaTools at;
    for(int i=0;i<size*size;i++)
	{
	int a = at.uniformeAB(0,size-1);
	int b = at.uniformeAB(0,size-1);
	int tmpA = ptrHostData[a];
	int tmpB = ptrHostData[b];
	ptrHostData[a]=tmpB;
	ptrHostData[b] = tmpA;
	}
    }

__host__ bool computeHistogramme(int n)
    {
    bool res = false;
    bool* ptrRes=&res;
    bool* ptrDevRes;
    dim3 dg = dim3(16,1,1);
    dim3 db = dim3(32,1,1);


    size_t size_res=sizeof(bool);
    HANDLE_ERROR(hipMalloc((void**) &ptrDevRes, size_res)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevRes,0,size_res)) ;

    int* ptrHostData = new int[n];
    swap(ptrHostData,n);

    int* ptrDevData;
    size_t sizeTabData = sizeof(int)*n;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevData, sizeTabData)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemcpy(ptrDevData,ptrHostData,sizeTabData,hipMemcpyHostToDevice)) ;

    int* ptrDevHistogramme;
    size_t sizeTabHisto = sizeof(int)*HISTOGRAMME_SIZE;
    HANDLE_ERROR(hipMalloc((void**) &ptrDevHistogramme, sizeTabHisto)); // Device memory allocation (*)
    HANDLE_ERROR(hipMemset(ptrDevHistogramme,0,sizeTabHisto));

    int length = HISTOGRAMME_SIZE;
    size_t sizeSM = sizeof(int)*length;

    computeHistogrammeGPU<<<dg,db,sizeSM>>>(n,ptrDevRes, ptrDevData,ptrDevHistogramme,HISTOGRAMME_SIZE);

    hipDeviceSynchronize();

    HANDLE_ERROR(hipMemcpy(ptrRes, ptrDevRes, size_res, hipMemcpyDeviceToHost));// Device -> Host

    // avoid memory leak, free memory
    HANDLE_ERROR(hipFree(ptrDevRes));
    HANDLE_ERROR(hipFree(ptrDevData));
    HANDLE_ERROR(hipFree(ptrDevHistogramme));
    return res;
    }

__device__ bool checkHistogramme(int* ptrDevResult,int length,int n){
    const int NBTHREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    int s = TID;
    bool isCorrect = true;
    int correctResult = n/HISTOGRAMME_SIZE;
    while(isCorrect&&s<length)
	{
	if(ptrDevResult[s]!=correctResult)
	    isCorrect=false;
	s+=NBTHREAD;
	}
    __syncthreads();
    return isCorrect;
}

__global__ void computeHistogrammeGPU(int n,bool* ptrDevResult, int* ptrDevData, int* ptrDevHistogramme,int length)
    {
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    extern __shared__ int tabSM[];
    CUDAUtils::initialize(tabSM,length);
    CUDAUtils::initialize(ptrDevHistogramme,length);
    reduceIntraThreadHistogramme(ptrDevData,tabSM, n); // add suffix, avoid multiple definition error
    __syncthreads();
    CUDAUtils::reduceInterBlock_v2(tabSM,ptrDevHistogramme,HISTOGRAMME_SIZE);
    *ptrDevResult = checkHistogramme(ptrDevHistogramme,length,n);
    }

__device__ void reduceIntraThreadHistogramme(int* ptrDevData, int* tabSM, int n)
    {
    const int NBTHREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocalBlock();
    int s = TID;
    while(s<n){
	int crtValue= ptrDevData[s];
	atomicAdd(&tabSM[crtValue],1);
	s+=NBTHREAD;
    }
    }



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

